#include <string>

#include "main.h"

int main(int argc, char** argv) {
    std::string config_file = "config.ini";
    if (argc > 1) {
        config_file = argv[1];
    }

    INIReader config(config_file);
    if (config.ParseError() < 0) {
        spdlog::info("Can't load {}", config_file);
        return 1;
    }

    // comm config
    std::string input_file = config.Get("comm", "dataset", "UNKNOWN");
    std::string log_level_str = config.Get("comm", "log_level", "info");
    spdlog::level::level_enum log_level = switch_log_level(log_level_str);
    spdlog::set_level(log_level);

    int device = config.GetInteger("comm", "device", 0);
    HRR(hipSetDevice(device));
    spdlog::info("Use device {}", device);

    // read dataset and preprocessing dataset
    CPUGraph cpu_graph(input_file);

    if (cpu_graph.edge_count > constant_comm::kMaxGraphEdgeCount) {
        spdlog::info("Input graph is too large! Exit ...");
        return 1;
    }

    // GPUGraph dcsr(cpu_graph);

    {
        Csr2DcsrDataTransfer cddt(input_file, &cpu_graph);
        cddt.transfer();
        GPUGraph& dcsr = cddt.d_graph;
        tc::approach::Polak::start_up(config, dcsr, argc, argv);
        tc::approach::TriCore::start_up(config, dcsr, argc, argv);
        tc::approach::HINDEX::start_up(config, dcsr, argc, argv);
        tc::approach::Green::start_up(config, dcsr, argc, argv);
        tc::approach::Hu::start_up(config, dcsr, argc, argv);
        tc::approach::TC_Check::start_up(config, dcsr, argc, argv);
        tc::approach::Fox::start_up(config, dcsr, argc, argv);
    }

    {
        Csr2RidDcsrDataTransfer crdt(input_file, &cpu_graph);
        crdt.transfer();
        GPUGraph& riddcsr = crdt.d_graph;
        tc::approach::Bisson::start_up(config, riddcsr, argc, argv);
        tc::approach::GroupTC::start_up(config, riddcsr, argc, argv);
        tc::approach::GroupTC_OPT::start_up(config, riddcsr, argc, argv);
    }

    {
        Csr2TrustDcsrDataTransfer ctdt(input_file, &cpu_graph);
        ctdt.transfer();
        GPUGraph& trustdcsr = ctdt.d_graph;
        tc::approach::GroupTC_HASH::start_up(config, trustdcsr, argc, argv);
        tc::approach::TRUST::start_up(config, trustdcsr, argc, argv);
    }

    return 0;
}


spdlog::level::level_enum switch_log_level(std::string log_level_str) {
    if (log_level_str == "trace") {
        return spdlog::level::trace;
    } else if (log_level_str == "debug") {
        return spdlog::level::debug;
    } else if (log_level_str == "info") {
        return spdlog::level::info;
    } else if (log_level_str == "warn") {
        return spdlog::level::warn;
    } else if (log_level_str == "err") {
        return spdlog::level::err;
    } else if (log_level_str == "critical") {
        return spdlog::level::critical;
    } else if (log_level_str == "off") {
        return spdlog::level::off;
    } else {
        return spdlog::level::info;
    }
}
