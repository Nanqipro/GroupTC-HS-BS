#include "hip/hip_runtime.h"
#include "cuda_graph_comm.h"

__global__ void cuda_graph_comm::cal_degree_and_zip_edge(int edge_count, int vertex_count, int *d_degree_arr, int *d_edge_arr, int *d_src_arr, int *d_adj_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int src = d_src_arr[i];
    int adj = d_adj_arr[i];
    d_edge_arr[i * 2] = src;
    d_edge_arr[i * 2 + 1] = adj;

    atomicAdd(d_degree_arr + src, 1);
    atomicAdd(d_degree_arr + adj, 1);
}

__global__ void cuda_graph_comm::redirect_edge(int edge_count, int vertex_count, int *d_degree_arr, int *d_edge_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int adj = d_edge_arr[i * 2];
    int src = d_edge_arr[i * 2 + 1];
    // redirect edge
    if (d_degree_arr[src] > d_degree_arr[adj] || (d_degree_arr[src] == d_degree_arr[adj] && src > adj))
    {
        d_edge_arr[i * 2] = src;
        d_edge_arr[i * 2 + 1] = adj;
    }
}

__global__ void cuda_graph_comm::redirect_edge(int edge_count, int vertex_count, int *d_degree_arr, int *d_src_arr, int *d_adj_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int src = d_src_arr[i];
    int dst = d_adj_arr[i];
    // redirect edge
    if (d_degree_arr[src] > d_degree_arr[dst] || (d_degree_arr[src] == d_degree_arr[dst] && src > dst))
    {
        d_adj_arr[i] = src;
        d_src_arr[i] = dst;
    }
}

__global__ void cuda_graph_comm::unzip_edge(int edge_count, int vertex_count, int *d_edge_arr, int *d_src_arr, int *d_adj_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    d_src_arr[i] = d_edge_arr[i * 2 + 1];
    d_adj_arr[i] = d_edge_arr[i * 2];
}

__global__ void cuda_graph_comm::recal_offset(int edge_count, int vertex_count, int *d_src_arr, index_t *d_offset_arr)
{
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i <= edge_count; i += step)
    {
        int prev = i > 0 ? d_src_arr[i - 1] : -1;
        int next = i < edge_count ? d_src_arr[i] : vertex_count;
        // 前一个元素小于后一个元素，才有可能出现 offset 的计算
        for (int j = prev + 1; j <= next; ++j)
            d_offset_arr[j] = i;
    }
}

__global__ void cuda_graph_comm::record_id(int edge_count, int vertex_count, int *d_id_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count)
    {
        return;
    }
    d_id_arr[i] = i;
}

__global__ void cuda_graph_comm::map_id(int edge_count, int vertex_count, int *d_id_arr, int *d_id_map_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count)
    {
        return;
    }
    d_id_map_arr[d_id_arr[i]] = i;
}

__global__ void cuda_graph_comm::redirect_edge_and_reassign_id(int edge_count, int vertex_count, int *d_id_map_arr, int *d_edge_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int src = d_id_map_arr[d_edge_arr[i * 2]];
    int adj = d_id_map_arr[d_edge_arr[i * 2 + 1]];
    if (src > adj)
    {
        int temp = src;
        src = adj;
        adj = temp;
    }
    d_edge_arr[i * 2] = adj;
    d_edge_arr[i * 2 + 1] = src;
}

__global__ void cuda_graph_comm::cal_degree(int edge_count, int vertex_count, int *d_degree_arr, int *d_src_arr, int *d_adj_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int src = d_src_arr[i];
    int adj = d_adj_arr[i];

    atomicAdd(d_degree_arr + src, 1);
    atomicAdd(d_degree_arr + adj, 1);
}

__global__ void cuda_graph_comm::cal_src_out_degree(int edge_count, int vertex_count, int *d_degree_arr, int *d_src_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    atomicAdd(d_degree_arr + d_src_arr[i], 1);
}

__global__ void cuda_graph_comm::record_id_and_part_graph_by_degree(int edge_count, int vertex_count, int *d_id_arr, int *d_degree_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count)
    {
        return;
    }
    d_id_arr[i] = i;
    int degree = d_degree_arr[i];
    if (degree < 2)
    {
        degree = 2;
    }
    else if (degree <= 100)
    {
        degree = 1;
    }
    else
    {
        degree = 0;
    }
    d_degree_arr[i] = degree;
}

__global__ void cuda_graph_comm::reassign_id(int edge_count, int vertex_count, int *d_id_map_arr, int *d_src_arr, int *d_adj_arr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    d_src_arr[i] = d_id_map_arr[d_src_arr[i]];
    d_adj_arr[i] = d_id_map_arr[d_adj_arr[i]];
}