#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <fstream>
#include <cstdio>
#include <vector>
#include <sstream>
#include <cmath>
#include <string>
#include "csr2trust_dcsr_data_transfer.h"
#include "cuda_graph_comm.h"
#include "spdlog/spdlog.h"
#include "constant_comm.h"

void Csr2TrustDcsrDataTransfer::transfer()
{

    vertex_t *d_degree_arr;
    vertex_t *d_id_arr;
    vertex_t *d_id_map_arr;

    vertex_t vertex_count = d_graph.vertex_count;
    vertex_t edge_count = d_graph.edge_count;
    vertex_t *d_src_arr = d_graph.src_list;
    vertex_t *d_adj_arr = d_graph.adj_list;
    index_t *d_offset_arr = d_graph.beg_pos;

    vertex_t *h_src_arr = h_graph.src_list;
    vertex_t *h_adj_arr = h_graph.adj_list;
    index_t *h_offset_arr = h_graph.beg_pos;

    size_t size_degree_arr = sizeof(vertex_t) * vertex_count;
    size_t size_offset_arr = sizeof(index_t) * (vertex_count + 1);
    size_t size_src_arr = sizeof(vertex_t) * edge_count;

    for (int i = 0; i < 10; i++)
    {
        spdlog::debug("arr[{0}]: {1}", i, h_offset_arr[i]);
    }

    HRR(hipMalloc((void **)&d_degree_arr, size_degree_arr));
    HRR(hipMalloc((void **)&d_id_arr, size_degree_arr));
    HRR(hipMalloc((void **)&d_id_map_arr, size_degree_arr));

    HRR(hipMemset(d_degree_arr, 0, size_degree_arr));

    int block_size = 1024;
    int vertex_grid_size = (vertex_count - 1) / block_size + 1;
    int edge_grid_size = (edge_count - 1) / block_size + 1;

    int max_degree = compute_max_degree();

    spdlog::info("CSR2TrustDCSR graph transfer start, graph max degree is {}", max_degree);

    double t_start = wtime();

    int iterations = constant_comm::kTransferIterations;
    for (int k = 0; k < iterations; k++)
    {
        cuda_graph_comm::cal_degree<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_src_arr, d_adj_arr);
        HRR(hipDeviceSynchronize());

        cuda_graph_comm::redirect_edge<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_src_arr, d_adj_arr);
        HRR(hipDeviceSynchronize());

        HRR(hipMemset(d_degree_arr, 0, size_degree_arr));
        cuda_graph_comm::cal_src_out_degree<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_src_arr);
        HRR(hipDeviceSynchronize());

        cuda_graph_comm::record_id_and_part_graph_by_degree<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_id_arr, d_degree_arr);
        HRR(hipDeviceSynchronize());

        thrust::device_ptr<vertex_t> d_id_ptr((vertex_t *)d_id_arr);
        thrust::sort_by_key(d_degree_arr, d_degree_arr + vertex_count, d_id_ptr);

        cuda_graph_comm::map_id<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_id_arr, d_id_map_arr);
        HRR(hipDeviceSynchronize());

        cuda_graph_comm::reassign_id<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_id_map_arr, d_src_arr, d_adj_arr);
        HRR(hipDeviceSynchronize());

        thrust::device_ptr<vertex_t> d_dst_ptr((vertex_t *)d_adj_arr);
        thrust::device_ptr<vertex_t> d_src_ptr((vertex_t *)d_src_arr);
        thrust::sort_by_key(d_src_ptr, d_src_ptr + edge_count, d_adj_arr);

        cuda_graph_comm::recal_offset<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_src_arr, d_offset_arr);
        HRR(hipDeviceSynchronize());
    }
    double t_end = wtime();

    spdlog::info("iterate {0} times, avg time consumption {1:.6f} s", iterations, (t_end - t_start) / iterations);

    cuda_graph_comm::check_array("d_src_arr", d_src_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_adj_arr", d_adj_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_offset_arr", d_offset_arr, vertex_count + 1, 0, 10);

    HRR(hipMemcpy(h_offset_arr, d_offset_arr, size_offset_arr, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(h_src_arr, d_src_arr, size_src_arr, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(h_adj_arr, d_adj_arr, size_src_arr, hipMemcpyDeviceToHost));

    max_degree = compute_max_degree();
    spdlog::info("CSR2TrustDCSR graph transfer finished, graph max degree is {}", max_degree);

    hipFree(d_degree_arr);
    hipFree(d_id_arr);
    hipFree(d_id_map_arr);
}
