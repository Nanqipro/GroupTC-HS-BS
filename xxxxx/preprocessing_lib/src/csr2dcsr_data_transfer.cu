#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <fstream>
#include <cstdio>
#include <vector>
#include <sstream>
#include <cmath>
#include <string>
#include "csr2dcsr_data_transfer.h"
#include "cuda_graph_comm.h"
#include "spdlog/spdlog.h"
#include "constant_comm.h"

void Csr2DcsrDataTransfer::transfer()
{

    vertex_t *d_degree_arr;
    vertex_t *d_edge_arr;

    vertex_t vertex_count = d_graph.vertex_count;
    vertex_t edge_count = d_graph.edge_count;
    vertex_t *d_src_arr = d_graph.src_list;
    vertex_t *d_adj_arr = d_graph.adj_list;
    index_t *d_offset_arr = d_graph.beg_pos;

    vertex_t *h_src_arr = h_graph.src_list;
    vertex_t *h_adj_arr = h_graph.adj_list;
    index_t *h_offset_arr = h_graph.beg_pos;

    size_t size_degree_arr = sizeof(vertex_t) * vertex_count;
    size_t size_offset_arr = sizeof(index_t) * (vertex_count + 1);
    size_t size_src_arr = sizeof(vertex_t) * edge_count;
    size_t size_edge_arr = size_src_arr * 2;

    for (int i = 0; i < 10; i++)
    {
        spdlog::debug("arr[{0}]: {1}", i, h_offset_arr[i]);
    }

    HRR(hipMalloc((void **)&d_degree_arr, size_degree_arr));
    HRR(hipMalloc((void **)&d_edge_arr, size_edge_arr));

    HRR(hipMemset(d_degree_arr, 0, size_degree_arr));

    int block_size = 1024;
    int grid_size = (edge_count - 1) / block_size + 1;

    int max_degree = compute_max_degree();

    spdlog::info("CSR2DCSR graph transfer start, graph max degree is {}", max_degree);

    double t_start = wtime();

    int iterations = constant_comm::kTransferIterations;
    for (int k = 0; k < iterations; k++)
    {
        cuda_graph_comm::cal_degree_and_zip_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_edge_arr, d_src_arr, d_adj_arr);
        HRR(hipDeviceSynchronize());

        cuda_graph_comm::redirect_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_edge_arr);
        HRR(hipDeviceSynchronize());

        thrust::device_ptr<uint64_t> sort_ptr((uint64_t *)d_edge_arr);
        thrust::sort(sort_ptr, sort_ptr + edge_count);

        cuda_graph_comm::unzip_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_edge_arr, d_src_arr, d_adj_arr);
        HRR(hipDeviceSynchronize());

        cuda_graph_comm::recal_offset<<<grid_size, block_size>>>(edge_count, vertex_count, d_src_arr, d_offset_arr);
        HRR(hipDeviceSynchronize());
    }
    double t_end = wtime();

    spdlog::info("iterate {0} times, avg time consumption {1:.6f} s", iterations, (t_end - t_start) / iterations);

    cuda_graph_comm::check_array("d_src_arr", d_src_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_adj_arr", d_adj_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_offset_arr", d_offset_arr, vertex_count + 1, 0, 10);

    HRR(hipMemcpy(h_offset_arr, d_offset_arr, size_offset_arr, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(h_src_arr, d_src_arr, size_src_arr, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(h_adj_arr, d_adj_arr, size_src_arr, hipMemcpyDeviceToHost));

    max_degree = compute_max_degree();
    spdlog::info("CSR2DCSR graph transfer finished, graph max degree is {}", max_degree);

    hipFree(d_degree_arr);
    hipFree(d_edge_arr);
}
