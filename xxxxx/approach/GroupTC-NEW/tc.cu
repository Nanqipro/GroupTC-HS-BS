#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include "../comm/cuda_comm.h"

#define block_bucketnum 256
#define subwarp_size 64
#define warp_step block_bucketnum / subwarp_size

int grid_size = NumberOfMPs() * 8;
// int block_size = WarpSize() * 2;

int edge_count;
int vertex_count;
int iterator_count = 100;

// __device__ int bin_search1(vertex_t *arr, int len, int val)
// {

// 	uint32_t Y;
// 	int32_t bot = 0;
// 	int32_t top = len - 1;
// 	int32_t r;
// 	while (top >= bot)
// 	{
// 		r = (top + bot) / 2;
// 		Y = arr[r];

// 		if (val == Y)
// 		{
// 			return 1;
// 		}
// 		if (val < Y)
// 		{
// 			top = r - 1;
// 		}
// 		else
// 		{
// 			bot = r + 1;
// 		}
// 	}
// 	return 0;
// }

// __device__ int bin_search(vertex_t *arr, int len, int val)
// {
// 	int ret = 0;
// 	int halfsize;
// 	int candidate;
// 	int temp = len;
// 	while (temp > 1)
// 	{
// 		halfsize = temp / 2;
// 		candidate = arr[ret + halfsize];
// 		ret += (candidate < val) ? halfsize : 0;
// 		temp -= halfsize;
// 	}
// 	ret += (arr[ret] < val);
// 	return ret < len && arr[ret] == val;
// }

__device__ int bin_search(vertex_t *arr, int len, int val, int &offset)
{
	int ret = 0;
	int halfsize;
	int candidate;
	int temp = len;
	while (temp > 1)
	{
		halfsize = temp / 2;
		candidate = arr[ret + halfsize];
		ret += (candidate < val) ? halfsize : 0;
		temp -= halfsize;
	}
	ret += (arr[ret] < val);
	offset = ret;
	return ret < len && arr[ret] == val;
}

__global__ void
grouptc_with_reduce(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, unsigned long long *GLOBAL_COUNT)
{

	// 共享内存中的 hashTable
	__shared__ int sh_tb_start[block_bucketnum];
	__shared__ int sh_tb_len[block_bucketnum];
	__shared__ int sh_ele_start[block_bucketnum];
	__shared__ int sh_ele_len[block_bucketnum];

	unsigned long long P_counter = 0;

	int bid = blockIdx.x;
	int tid = threadIdx.x;

	for (int i = bid * block_bucketnum; i < edge_count; i += gridDim.x * block_bucketnum)
	{
		// if (i + tid < edge_count)
		// {
		// 	int src = src_list[i + tid];
		// 	int dst = adj_list[i + tid];
		// 	int temp;

		// 	int tb_start, tb_len, ele_start, ele_len;
		// 	tb_start = i + tid + 1;
		// 	// tb_start = beg_pos[src];
		// 	tb_len = beg_pos[src + 1] - tb_start;
		// 	ele_start = beg_pos[dst];
		// 	ele_len = beg_pos[dst + 1] - ele_start;

		// 	if (tb_len * 2 < ele_len)
		// 	{
		// 		temp = tb_start;
		// 		tb_start = ele_start;
		// 		ele_start = temp;

		// 		temp = tb_len;
		// 		tb_len = ele_len;
		// 		ele_len = temp;
		// 	}

		// 	sh_tb_start[tid] = tb_start;
		// 	sh_tb_len[tid] = tb_len;
		// 	sh_ele_start[tid] = ele_start;
		// 	sh_ele_len[tid] = ele_len;
		// }

		// __syncthreads();

		int now = i + tid / subwarp_size;
		int end = min(edge_count, block_bucketnum + i);
		int workid = tid % subwarp_size;
		int offset = 0;
		int last_now = -1;

		// 获取二跳邻居节点
		int neighbor = adj_list[now];
		int neighbor_start = beg_pos[neighbor];
		int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
		while (now < end)
		{
			// 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
			while (now < end && workid >= neighbor_degree)
			{
				now += warp_step;
				if (now < end)
				{
					workid -= neighbor_degree;
					neighbor = adj_list[now];
					neighbor_start = beg_pos[neighbor];
					neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
				}
			}

			if (now < end)
			{
				// offset = last_now == now ? offset : 0;
				int src = src_list[now];
				int src_start = beg_pos[src];
				int src_degree = beg_pos[src + 1] - src_start;

				P_counter += bin_search(adj_list + src_start, src_degree, adj_list[neighbor_start + workid], offset);

				// P_counter += bin_search(adj_list + sh_tb_start[now] , sh_tb_len[now], adj_list[sh_ele_start[now] + workid]);
				// last_now = now;
			}
			workid += subwarp_size;
		}
		__syncthreads();
	}

	GLOBAL_COUNT[bid * block_bucketnum + tid] = P_counter;
}

__global__ void
grouptc_with_atomic(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, unsigned long long *GLOBAL_COUNT)
{

	// 共享内存中的 hashTable
	__shared__ int sh_tb_start[block_bucketnum];
	__shared__ int sh_tb_len[block_bucketnum];
	__shared__ int sh_ele_start[block_bucketnum];
	__shared__ int sh_ele_len[block_bucketnum];
	__shared__ unsigned long long B_counter;

	unsigned long long P_counter = 0;

	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if (tid == 0)
	{
		B_counter = 0;
	}
	__syncthreads();

	for (int i = bid * block_bucketnum; i < edge_count; i += gridDim.x * block_bucketnum)
	{
		if (i + tid < edge_count)
		{
			int src = src_list[i + tid];
			int dst = adj_list[i + tid];
			int temp;

			int tb_start, tb_len, ele_start, ele_len;
			tb_start = i + tid + 1;
			// tb_start = beg_pos[src];
			tb_len = beg_pos[src + 1] - tb_start;
			ele_start = beg_pos[dst];
			ele_len = beg_pos[dst + 1] - ele_start;

			if (tb_len * 2 < ele_len)
			{
				temp = tb_start;
				tb_start = ele_start;
				ele_start = temp;

				temp = tb_len;
				tb_len = ele_len;
				ele_len = temp;
			}

			sh_tb_start[tid] = tb_start;
			sh_tb_len[tid] = tb_len;
			sh_ele_start[tid] = ele_start;
			sh_ele_len[tid] = ele_len;
		}

		__syncthreads();

		int now = tid / subwarp_size;
		int end = min(edge_count - i, block_bucketnum);
		int workid = tid % subwarp_size;
		int offset = 0;
		int last_now = -1;

		// 获取二跳邻居节点
		int neighbor_degree = sh_ele_len[now];
		while (now < end)
		{
			// 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
			while (now < end && workid >= neighbor_degree)
			{
				now += warp_step;
				if (now < end)
				{
					workid -= neighbor_degree;
					neighbor_degree = sh_ele_len[now];
				}
			}

			if (now < end)
			{
				offset = last_now == now ? offset : 0;
				P_counter += bin_search(adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
				last_now = now;
			}
			workid += subwarp_size;
		}
		__syncthreads();
	}

	P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 16);
	P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 8);
	P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 4);
	P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 2);
	P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 1);

	if (tid % 32 == 0)
	{
		atomicAdd(&B_counter, P_counter);
	}
	__syncthreads();
	if (tid == 0)
	{
		atomicAdd(GLOBAL_COUNT, B_counter);
	}
}

void gpu_run_with_reduce(vertex_t *d_source, vertex_t *d_adj, index_t *d_offset, unsigned long long *results)
{

	double t_start, total_kernel_use = 0;
	uint64_t count;
	hipProfilerStop();
	// hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
	for (int i = 0; i < iterator_count; i++)
	{
		hipMemset(results, grid_size * block_bucketnum * sizeof(unsigned long long), 0);
		t_start = wtime();
		hipProfilerStart();
		grouptc_with_reduce<<<grid_size, block_bucketnum>>>(d_source, d_adj, d_offset, edge_count, vertex_count, results);
		HRR(hipDeviceSynchronize());
		hipProfilerStop();
		thrust::device_ptr<unsigned long long> ptr(results);
		count = thrust::reduce(ptr, ptr + (grid_size * block_bucketnum));
		total_kernel_use += wtime() - t_start;
		if (i == 0)
		{
			if (total_kernel_use > 0.1 && iterator_count != 1)
			{
				iterator_count = 10;
			}
		}
	}

	printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
	printf("triangle count %ld \n\n", count);
}

void gpu_run_with_atomic(vertex_t *d_source, vertex_t *d_adj, index_t *d_offset)
{

	double t_start, total_kernel_use = 0;
	unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long));
	unsigned long long *results;

	HRR(hipMalloc(&results, sizeof(unsigned long long)));

	for (int i = 0; i < iterator_count; i++)
	{
		HRR(hipMemset(results, 0, sizeof(unsigned long long)));
		hipProfilerStart();
		t_start = wtime();
		grouptc_with_atomic<<<grid_size, block_bucketnum>>>(d_source, d_adj, d_offset, edge_count, vertex_count, results);
		HRR(hipDeviceSynchronize());
		total_kernel_use += wtime() - t_start;
		hipProfilerStop();
		if (i == 0)
		{
			if (total_kernel_use > 0.1 && iterator_count != 1)
			{
				iterator_count = 10;
			}
		}
	}
	HRR(hipMemcpy(counter, results, sizeof(unsigned long long), hipMemcpyDeviceToHost));
	printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
	printf("triangle count %ld \n\n", counter[0]);

	free(counter);
	HRR(hipFree(results));
}

void TC_gpu_with_reduce(graph *graph_d)
{

	long int edge_size = sizeof(vertex_t) * edge_count;
	long int offset_size = sizeof(index_t) * (vertex_count + 1);

	vertex_t *d_source, *d_adj;
	index_t *d_offset;
	unsigned long long *results;
	HRR(hipMalloc(&d_source, edge_size));
	HRR(hipMalloc(&d_adj, edge_size));
	HRR(hipMalloc(&d_offset, offset_size));
	// HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));
	// HRR(hipMalloc(&results, edge_count * sizeof(unsigned long long)));

	HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	HRR(hipMemcpy(d_source, graph_d->source_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj, graph_d->adj_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_offset, graph_d->beg_pos, offset_size, hipMemcpyHostToDevice));

	gpu_run_with_reduce(d_source, d_adj, d_offset, results);

	HRR(hipFree(d_source));
	HRR(hipFree(d_adj));
	HRR(hipFree(d_offset));
	HRR(hipFree(results));
}

void TC_gpu_with_atomic(graph *graph_d)
{

	long int edge_size = sizeof(vertex_t) * edge_count;
	long int offset_size = sizeof(index_t) * (vertex_count + 1);

	vertex_t *d_source, *d_adj;
	index_t *d_offset;
	HRR(hipMalloc(&d_source, edge_size));
	HRR(hipMalloc(&d_adj, edge_size));
	HRR(hipMalloc(&d_offset, offset_size));

	HRR(hipMemcpy(d_source, graph_d->source_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj, graph_d->adj_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_offset, graph_d->beg_pos, offset_size, hipMemcpyHostToDevice));

	gpu_run_with_atomic(d_source, d_adj, d_offset);

	HRR(hipFree(d_source));
	HRR(hipFree(d_adj));
	HRR(hipFree(d_offset));
}

int main(int argc, char **argv)
{
	string file = argv[1];
	if (argc >= 4)
	{
		iterator_count = atoi(argv[3]);
	}
	if (argc >= 5)
	{
		grid_size = atoi(argv[4]);
	}
	hipSetDevice(atoi(argv[2]));
	graph *graph_d = readGraph(file);
	edge_count = graph_d->edge_count;
	vertex_count = graph_d->vertex_count;

	cout << "dataset\t" << file << endl;
	cout << "Number of nodes: " << vertex_count
		 << ", number of edges: " << edge_count << endl;

	if (edge_count > 1e8)
	{
		grid_size = edge_count / block_bucketnum / 20;
		TC_gpu_with_reduce(graph_d);
	}
	else
	{
		grid_size = NumberOfMPs() * 8;
		TC_gpu_with_atomic(graph_d);
	}

	return 0;
}
