#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "../comm/cuda_comm.h"

#define block_bucketnum 256
#define subwarp_size 32
#define warp_step block_bucketnum / subwarp_size

int iter_batch = 0;
int grid_size = NumberOfMPs() * 8;
int block_size = WarpSize() * 2;

int edge_count;
int vertex_count;
int iterator_count = 100;

__device__ int bin_search1(vertex_t *arr, int len, int val)
{

	uint32_t Y;
	int32_t bot = 0;
	int32_t top = len - 1;
	int32_t r;
	while (top >= bot)
	{
		r = (top + bot) / 2;
		Y = arr[r];

		if (val == Y)
		{
			return 1;
		}
		if (val < Y)
		{
			top = r - 1;
		}
		else
		{
			bot = r + 1;
		}
	}
	return 0;
}

// __device__ int bin_search(vertex_t *arr, int len, int val)
// {
// 	int ret = 0;
// 	int halfsize;
// 	int candidate;
// 	int temp = len;
// 	while (temp > 1)
// 	{
// 		halfsize = temp / 2;
// 		candidate = arr[ret + halfsize];
// 		ret += (candidate < val) ? halfsize : 0;
// 		temp -= halfsize;
// 	}
// 	ret += (arr[ret] < val);
// 	return ret < len && arr[ret] == val;
// }

__device__ inline int bin_search(vertex_t *arr, int start, int end, int val, int &last_position)
{
	int len = end - start;
	arr += start;
	int ret = 0;
	int halfsize;
	int candidate;
	int temp = len;
	while (temp > 1)
	{
		halfsize = temp / 2;
		candidate = arr[ret + halfsize];
		ret += (candidate < val) ? halfsize : 0;
		temp -= halfsize;
	}
	ret += (arr[ret] < val);
	last_position = ret + start;
	return ret < len && arr[ret] == val;
}

__global__ void
grouptc(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, unsigned long long *GLOBAL_COUNT, int iter_batch)
{

	__shared__ int sh_tb_src[block_bucketnum];
	__shared__ int sh_tb_start[block_bucketnum];
	__shared__ int sh_tb_len[block_bucketnum];
	__shared__ int sh_ele_start[block_bucketnum];
	__shared__ int sh_ele_len[block_bucketnum];

	int bid = blockIdx.x;
	int tid = threadIdx.x;

	unsigned long long P_counter = 0;
	for (int i = bid * block_bucketnum; i < edge_count; i += gridDim.x * block_bucketnum)
	{
		if (i + tid < edge_count)
		{
			int src = src_list[i + tid];
			int dst = adj_list[i + tid];
			int temp;

			int tb_start, tb_len, ele_start, ele_len;
			tb_start = i + tid + 1;
			// tb_start = beg_pos[src];
			tb_len = beg_pos[src + 1] - tb_start;
			ele_start = beg_pos[dst];
			ele_len = beg_pos[dst + 1] - ele_start;

			if (tb_len * 2 < ele_len)
			{
				src = dst;
				temp = tb_start;
				tb_start = ele_start;
				ele_start = temp;

				temp = tb_len;
				tb_len = ele_len;
				ele_len = temp;
			}
			sh_tb_src[tid] = src;
			sh_tb_start[tid] = tb_start;
			sh_tb_len[tid] = tb_len;
			sh_ele_start[tid] = ele_start;
			sh_ele_len[tid] = ele_len;
		}

		__syncthreads();

		// int now = (tid / subwarp_size) * subwarp_size;
		// int end = min(edge_count - i, now + subwarp_size);
		int now = tid / subwarp_size;
		int end = min(edge_count - i, block_bucketnum);
		int workid = tid % subwarp_size;
		int last_position = 0;
		int last_src = -1;
		int last_val = 0;
		int last_search;

		// 获取二跳邻居节点
		int neighbor_degree = sh_ele_len[now];
		while (now < end)
		{
			// 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
			while (now < end && workid >= neighbor_degree)
			{
				// now++;
				now += warp_step;
				if (now < end)
				{
					workid -= neighbor_degree;
					neighbor_degree = sh_ele_len[now];
				}
			}

			if (now < end)
			{
				// int tb_start = sh_tb_start[now];
				// int tb_len = sh_tb_len[now];
				// offset = last_now == now ? offset : 0;
				int val = adj_list[sh_ele_start[now] + workid];

				// P_counter += val % 2;
				// for (int iterBinSearch = 0; iterBinSearch < iter_batch; iterBinSearch++)
				// {
				// if (bid == 0 && tid == 0)
				// {
				// 	printf("%d \n", (unsigned long long)((i + now + tid) * 1277 + iterBinSearch * (long long)1249) % edge_count);
				// }
				// P_counter += adj_list[(unsigned long long)((i + now + tid) * 1277 + iterBinSearch * (long long)1249) % tb_len + tb_start];
				// }/
				int src = sh_tb_src[now];
				int tb_start = sh_tb_start[now];
				int tb_end = tb_start + sh_tb_len[now];

				if (src == last_src)
				{
					if (val > last_val)
					{
						tb_start = max(last_position - 1, tb_start);
					}
					else if (val < last_val)
					{
						tb_end = last_position;
					}
				}

				if (src != last_src || val != last_val)
				{
					// P_counter += bin_search(adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
					last_search = bin_search(adj_list, tb_start, tb_end, val, last_position);
					last_src = src;
					last_val = val;
				}

				P_counter += last_search;
			}
			workid += subwarp_size;
		}
		__syncthreads();
	}

	// atomicAdd(GLOBAL_COUNT2, P_counter);
	GLOBAL_COUNT[bid * block_bucketnum + tid] = P_counter;
}

void gpu_run(vertex_t *d_source, vertex_t *d_adj, index_t *d_offset, unsigned long long *results)
{

	double t_start, total_kernel_use = 0;
	uint64_t count;
	hipProfilerStop();
	for (int i = 0; i < iterator_count; i++)
	{
		hipMemset(results, grid_size * block_bucketnum * sizeof(unsigned long long), 0);
		t_start = wtime();
		hipProfilerStart();
		grouptc<<<grid_size, block_bucketnum>>>(d_source, d_adj, d_offset, edge_count, vertex_count, results, iter_batch);
		HRR(hipDeviceSynchronize());
		hipProfilerStop();
		thrust::device_ptr<unsigned long long> ptr(results);
		count = thrust::reduce(ptr, ptr + (grid_size * block_bucketnum));
		double ee = wtime();
		total_kernel_use += ee - t_start;
		if (i == 0)
		{
			if (ee - t_start > 0.1 && iterator_count != 1)
			{
				iterator_count = 10;
			}
		}
	}

	printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
	printf("triangle count %ld \n\n", count);
}

void TC_gpu(graph *graph_d)
{

	long int edge_size = sizeof(vertex_t) * edge_count;
	long int offset_size = sizeof(index_t) * (vertex_count + 1);

	vertex_t *d_source, *d_adj;
	index_t *d_offset;
	unsigned long long *results;
	HRR(hipMalloc(&d_source, edge_size));
	HRR(hipMalloc(&d_adj, edge_size));
	HRR(hipMalloc(&d_offset, offset_size));
	// HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	// HRR(hipMalloc(&results, edge_count * sizeof(unsigned long long)));
	HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	HRR(hipMemcpy(d_source, graph_d->source_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj, graph_d->adj_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_offset, graph_d->beg_pos, offset_size, hipMemcpyHostToDevice));

	gpu_run(d_source, d_adj, d_offset, results);

	HRR(hipFree(d_source));
	HRR(hipFree(d_adj));
	HRR(hipFree(d_offset));
	HRR(hipFree(results));
}

int main(int argc, char **argv)
{
	string file = argv[1];
	if (argc >= 4)
	{
		iterator_count = atoi(argv[3]);
	}
	if (argc >= 5)
	{
		iter_batch = atoi(argv[4]);
	}
	// if (argc >= 5)
	// {
	// 	grid_size = atoi(argv[4]);
	// 	block_size = atoi(argv[5]);
	// }

	hipSetDevice(atoi(argv[2]));

	graph *graph_d = readGraph(file);
	edge_count = graph_d->edge_count;
	vertex_count = graph_d->vertex_count;

	grid_size = edge_count / block_bucketnum / 20;

	cout << "dataset\t" << file << endl;
	cout << "Number of nodes: " << vertex_count
		 << ", number of edges: " << edge_count << endl;

	TC_gpu(graph_d);

	return 0;
}
