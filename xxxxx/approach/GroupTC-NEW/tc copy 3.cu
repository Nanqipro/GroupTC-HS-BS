#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "../comm/cuda_comm.h"

#define block_bucketnum 256
#define subwarp_size 32
#define warp_step block_bucketnum / subwarp_size

int iter_batch = 0;
int grid_size = NumberOfMPs() * 8;
int block_size = WarpSize() * 2;

int edge_count;
int vertex_count;
int iterator_count = 100;

__device__ int bin_search1(vertex_t *arr, vertex_t *sh_arr, int offset, int len, int val)
{

	uint32_t Y;
	int32_t bot = 0;
	int32_t top = len - 1;
	int32_t r;
	while (top >= bot)
	{
		r = (top + bot) / 2;
		if (r + offset < block_bucketnum)
		{
			Y = sh_arr[r];
		}
		else
		{
			Y = arr[r];
		}

		if (val == Y)
		{
			return 1;
		}
		if (val < Y)
		{
			top = r - 1;
		}
		else
		{
			bot = r + 1;
		}
	}
	return 0;
}

// __device__ int bin_search(vertex_t *arr, int len, int val)
// {
// 	int ret = 0;
// 	int halfsize;
// 	int candidate;
// 	int temp = len;
// 	while (temp > 1)
// 	{
// 		halfsize = temp / 2;
// 		candidate = arr[ret + halfsize];
// 		ret += (candidate < val) ? halfsize : 0;
// 		temp -= halfsize;
// 	}
// 	ret += (arr[ret] < val);
// 	return ret < len && arr[ret] == val;
// }

__device__ inline int bin_search(vertex_t *arr, vertex_t *sh_arr, int offset, int len, int val)
{
	int ret = 0;
	int halfsize;
	int candidate;
	int temp = len;
	while (temp > 1)
	{
		halfsize = temp / 2;
		// candidate = offset + ret + halfsize < block_bucketnum ? sh_arr[ret + halfsize] : arr[ret + halfsize];
		candidate = arr[ret + halfsize];
		ret += (candidate < val) ? halfsize : 0;
		temp -= halfsize;
	}
	ret += (arr[ret] < val);
	return ret < len && arr[ret] == val;
	// ret += (sh_arr[ret] < val);
	// return ret < len && sh_arr[ret] == val;
}

__global__ void
grouptc(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, unsigned long long *GLOBAL_COUNT, int iter_batch)
{

	__shared__ int sh_tb[block_bucketnum];
	__shared__ int sh_tb_start[block_bucketnum];
	__shared__ int sh_tb_len[block_bucketnum];
	__shared__ int sh_ele_start[block_bucketnum];
	__shared__ int sh_ele_len[block_bucketnum];

	int bid = blockIdx.x;
	int tid = threadIdx.x;

	unsigned long long P_counter = 0;
	for (int i = bid * block_bucketnum; i < edge_count; i += gridDim.x * block_bucketnum)
	{
		if (i + tid < edge_count)
		{
			int src = src_list[i + tid];
			int dst = adj_list[i + tid];

			sh_tb[tid] = dst;
			sh_tb_start[tid] = i + tid + 1;
			sh_tb_len[tid] = beg_pos[src + 1] - (i + tid + 1);
			sh_ele_start[tid] = beg_pos[dst];
			sh_ele_len[tid] = beg_pos[dst + 1] - sh_ele_start[tid];
		}

		__syncthreads();

		int now = tid / subwarp_size;
		int end = min(edge_count - i, block_bucketnum);
		int workid = tid % subwarp_size;

		// 获取二跳邻居节点
		int neighbor_degree = sh_ele_len[now];
		while (now < end)
		{
			// 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
			while (now < end && workid >= neighbor_degree)
			{
				now += warp_step;
				if (now < end)
				{
					workid -= neighbor_degree;
					neighbor_degree = sh_ele_len[now];
				}
			}

			if (now < end)
			{
				int val = adj_list[sh_ele_start[now] + workid];

				int tb_start = sh_tb_start[now];
				int tb_len = sh_tb_len[now];

				P_counter += bin_search(adj_list + tb_start, sh_tb + tb_start - i, tb_start - i, tb_len, val);
			}
			workid += subwarp_size;
		}
		__syncthreads();
	}

	// atomicAdd(GLOBAL_COUNT2, P_counter);
	GLOBAL_COUNT[bid * block_bucketnum + tid] = P_counter;
}

void gpu_run(vertex_t *d_source, vertex_t *d_adj, index_t *d_offset, unsigned long long *results)
{

	double t_start, total_kernel_use = 0;
	uint64_t count;
	hipProfilerStop();
	for (int i = 0; i < iterator_count; i++)
	{
		hipMemset(results, grid_size * block_bucketnum * sizeof(unsigned long long), 0);
		t_start = wtime();
		hipProfilerStart();
		grouptc<<<grid_size, block_bucketnum>>>(d_source, d_adj, d_offset, edge_count, vertex_count, results, iter_batch);
		HRR(hipDeviceSynchronize());
		hipProfilerStop();
		thrust::device_ptr<unsigned long long> ptr(results);
		count = thrust::reduce(ptr, ptr + (grid_size * block_bucketnum));
		double ee = wtime();
		total_kernel_use += ee - t_start;
		if (i == 0)
		{
			if (ee - t_start > 0.1 && iterator_count != 1)
			{
				iterator_count = 10;
			}
		}
	}

	printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
	printf("triangle count %ld \n\n", count);
}

void TC_gpu(graph *graph_d)
{

	long int edge_size = sizeof(vertex_t) * edge_count;
	long int offset_size = sizeof(index_t) * (vertex_count + 1);

	vertex_t *d_source, *d_adj;
	index_t *d_offset;
	unsigned long long *results;
	HRR(hipMalloc(&d_source, edge_size));
	HRR(hipMalloc(&d_adj, edge_size));
	HRR(hipMalloc(&d_offset, offset_size));
	// HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	// HRR(hipMalloc(&results, edge_count * sizeof(unsigned long long)));
	HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	HRR(hipMemcpy(d_source, graph_d->source_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj, graph_d->adj_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_offset, graph_d->beg_pos, offset_size, hipMemcpyHostToDevice));

	gpu_run(d_source, d_adj, d_offset, results);

	HRR(hipFree(d_source));
	HRR(hipFree(d_adj));
	HRR(hipFree(d_offset));
	HRR(hipFree(results));
}

int main(int argc, char **argv)
{
	string file = argv[1];
	if (argc >= 4)
	{
		iterator_count = atoi(argv[3]);
	}
	if (argc >= 5)
	{
		iter_batch = atoi(argv[4]);
	}
	// if (argc >= 5)
	// {
	// 	grid_size = atoi(argv[4]);
	// 	block_size = atoi(argv[5]);
	// }

	hipSetDevice(atoi(argv[2]));

	graph *graph_d = readGraph(file);
	edge_count = graph_d->edge_count;
	vertex_count = graph_d->vertex_count;

	grid_size = edge_count / block_bucketnum / 20;

	cout << "dataset\t" << file << endl;
	cout << "Number of nodes: " << vertex_count
		 << ", number of edges: " << edge_count << endl;

	TC_gpu(graph_d);

	return 0;
}
