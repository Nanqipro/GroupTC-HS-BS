#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include "../comm/cuda_comm.h"

#define block_bucketnum 256
#define subwarp_size 256
#define warp_step block_bucketnum / subwarp_size
#define BATCH_SIZE 4

int grid_size = NumberOfMPs() * 8;
int block_size = WarpSize() * 2;

int edge_count;
int vertex_count;
int iterator_count = 100;

__device__ int bin_search1(vertex_t *arr, int len, int val)
{

	uint32_t Y;
	int32_t bot = 0;
	int32_t top = len - 1;
	int32_t r;
	while (top >= bot)
	{
		r = (top + bot) / 2;
		Y = arr[r];

		if (val == Y)
		{
			return 1;
		}
		if (val < Y)
		{
			top = r - 1;
		}
		else
		{
			bot = r + 1;
		}
	}
	return 0;
}

// __device__ int bin_search(vertex_t *arr, int len, int val)
// {
// 	int ret = 0;
// 	int halfsize;
// 	int candidate;
// 	int temp = len;
// 	while (temp > 1)
// 	{
// 		halfsize = temp / 2;
// 		candidate = arr[ret + halfsize];
// 		ret += (candidate < val) ? halfsize : 0;
// 		temp -= halfsize;
// 	}
// 	ret += (arr[ret] < val);
// 	return ret < len && arr[ret] == val;
// }

__device__ int bin_search(vertex_t *arr, int len, int val, int &offset)
{
	int ret = 0;
	int halfsize;
	int candidate;
	int temp = len;
	while (temp > 1)
	{
		halfsize = temp / 2;
		candidate = arr[ret + halfsize];
		ret += (candidate < val) ? halfsize : 0;
		temp -= halfsize;
	}
	ret += (arr[ret] < val);
	offset = ret;
	return ret < len && arr[ret] == val;
}

// template <const int EPB, const int NPT, const int SUBWARP_SIZE>
__global__ void grouptc(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, unsigned long long *GLOBAL_COUNT)
{

	// 共享内存中的 hashTable
	__shared__ int sh_tb_start[block_bucketnum];
	__shared__ int sh_tb_len[block_bucketnum];
	__shared__ int sh_ele_start[block_bucketnum];
	__shared__ int sh_ele_len[block_bucketnum];

	__shared__ int sh_cache_parent[block_bucketnum * BATCH_SIZE];
	__shared__ int sh_cache_value[block_bucketnum * BATCH_SIZE];

	unsigned long long P_counter = 0;

	int bid = blockIdx.x;
	int tid = threadIdx.x;

	for (int blockEdgeIdx = bid * block_bucketnum; blockEdgeIdx < edge_count; blockEdgeIdx += gridDim.x * block_bucketnum)
	{
		if (blockEdgeIdx + tid < edge_count)
		{
			int src = src_list[blockEdgeIdx + tid];
			int dst = adj_list[blockEdgeIdx + tid];
			int temp;

			int tb_start, tb_len, ele_start, ele_len;
			tb_start = blockEdgeIdx + tid + 1;
			tb_len = beg_pos[src + 1] - tb_start;
			ele_start = beg_pos[dst];
			ele_len = beg_pos[dst + 1] - ele_start;

			if (tb_len * 2 < ele_len)
			{
				temp = tb_start;
				tb_start = ele_start;
				ele_start = temp;

				temp = tb_len;
				tb_len = ele_len;
				ele_len = temp;
			}

			sh_tb_start[tid] = tb_start;
			sh_tb_len[tid] = tb_len;
			sh_ele_start[tid] = ele_start;
			sh_ele_len[tid] = ele_len;
		}

		__syncthreads();

		int warpID = tid / subwarp_size;
		int now = warpID * subwarp_size;
		int end = min(edge_count - blockEdgeIdx, (warpID + 1) * subwarp_size);
		int workid = tid % subwarp_size;
		int offset = 0;
		int last_now = -1;

		// 获取二跳邻居节点
		int neighbor_degree = sh_ele_len[now];
		while (now < end)
		{
			for (int iter = 0; iter < BATCH_SIZE; iter++)
			{
				// 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
				while (now < end && workid >= neighbor_degree)
				{
					now++;
					if (now < end)
					{
						workid -= neighbor_degree;
						neighbor_degree = sh_ele_len[now];
					}
				}

				if (now < end)
				{
					offset = last_now == now ? offset : 0;
					int val = adj_list[sh_ele_start[now] + workid];

					sh_cache_parent[iter * block_bucketnum + tid] = now;
					sh_cache_value[iter * block_bucketnum + tid] = val;

					// // load to register
					// // tmp[iter] = val;
					// P_counter += bin_search(adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
					// last_now = now;
				}
			}
			// __syncthreads();

			// compute
			int parent, val;
			for (int iter = 0; iter < BATCH_SIZE; iter++)
			{
				parent = sh_cache_parent[iter + tid * BATCH_SIZE];
				val = sh_cache_value[iter + tid * BATCH_SIZE];
				P_counter += bin_search(adj_list + (sh_tb_start[parent] + offset), sh_tb_len[parent] - offset, val , offset);
				last_now = parent;
			}

			workid += subwarp_size;
		}
		__syncthreads();
	}

	// atomicAdd(GLOBAL_COUNT2, P_counter);
	GLOBAL_COUNT[bid * block_bucketnum + tid] = P_counter;
}

void gpu_run(vertex_t *d_source, vertex_t *d_adj, index_t *d_offset, unsigned long long *results)
{

	double t_start, total_kernel_use = 0;
	uint64_t count;
	hipProfilerStop();
	for (int i = 0; i < iterator_count; i++)
	{
		hipMemset(results, grid_size * block_bucketnum * sizeof(unsigned long long), 0);
		t_start = wtime();
		hipProfilerStart();
		grouptc<<<grid_size, block_bucketnum>>>(d_source, d_adj, d_offset, edge_count, vertex_count, results);
		HRR(hipDeviceSynchronize());
		hipProfilerStop();
		thrust::device_ptr<unsigned long long> ptr(results);
		count = thrust::reduce(ptr, ptr + (grid_size * block_bucketnum));
		double ee = wtime();
		total_kernel_use += ee - t_start;
		if (i == 0)
		{
			if (ee - t_start > 0.1 && iterator_count != 1)
			{
				iterator_count = 10;
			}
		}
	}

	printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
	printf("triangle count %ld \n\n", count);
}

void TC_gpu(graph *graph_d)
{

	long int edge_size = sizeof(vertex_t) * edge_count;
	long int offset_size = sizeof(index_t) * (vertex_count + 1);

	vertex_t *d_source, *d_adj;
	index_t *d_offset;
	unsigned long long *results;
	HRR(hipMalloc(&d_source, edge_size));
	HRR(hipMalloc(&d_adj, edge_size));
	HRR(hipMalloc(&d_offset, offset_size));
	// HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	// HRR(hipMalloc(&results, edge_count * sizeof(unsigned long long)));
	HRR(hipMalloc(&results, grid_size * block_bucketnum * sizeof(unsigned long long)));

	HRR(hipMemcpy(d_source, graph_d->source_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj, graph_d->adj_list, edge_size, hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_offset, graph_d->beg_pos, offset_size, hipMemcpyHostToDevice));

	gpu_run(d_source, d_adj, d_offset, results);

	HRR(hipFree(d_source));
	HRR(hipFree(d_adj));
	HRR(hipFree(d_offset));
	HRR(hipFree(results));
}

int main(int argc, char **argv)
{
	string file = argv[1];
	if (argc >= 4)
	{
		iterator_count = atoi(argv[3]);
	}
	if (argc >= 5)
	{
		grid_size = atoi(argv[4]);
		block_size = atoi(argv[5]);
	}

	hipSetDevice(atoi(argv[2]));

	graph *graph_d = readGraph(file);
	edge_count = graph_d->edge_count;
	vertex_count = graph_d->vertex_count;

	cout << "dataset\t" << file << endl;
	cout << "Number of nodes: " << vertex_count
		 << ", number of edges: " << edge_count << endl;

	TC_gpu(graph_d);

	return 0;
}
