#include "hip/hip_runtime.h"
#include <arpa/inet.h>
#include <assert.h>
#include <errno.h>
#include <math.h>
#include <netdb.h>
#include <netinet/in.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <unistd.h>

#include <iostream>
#include <iterator>
#include <queue>
#include <set>

#include "../comm/cuda_comm.h"
using namespace std;

struct arguments {
    int edge_count;
    long long count;
    double time;
    int degree;
    int vertices;
};

struct arguments Triangle_count(char input[100], struct arguments args, int threads, int blocks, int chunk_size);

int iterator_count = 100;

int main(int argc, char *argv[]) {
    char *name = argv[1];
    int device = atoi(argv[2]);
    iterator_count = atoi(argv[3]);
    // int N_THREADS = atoi(argv[2]);
    // int N_BLOCKS = atoi(argv[3]);
    // int chunk_size = atoi(argv[4]);
    int N_THREADS = 1024;
    int N_BLOCKS = 1024;
    int chunk_size = 1;
    struct arguments args = {};
    //  call the function
    // long long sum = 0;
    // double time = 0;
    hipSetDevice(device);
    args = Triangle_count(name, args, N_THREADS, N_BLOCKS, chunk_size);
    // time = args.time;
    // sum = args.count;
    // printf("%s,%d,%d,%lld,%f,%f \n", argv[1], args.vertices, args.edge_count, sum, time, (args.edge_count / time / 1000000000));
    return 0;
}

// #define dynamic
#define static
#define without_combination 0
#define use_static 0

#define BLOCK_SIZE 1024
#define GROUP_SIZE 1024
#define WARP_SIZE 64

#define shared_BLOCK_BUCKET_SIZE 6
#define shared_GROUP_BUCKET_SIZE 4
#define SUM_SIZE 1
#define USE_CTA 100
#define USE_WARP 1

#define block_bucketnum 1024
#define group_bucketnum 1024
#define BLOCK_MODULO 1023
#define GROUP_MODULO 1023

#define EDGE_CHUNK 512
#define shared_CHUNK_CACHE_SIZE 640
#define BLOCK_BUCKET_SIZE 100
#define GROUP_BUCKET_SIZE 100

using namespace std;

__device__ int linear_search_block(int neighbor, int *shared_partition, int *partition, int len, int bin, int BIN_START) {
    for (;;) {
        // int i = bin;
        // int len = bin_count[i];
        // int step = 0;
        // int nowlen;
        // if (len < shared_BLOCK_BUCKET_SIZE)
        //    nowlen = len;
        // else
        //    nowlen = shared_BLOCK_BUCKET_SIZE;
        // while (step < nowlen)
        // {
        //    if (shared_partition[i] == neighbor)
        //    {
        //       return 1;
        //    }
        //    i += block_bucketnum;
        //    step += 1;
        // }

        len -= shared_BLOCK_BUCKET_SIZE;
        int i = bin + BIN_START;
        int step = 0;
        while (step < len) {
            if (partition[i] == neighbor) {
                return 1;
            }
            i += block_bucketnum;
            step += 1;
        }
        if (len + shared_BLOCK_BUCKET_SIZE < 99) break;
        bin++;
    }
    return 0;
}

__device__ int linear_search_group(int neighbor, int *shared_partition, int *partition, int len, int bin, int BIN_START) {
    // int i = bin;
    // int len = bin_count[i];

    // int step = 0;
    // int nowlen = len < shared_GROUP_BUCKET_SIZE ? len : shared_GROUP_BUCKET_SIZE;
    // while (step < nowlen)
    // {
    //    if (shared_partition[i] == neighbor)
    //    {
    //       return 1;
    //    }
    //    i += group_bucketnum;
    //    step += 1;
    // }

    len -= shared_GROUP_BUCKET_SIZE;
    int i = bin + BIN_START;
    int step = 0;
    while (step < len) {
        if (partition[i] == neighbor) {
            return 1;
        }
        i += group_bucketnum;
        step += 1;
    }

    return 0;
}

int my_binary_search(int len, int val, index_t *beg) {
    int l = 0, r = len;
    while (l < r - 1) {
        int mid = (l + r) / 2;
        if (beg[mid + 1] - beg[mid] > val)
            l = mid;
        else
            r = mid;
    }
    if (beg[l + 1] - beg[l] <= val) return -1;
    return l;
}

__global__ void trust_block(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, int *partition,
                            unsigned long long *GLOBAL_COUNT, int T_Group, int *G_INDEX, int CHUNK_SIZE, int warpfirstvertex, int warpfirstedge,
                            int nocomputefirstvertex, int nocomputefirstedge) {
    // hashTable bucket 计数器
    __shared__ int bin_count[block_bucketnum];
    // 共享内存中的 hashTable
    __shared__ int shared_partition[block_bucketnum * shared_BLOCK_BUCKET_SIZE];
    unsigned long long __shared__ G_counter;

    if (threadIdx.x == 0) {
        G_counter = 0;
    }

    int BIN_START = blockIdx.x * block_bucketnum * BLOCK_BUCKET_SIZE;
    unsigned long long P_counter = 0;

    // CTA for large degree vertex
    int vertex = blockIdx.x * CHUNK_SIZE;
    int vertex_end = vertex + CHUNK_SIZE;
    __shared__ int ver;

    while (vertex < warpfirstvertex)
    // while (0)
    {
        // if (degree<=USE_CTA) break;
        int group_start = beg_pos[vertex];
        int end = beg_pos[vertex + 1];
        int now = threadIdx.x + group_start;
        // int MODULO = block_bucketnum - 1;
        // clean bin_count
        // 初始化 hashTable bucket 计数器
        for (int i = threadIdx.x; i < block_bucketnum; i += BLOCK_SIZE) bin_count[i] = 0;
        __syncthreads();

        // count hash bin
        // 生成 hashTable
        while (now < end) {
            int temp = adj_list[now];
            int bin = temp & BLOCK_MODULO;
            int index;
            index = atomicAdd(&bin_count[bin], 1);
            if (index < shared_BLOCK_BUCKET_SIZE) {
                shared_partition[index * block_bucketnum + bin] = temp;
            } else if (index < BLOCK_BUCKET_SIZE) {
                index = index - shared_BLOCK_BUCKET_SIZE;
                partition[index * block_bucketnum + bin + BIN_START] = temp;
            }
            now += blockDim.x;
        }
        __syncthreads();

        // list intersection
        now = beg_pos[vertex];
        end = beg_pos[vertex + 1];
        int superwarp_ID = threadIdx.x / 64;
        int superwarp_TID = threadIdx.x % 64;
        int workid = superwarp_TID;
        now = now + superwarp_ID;
        // 获取二跳邻居节点
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += 16;
                workid -= neighbor_degree;
                neighbor = adj_list[now];
                neighbor_start = beg_pos[neighbor];
                neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
            }
            if (now < end) {
                int temp_adj = adj_list[neighbor_start + workid];
                int bin = temp_adj & BLOCK_MODULO;

                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + block_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + block_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + block_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + block_bucketnum * 3] == temp_adj : 0;
                P_counter += len > 4 ? shared_partition[bin + block_bucketnum * 4] == temp_adj : 0;
                P_counter += len > 5 ? shared_partition[bin + block_bucketnum * 5] == temp_adj : 0;

                if (len > shared_BLOCK_BUCKET_SIZE) {
                    P_counter += linear_search_block(temp_adj, shared_partition, partition, len, bin, BIN_START);
                }
            }
            // __syncthreads();
            workid += 64;
        }

        __syncthreads();
        // if (vertex>1) break;
        vertex++;
        if (vertex == vertex_end) {
            if (threadIdx.x == 0) {
                ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
            }
            __syncthreads();
            vertex = ver;
            vertex_end = vertex + CHUNK_SIZE;
        }
    }

    // EDGE CHUNK for small degree vertex
    __shared__ int group_start;
    __shared__ int group_size;

    int *shared_src = shared_partition + group_bucketnum * shared_GROUP_BUCKET_SIZE;
    int *shared_adj_start = shared_src + shared_CHUNK_CACHE_SIZE;
    int *shared_adj_degree = shared_adj_start + shared_CHUNK_CACHE_SIZE;

    for (int group_offset = warpfirstedge + blockIdx.x * EDGE_CHUNK; group_offset < nocomputefirstedge; group_offset += gridDim.x * EDGE_CHUNK) {
        // compute group start and end
        if (threadIdx.x == 0) {
            int src = src_list[group_offset];
            int src_start = beg_pos[src];
            int src_end = beg_pos[src + 1];
            group_start = ((src_start == group_offset) ? src_start : src_end);

            src = src_list[min(group_offset + EDGE_CHUNK, nocomputefirstedge) - 1];
            group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
        }

        // cache start
        for (int i = threadIdx.x; i < group_bucketnum; i += blockDim.x) bin_count[i] = 0;

        __syncthreads();

        for (int i = threadIdx.x; i < group_size; i += BLOCK_SIZE) {
            int temp_src = src_list[i + group_start];
            int temp_adj = adj_list[i + group_start];

            shared_src[i] = temp_src;
            shared_adj_start[i] = beg_pos[temp_adj];
            shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

            int bin = (temp_src + temp_adj) & GROUP_MODULO;
            int index = atomicAdd(&bin_count[bin], 1);

            if (index < shared_GROUP_BUCKET_SIZE) {
                shared_partition[index * group_bucketnum + bin] = temp_adj;
            } else if (index < GROUP_BUCKET_SIZE) {
                index = index - shared_GROUP_BUCKET_SIZE;
                partition[index * group_bucketnum + bin + BIN_START] = temp_adj;
            }
        }
        __syncthreads();

        // compute 2 hop neighbors
        int now = threadIdx.x / WARP_SIZE;
        int workid = threadIdx.x % WARP_SIZE;

        while (now < group_size) {
            int neighbor_degree = shared_adj_degree[now];
            while (now < group_size && workid >= neighbor_degree) {
                now += BLOCK_SIZE / WARP_SIZE;
                workid -= neighbor_degree;
                neighbor_degree = shared_adj_degree[now];
            }

            if (now < group_size) {
                int temp_src = shared_src[now];
                int temp_adj = adj_list[shared_adj_start[now] + workid];
                int bin = (temp_src + temp_adj) & GROUP_MODULO;
                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + group_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + group_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + group_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + group_bucketnum * 3] == temp_adj : 0;

                if (len > shared_GROUP_BUCKET_SIZE) {
                    P_counter += linear_search_group(temp_adj, shared_partition, partition, len, bin, BIN_START);
                }
            }
            workid += WARP_SIZE;
        }
        __syncthreads();
    }

    atomicAdd(&G_counter, P_counter);

    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
    }
}

__global__ void trust_group(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, int edge_count, int vertex_count, int *partition,
                            unsigned long long *GLOBAL_COUNT, int T_Group, int *G_INDEX, int CHUNK_SIZE, int warpfirstvertex, int warpfirstedge,
                            int nocomputefirstvertex, int nocomputefirstedge) {
    // hashTable bucket 计数器
    __shared__ int bin_count[group_bucketnum];
    // 共享内存中的 hashTable
    __shared__ int shared_partition[group_bucketnum * shared_BLOCK_BUCKET_SIZE];
    unsigned long long __shared__ G_counter;

    if (threadIdx.x == 0) {
        G_counter = 0;
    }

    int BIN_START = blockIdx.x * group_bucketnum * BLOCK_BUCKET_SIZE;
    unsigned long long P_counter = 0;

    __shared__ int group_start;
    __shared__ int group_size;

    int *shared_src = shared_partition + group_bucketnum * shared_GROUP_BUCKET_SIZE;
    int *shared_adj_start = shared_src + shared_CHUNK_CACHE_SIZE;
    int *shared_adj_degree = shared_adj_start + shared_CHUNK_CACHE_SIZE;

    for (int group_offset = warpfirstedge + blockIdx.x * EDGE_CHUNK; group_offset < nocomputefirstedge; group_offset += gridDim.x * EDGE_CHUNK) {
        // compute group start and end
        if (threadIdx.x == 0) {
            int src = src_list[group_offset];
            int src_start = beg_pos[src];
            int src_end = beg_pos[src + 1];
            group_start = ((src_start == group_offset) ? src_start : src_end);

            src = src_list[min(group_offset + EDGE_CHUNK, nocomputefirstedge) - 1];
            group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
        }

        // cache start
        for (int i = threadIdx.x; i < group_bucketnum; i += blockDim.x) bin_count[i] = 0;

        __syncthreads();

        for (int i = threadIdx.x; i < group_size; i += BLOCK_SIZE) {
            int temp_src = src_list[i + group_start];
            int temp_adj = adj_list[i + group_start];

            shared_src[i] = temp_src;
            shared_adj_start[i] = beg_pos[temp_adj];
            shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

            int bin = (temp_src + temp_adj) & GROUP_MODULO;
            int index = atomicAdd(&bin_count[bin], 1);

            if (index < shared_GROUP_BUCKET_SIZE) {
                shared_partition[index * group_bucketnum + bin] = temp_adj;
            } else if (index < GROUP_BUCKET_SIZE) {
                index = index - shared_GROUP_BUCKET_SIZE;
                partition[index * group_bucketnum + bin + BIN_START] = temp_adj;
            }
        }
        __syncthreads();

        // for (int i = threadIdx.x; i < group_bucketnum; i += BLOCK_SIZE)
        // {
        //    if (bin_count[i] == 2)
        //    {
        //       P_counter++;
        //    }
        // }

        // __syncthreads();

        // 获取二跳邻居节点
        int now = threadIdx.x / WARP_SIZE;
        int workid = threadIdx.x % WARP_SIZE;

        while (now < group_size) {
            int neighbor_degree = shared_adj_degree[now];
            while (now < group_size && workid >= neighbor_degree) {
                now += GROUP_SIZE / WARP_SIZE;
                workid -= neighbor_degree;
                neighbor_degree = shared_adj_degree[now];
            }

            if (now < group_size) {
                int temp_src = shared_src[now];
                int temp_adj = adj_list[shared_adj_start[now] + workid];
                int bin = (temp_src + temp_adj) & GROUP_MODULO;
                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + group_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + group_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + group_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + group_bucketnum * 3] == temp_adj : 0;

                if (len > shared_GROUP_BUCKET_SIZE) {
                    P_counter += linear_search_group(temp_adj, shared_partition, partition, len, bin, BIN_START);
                }
            }
            workid += WARP_SIZE;
        }
        __syncthreads();
    }

    atomicAdd(&G_counter, P_counter);

    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
    }
}

struct arguments Triangle_count(char name[100], struct arguments args, int n_threads, int n_blocks, int chunk_size) {
    int T_Group = 32;
    int BUCKET_SIZE = 100;
    int total = n_blocks * block_bucketnum * BUCKET_SIZE;
    unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);
    string json_file = name;
    graph *graph_d = new graph(json_file);
    index_t vertex_count = graph_d->vertex_count;
    index_t edge_count = graph_d->edge_count;
    index_t edges = graph_d->edge_count;
    int maxDegree = 0;
    for (int i = 1; i <= graph_d->vertex_count; i++) {
        int degree = graph_d->beg_pos[i] - graph_d->beg_pos[i - 1];
        if (degree > maxDegree) {
            maxDegree = degree;
        }
    }

    cout << "dataset\t" << json_file << endl;
    cout << "Number of nodes: " << vertex_count << ", number of edges: " << edge_count << endl;
    // cout << "load graph file:" << name << "  vCount:" << graph_d->vertex_count << "  eCount:" << graph_d->edge_count << "  maxDegree:" << maxDegree
    // << endl;

    // ofstream outFile("/home/LiJB/cuda_project/TRUST/output/adj_list.txt", ios::out);
    // for (int i = 0; i < vertex_count; i++)
    // {
    //    int start = graph_d->beg_pos[i];
    //    int end = graph_d->beg_pos[i + 1];
    //    for (int j = start; j < end; j++)
    //    {
    //       outFile << i << "  " << graph_d->adj_list[j] << endl;
    //    }
    // }

    /* Preprocessing Step to calculate the ratio */
    int *prefix = (int *)malloc(sizeof(int) * vertex_count);

    int warpfirstvertex = my_binary_search(vertex_count, USE_CTA, graph_d->beg_pos) + 1;
    int warpfirstedge = graph_d->beg_pos[warpfirstvertex];
    int nocomputefirstvertex = my_binary_search(vertex_count, USE_WARP, graph_d->beg_pos) + 1;
    int nocomputefirstedge = graph_d->beg_pos[nocomputefirstvertex];

    printf("warpfirstvertex %d  warpfirstedge %d\n", warpfirstvertex, warpfirstedge);
    printf("nocomputefirstvertex %d  nocomputefirstedge %d\n", nocomputefirstvertex, nocomputefirstedge);

    int *BIN_MEM;
    unsigned long long *GLOBAL_COUNT;
    int *G_INDEX;
    index_t *d_beg_pos;
    vertex_t *d_src_list;
    vertex_t *d_adj_list;
    HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
    HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));
    HRR(hipMalloc((void **)&d_beg_pos, sizeof(index_t) * (vertex_count + 1)));
    HRR(hipMalloc((void **)&d_src_list, sizeof(vertex_t) * (edge_count)));
    HRR(hipMalloc((void **)&d_adj_list, sizeof(vertex_t) * (edge_count)));
    // Swap edge list count with Eend - estart; --> gives error; may add some more

    int nowindex[3];
    nowindex[0] = chunk_size * n_blocks * n_threads / T_Group;
    nowindex[1] = chunk_size * n_blocks;
    nowindex[2] = warpfirstvertex + chunk_size * (n_blocks * n_threads / T_Group);
    // unsigned long long cou=0;
    // int nowindex=0;

    HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_beg_pos, graph_d->beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_src_list, graph_d->source_list, sizeof(vertex_t) * edge_count, hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_adj_list, graph_d->adj_list, sizeof(vertex_t) * edge_count, hipMemcpyHostToDevice));
    double t1 = wtime();
    double cmp_time;
    HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * total));

    double total_kernel_use = 0;
    double startKernel, ee = 0;
    int block_kernel_grid_size = min(max(warpfirstvertex, 1), n_blocks);
    int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (EDGE_CHUNK * 10), n_blocks);
    // int block_kernel_grid_size =  n_blocks;
    // int group_kernel_grid_size = n_blocks;

    for (int i = 0; i < iterator_count; i++) {
        HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
        double time_start = clock();
        startKernel = wtime();
        hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
        // trust<<<n_blocks, n_threads>>>(d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, BUCKET_SIZE, T_Group,
        // G_INDEX, chunk_size, warpfirstvertex, warpfirstedge, nocomputefirstvertex, nocomputefirstedge);
        trust_block<<<group_kernel_grid_size, BLOCK_SIZE>>>(d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT,
                                                            T_Group, G_INDEX, chunk_size, warpfirstvertex, warpfirstedge, nocomputefirstvertex,
                                                            nocomputefirstedge);
        // HRR(hipDeviceSynchronize());
        // trust_group<<<group_kernel_grid_size, GROUP_SIZE>>>(d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT,
        // T_Group, G_INDEX, chunk_size, warpfirstvertex, warpfirstedge, nocomputefirstvertex, nocomputefirstedge);
        HRR(hipDeviceSynchronize());
        HRR(hipGetLastError());
        ee = wtime();
        total_kernel_use += ee - startKernel;
        if (i == 0) {
            if (ee - startKernel > 0.1 && iterator_count != 1) {
                iterator_count = 10;
            }
        }
        // cout << "kernel use " << ee - startKernel << endl;
        cmp_time = clock() - time_start;
    }

    // HRR(hipFree(BIN_MEM));
    cmp_time = cmp_time / CLOCKS_PER_SEC;
    HRR(hipFree(BIN_MEM));

    HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));
    printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
    printf("triangle count %lld \n\n", counter[0]);
    // printf("xxx %lld \n\n", counter[1]);
    // cout << "total triangle count: " << counter[0] << endl
    //      << endl;
    // printf("avg kernel use %lf s\n\n", total_kernel_use / iterator_count);
    HRR(hipFree(GLOBAL_COUNT));
    HRR(hipFree(G_INDEX));
    HRR(hipFree(d_beg_pos));
    HRR(hipFree(d_src_list));
    HRR(hipFree(d_adj_list));
    free(prefix);
    delete graph_d;
    args.time = cmp_time;
    args.count = counter[0];

    args.edge_count = edges;
    args.degree = edges / vertex_count;
    args.vertices = vertex_count;
    return args;
}
