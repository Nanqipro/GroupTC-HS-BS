#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <errno.h>
#include <netdb.h>
#include <queue>
#include <set>
#include <iterator>
#include "../comm/cuda_comm.h"
#include <math.h>
using namespace std;

struct arguments
{
   int edge_count;
   long long count;
   double time;
   int degree;
   int vertices;
};

struct arguments Triangle_count(char input[100], struct arguments args, int threads, int blocks, int chunk_size);

int iterator_count = 100;

int main(int argc, char* argv[])
{
   char* name = argv[1];
   int device = atoi(argv[2]);
   iterator_count = atoi(argv[3]);
   // int N_THREADS = atoi(argv[2]);
   // int N_BLOCKS = atoi(argv[3]);
   // int chunk_size = atoi(argv[4]);
   int N_THREADS = 1024;
   int N_BLOCKS = 1024;
   int chunk_size = 1;
   struct arguments args = {};
   //  call the function
   // long long sum = 0;
   // double time = 0;
   hipSetDevice(device);
   args = Triangle_count(name, args, N_THREADS, N_BLOCKS, chunk_size);
   // time = args.time;
   // sum = args.count;
   // printf("%s,%d,%d,%lld,%f,%f \n", argv[1], args.vertices, args.edge_count, sum, time, (args.edge_count / time / 1000000000));
   return 0;
}

// #define dynamic
#define static
#define shared_BUCKET_SIZE 6
#define SUM_SIZE 1
#define USE_CTA 100
#define USE_WARP 2
#define without_combination 0
#define use_static 0

#define block_bucketnum 1024
#define warp_bucketnum 32

using namespace std;

__device__ int linear_search(int neighbor, int* shared_partition, int* partition, int* bin_count, int bin, int BIN_START)
{

   for (;;)
   {
      int i = bin;
      int len = bin_count[i];
      int step = 0;
      int nowlen;
      if (len < shared_BUCKET_SIZE)
         nowlen = len;
      else
         nowlen = shared_BUCKET_SIZE;
      while (step < nowlen)
      {
         if (shared_partition[i] == neighbor)
         {
            return 1;
         }
         i += block_bucketnum;
         step += 1;
      }

      len -= shared_BUCKET_SIZE;
      i = bin + BIN_START;
      step = 0;
      while (step < len)
      {
         if (partition[i] == neighbor)
         {
            return 1;
         }
         i += block_bucketnum;
         step += 1;
      }
      if (len + shared_BUCKET_SIZE < 99)
         break;
      bin++;
   }
   return 0;
}

int my_binary_search(int len, int val, index_t* beg)
{
   int l = 0, r = len;
   while (l < r - 1)
   {
      int mid = (l + r) / 2;
      if (beg[mid + 1] - beg[mid] > val)
         l = mid;
      else
         r = mid;
   }
   if (beg[l + 1] - beg[l] <= val)
      return -1;
   return l;
}

__global__ void
trust(vertex_t* adj_list, index_t* beg_pos, int edge_count, int vertex_count, int* partition, unsigned long long* GLOBAL_COUNT, int BUCKET_SIZE, int T_Group, int* G_INDEX, int CHUNK_SIZE, int warpfirstvertex)
{

   // int tid=threadIdx.x+blockIdx.x*blockDim.x;
   // hashTable bucket 计数器
   __shared__ int bin_count[block_bucketnum];
   // 共享内存中的 hashTable
   __shared__ int shared_partition[block_bucketnum * shared_BUCKET_SIZE + 1];
   // useless[threadIdx.x]=1;
   unsigned long long __shared__ G_counter;
   int WARPSIZE = 32;
   if (threadIdx.x == 0)
   {
      G_counter = 0;
   }

   int BIN_START = blockIdx.x * block_bucketnum * BUCKET_SIZE;
   // __syncthreads();
   unsigned long long P_counter = 0;

   // start_time = clock64();
   // CTA for large degree vertex
   int vertex = blockIdx.x * CHUNK_SIZE;
   int vertex_end = vertex + CHUNK_SIZE;
   __shared__ int ver;
   while (vertex < warpfirstvertex)
      // while (0)
   {
      // if (degree<=USE_CTA) break;
      int start = beg_pos[vertex];
      int end = beg_pos[vertex + 1];
      int now = threadIdx.x + start;
      int MODULO = block_bucketnum - 1;
      // int divide=(vert_count/blockDim.x);
      int BIN_OFFSET = 0;
      // clean bin_count
      // 初始化 hashTable bucket 计数器
      for (int i = threadIdx.x; i < block_bucketnum; i += blockDim.x)
         bin_count[i] = 0;
      __syncthreads();

      // start_time = clock64();
      // count hash bin
      // 生成 hashTable
      while (now < end)
      {
         int temp = adj_list[now];
         int bin = temp & MODULO;
         int index;
         index = atomicAdd(&bin_count[bin], 1);
         if (index < shared_BUCKET_SIZE)
         {
            shared_partition[index * block_bucketnum + bin] = temp;
         }
         else if (index < BUCKET_SIZE)
         {
            index = index - shared_BUCKET_SIZE;
            partition[index * block_bucketnum + bin + BIN_START] = temp;
         }
         now += blockDim.x;
      }
      __syncthreads();

      // unsigned long long hash_time=clock64()-start_time;
      // start_time = clock64();
      // list intersection
      now = beg_pos[vertex];
      end = beg_pos[vertex + 1];
      int superwarp_ID = threadIdx.x / 64;
      int superwarp_TID = threadIdx.x % 64;
      int workid = superwarp_TID;
      now = now + superwarp_ID;
      // 获取二跳邻居节点
      int neighbor = adj_list[now];
      int neighbor_start = beg_pos[neighbor];
      int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
      while (now < end)
      {
         // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
         while (now < end && workid >= neighbor_degree)
         {
            now += 16;
            workid -= neighbor_degree;
            neighbor = adj_list[now];
            neighbor_start = beg_pos[neighbor];
            neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
         }
         if (now < end)
         {
            int temp = adj_list[neighbor_start + workid];
            int bin = temp & MODULO;
            P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
         }
         // __syncthreads();
         workid += 64;
      }

      __syncthreads();
      // if (vertex>1) break;
      vertex++;
      if (vertex == vertex_end)
      {
         if (threadIdx.x == 0)
         {
            ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
         }
         __syncthreads();
         vertex = ver;
         vertex_end = vertex + CHUNK_SIZE;
      }
      // __syncthreads();
   }

   // warp method
   int WARPID = threadIdx.x / WARPSIZE;
   int WARP_TID = threadIdx.x % WARPSIZE;
   vertex = warpfirstvertex + ((WARPID + blockIdx.x * blockDim.x / WARPSIZE)) * CHUNK_SIZE;
   vertex_end = vertex + CHUNK_SIZE;
   while (vertex < vertex_count)
   {
      int degree = beg_pos[vertex + 1] - beg_pos[vertex];
      if (degree < USE_WARP)
         break;
      int start = beg_pos[vertex];
      int end = beg_pos[vertex + 1];
      int now = WARP_TID + start;
      int MODULO = warp_bucketnum - 1;
      int BIN_OFFSET = WARPID * warp_bucketnum;
      // clean bin_count

      for (int i = BIN_OFFSET + WARP_TID; i < BIN_OFFSET + warp_bucketnum; i += WARPSIZE)
         bin_count[i] = 0;
      // bin_count[threadIdx.x]=0;
      //__syncwarp();

      // count hash bin
      while (now < end)
      {
         int temp = adj_list[now];
         int bin = temp & MODULO;
         bin += BIN_OFFSET;
         int index;
         index = atomicAdd(&bin_count[bin], 1);
         if (index < shared_BUCKET_SIZE)
         {
            shared_partition[index * block_bucketnum + bin] = temp;
         }
         else if (index < BUCKET_SIZE)
         {
            index = index - shared_BUCKET_SIZE;
            partition[index * block_bucketnum + bin + BIN_START] = temp;
         }
         now += WARPSIZE;
      }
      //__syncwarp();

      now = beg_pos[vertex];
      end = beg_pos[vertex + 1];

      int workid = WARP_TID;
      while (now < end)
      {
         int neighbor = adj_list[now];
         int neighbor_start = beg_pos[neighbor];
         int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;

         while (now < end && workid >= neighbor_degree)
         {
            now++;
            workid -= neighbor_degree;
            neighbor = adj_list[now];
            neighbor_start = beg_pos[neighbor];
            neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
         }
         if (now < end)
         {
            int temp = adj_list[neighbor_start + workid];
            int bin = temp & MODULO;
            P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
         }
         //__syncwarp();
         now = __shfl_sync(0xffffffff, now, 31);
         workid = __shfl_sync(0xffffffff, workid, 31);
         workid += WARP_TID + 1;

         // workid+=WARPSIZE;
      }
      //__syncwarp();
      vertex++;
      if (vertex == vertex_end)
      {
         if (WARP_TID == 0)
         {
            vertex = atomicAdd(&G_INDEX[2], CHUNK_SIZE);
         }
         //__syncwarp();
         vertex = __shfl_sync(0xffffffff, vertex, 0);
         vertex_end = vertex + CHUNK_SIZE;
      }
   }

   atomicAdd(&G_counter, P_counter);

   __syncthreads();
   if (threadIdx.x == 0)
   {
      atomicAdd(&GLOBAL_COUNT[0], G_counter);
   }
}

struct arguments Triangle_count(char name[100], struct arguments args, int n_threads, int n_blocks, int chunk_size)
{

   int T_Group = 32;
   int BUCKET_SIZE = 100;
   int total = n_blocks * block_bucketnum * BUCKET_SIZE;
   unsigned long long* counter = (unsigned long long*)malloc(sizeof(unsigned long long) * 10);
   string json_file = name;
   graph* graph_d = new graph(json_file);
   index_t vertex_count = graph_d->vertex_count;
   index_t edge_count = graph_d->edge_count;
   index_t edges = graph_d->edge_count;
   int maxDegree = 0;
   for (int i = 1; i <= graph_d->vertex_count; i++)
   {
      int degree = graph_d->beg_pos[i] - graph_d->beg_pos[i - 1];
      if (degree > maxDegree)
      {
         maxDegree = degree;
      }
   }

   cout << "dataset\t" << json_file << endl;
   cout << "Number of nodes: " << vertex_count
      << ", number of edges: " << edge_count << endl;
   // cout << "load graph file:" << name << "  vCount:" << graph_d->vertex_count << "  eCount:" << graph_d->edge_count << "  maxDegree:" << maxDegree << endl;

   // ofstream outFile("/home/LiJB/cuda_project/TRUST/output/adj_list.txt", ios::out);
   // for (int i = 0; i < vertex_count; i++)
   // {
   //    int start = graph_d->beg_pos[i];
   //    int end = graph_d->beg_pos[i + 1];
   //    for (int j = start; j < end; j++)
   //    {
   //       outFile << i << "  " << graph_d->adj_list[j] << endl;
   //    }
   // }

   /* Preprocessing Step to calculate the ratio */
   int* prefix = (int*)malloc(sizeof(int) * vertex_count);

   int warpfirstvertex = my_binary_search(vertex_count, USE_CTA, graph_d->beg_pos) + 1;

   int* BIN_MEM;
   unsigned long long* GLOBAL_COUNT;
   int* G_INDEX;
   index_t* d_beg_pos;
   vertex_t* d_adj_list;
   HRR(hipMalloc((void**)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
   HRR(hipMalloc((void**)&G_INDEX, sizeof(int) * 3));
   HRR(hipMalloc((void**)&d_beg_pos, sizeof(index_t) * (vertex_count + 1)));
   HRR(hipMalloc((void**)&d_adj_list, sizeof(vertex_t) * (edge_count)));
   // Swap edge list count with Eend - estart; --> gives error; may add some more

   int nowindex[3];
   nowindex[0] = chunk_size * n_blocks * n_threads / T_Group;
   nowindex[1] = chunk_size * n_blocks;
   nowindex[2] = warpfirstvertex + chunk_size * (n_blocks * n_threads / T_Group);
   // unsigned long long cou=0;
   // int nowindex=0;

   HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
   HRR(hipMemcpy(d_beg_pos, graph_d->beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));
   HRR(hipMemcpy(d_adj_list, graph_d->adj_list, sizeof(vertex_t) * edge_count, hipMemcpyHostToDevice));
   double t1 = wtime();
   double cmp_time;
   HRR(hipMalloc((void**)&BIN_MEM, sizeof(int) * total));

   double total_kernel_use = 0;
   double startKernel, ee = 0;
   for (int i = 0; i < iterator_count; i++)
   {
      HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
      double time_start = clock();
      startKernel = wtime();
      hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
      trust << <n_blocks, n_threads >> > (d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, BUCKET_SIZE, T_Group, G_INDEX, chunk_size, warpfirstvertex);
      HRR(hipDeviceSynchronize());
      ee = wtime();
      total_kernel_use += ee - startKernel;
      // cout << "kernel use " << ee - startKernel << endl;
      cmp_time = clock() - time_start;
   }

   // HRR(hipFree(BIN_MEM));
   cmp_time = cmp_time / CLOCKS_PER_SEC;
   HRR(hipFree(BIN_MEM));

   HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));
   printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
   printf("triangle count %ld \n\n", counter[0]);
   // cout << "total triangle count: " << counter[0] << endl
   //      << endl;
   // printf("avg kernel use %lf s\n\n", total_kernel_use / iterator_count);
   HRR(hipFree(GLOBAL_COUNT));
   HRR(hipFree(G_INDEX));
   HRR(hipFree(d_beg_pos));
   HRR(hipFree(d_adj_list));
   free(prefix);
   delete graph_d;
   args.time = cmp_time;
   args.count = counter[0];

   args.edge_count = edges;
   args.degree = edges / vertex_count;
   args.vertices = vertex_count;
   return args;
}