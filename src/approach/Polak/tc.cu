#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "approach/Polak/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

__global__ void tc::approach::Polak::calculate_triangles(uint edge_count, vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos,
                                                         unsigned long long* results, int deviceCount, int deviceIdx) {
    int from = gridDim.x * blockDim.x * deviceIdx + blockDim.x * blockIdx.x + threadIdx.x;
    int step = deviceCount * gridDim.x * blockDim.x;
    unsigned long long count = 0;

    for (int i = from; i < edge_count; i += step) {
        int u = src_list[i], v = adj_list[i];
        int u_it = beg_pos[u], u_end = beg_pos[u + 1];
        int v_it = beg_pos[v], v_end = beg_pos[v + 1];

        int a = adj_list[u_it], b = adj_list[v_it];
        while (u_it < u_end && v_it < v_end) {
            int d = a - b;
            if (d <= 0) a = adj_list[++u_it];
            if (d >= 0) b = adj_list[++v_it];
            if (d == 0) {
                ++count;
            }
        }
    }

    results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void tc::approach::Polak::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    vertex_t edge_count = gpu_graph.edge_count;

    int grid_size = 640;
    int block_size = 64;
    double t_start, total_kernel_use = 0;
    uint64_t count;

    unsigned long long* d_results;
    HRR(hipMalloc(&d_results, grid_size * block_size * sizeof(unsigned long long)));

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(d_results, 0, grid_size * block_size * sizeof(unsigned long long)));

        t_start = wtime();
        tc::approach::Polak::calculate_triangles<<<grid_size, block_size>>>(edge_count, d_src, d_adj, d_beg_pos, d_results);
        HRR(hipDeviceSynchronize());
        thrust::device_ptr<unsigned long long> ptr(d_results);
        count = thrust::reduce(ptr, ptr + (grid_size * block_size));

        double ee = wtime();
        total_kernel_use += ee - t_start;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - t_start > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, triangle_count, iteration_count, avg kernel time/s
    auto logger = spdlog::get("Polak_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "Polak", gpu_graph.input_dir, count, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'Polak_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", count);

    HRR(hipFree(d_results));
}

void tc::approach::Polak::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "Polak", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("Polak before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::Polak::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Polak after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after Polak runs.");
        }
    }
}
