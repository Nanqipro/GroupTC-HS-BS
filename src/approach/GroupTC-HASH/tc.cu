#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/GroupTC-HASH/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

__device__ int tc::approach::GroupTC_HASH::linear_search_block(int neighbor, int *partition, int len, int bin, int BIN_START) {
    for (;;) {
        len -= GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
        int i = bin + BIN_START;
        int step = 0;
        while (step < len) {
            if (partition[i] == neighbor) {
                return 1;
            }
            i += GroupTC_HASH_block_bucketnum;
            step += 1;
        }
        if (len + GroupTC_HASH_shared_BLOCK_BUCKET_SIZE < 99) break;
        bin++;
    }
    return 0;
}

__device__ int tc::approach::GroupTC_HASH::linear_search_group(int neighbor, int *partition, int len, int bin, int BIN_START) {
    len -= GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
    int i = bin + BIN_START;
    int step = 0;
    while (step < len) {
        if (partition[i] == neighbor) {
            return 1;
        }
        i += GroupTC_HASH_group_bucketnum;
        step += 1;
    }

    return 0;
}

int tc::approach::GroupTC_HASH::my_binary_search(int len, int val, index_t *beg) {
    int l = 0, r = len;
    while (l < r - 1) {
        int mid = (l + r) / 2;
        if (beg[mid + 1] - beg[mid] > val)
            l = mid;
        else
            r = mid;
    }
    if (beg[l + 1] - beg[l] <= val) return -1;
    return l;
}

__global__ void tc::approach::GroupTC_HASH::grouptc_hash(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count, uint vertex_count,
                                                         int *partition, unsigned long long *GLOBAL_COUNT, int T_Group, int *G_INDEX, int CHUNK_SIZE,
                                                         int warpfirstvertex, int warpfirstedge, int nocomputefirstvertex, int nocomputefirstedge) {
    // hashTable bucket 计数器
    __shared__ int bin_count[GroupTC_HASH_block_bucketnum];
    // 共享内存中的 hashTable
    __shared__ int shared_partition[GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE];
    unsigned long long __shared__ G_counter;

    if (threadIdx.x == 0) {
        G_counter = 0;
    }

    int BIN_START = blockIdx.x * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE;
    unsigned long long P_counter = 0;

    // CTA for large degree vertex
    int vertex = blockIdx.x * CHUNK_SIZE;
    int vertex_end = vertex + CHUNK_SIZE;
    __shared__ int ver;

    while (vertex < warpfirstvertex)
    // while (0)
    {
        // if (degree<=USE_CTA) break;
        int group_start = beg_pos[vertex];
        int end = beg_pos[vertex + 1];
        int now = threadIdx.x + group_start;
        // int MODULO = GroupTC_HASH_block_bucketnum - 1;
        // clean bin_count
        // 初始化 hashTable bucket 计数器
        for (int i = threadIdx.x; i < GroupTC_HASH_block_bucketnum; i += GroupTC_HASH_BLOCK_SIZE) bin_count[i] = 0;
        __syncthreads();

        // count hash bin
        // 生成 hashTable
        while (now < end) {
            int temp = adj_list[now];
            int bin = temp & GroupTC_HASH_BLOCK_MODULO;
            int index;
            index = atomicAdd(&bin_count[bin], 1);
            if (index < GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
                shared_partition[index * GroupTC_HASH_block_bucketnum + bin] = temp;
            } else if (index < GroupTC_HASH_BLOCK_BUCKET_SIZE) {
                index = index - GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
                partition[index * GroupTC_HASH_block_bucketnum + bin + BIN_START] = temp;
            }
            now += blockDim.x;
        }
        __syncthreads();

        // list intersection
        now = beg_pos[vertex];
        end = beg_pos[vertex + 1];
        int superwarp_ID = threadIdx.x / 64;
        int superwarp_TID = threadIdx.x % 64;
        int workid = superwarp_TID;
        now = now + superwarp_ID;
        // 获取二跳邻居节点
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += 16;
                workid -= neighbor_degree;
                neighbor = adj_list[now];
                neighbor_start = beg_pos[neighbor];
                neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
            }
            if (now < end) {
                int temp_adj = adj_list[neighbor_start + workid];
                int bin = temp_adj & GroupTC_HASH_BLOCK_MODULO;

                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 3] == temp_adj : 0;
                P_counter += len > 4 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 4] == temp_adj : 0;
                P_counter += len > 5 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 5] == temp_adj : 0;

                if (len > GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
                    P_counter += tc::approach::GroupTC_HASH::linear_search_block(temp_adj, partition, len, bin, BIN_START);
                }
            }
            // __syncthreads();
            workid += 64;
        }

        __syncthreads();
        // if (vertex>1) break;
        vertex++;
        if (vertex == vertex_end) {
            if (threadIdx.x == 0) {
                ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
            }
            __syncthreads();
            vertex = ver;
            vertex_end = vertex + CHUNK_SIZE;
        }
    }

    // EDGE CHUNK for small degree vertex
    __shared__ int group_start;
    __shared__ int group_size;

    int *shared_src = shared_partition + GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
    int *shared_adj_start = shared_src + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;
    int *shared_adj_degree = shared_adj_start + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;

    for (int group_offset = warpfirstedge + blockIdx.x * GroupTC_HASH_EDGE_CHUNK; group_offset < nocomputefirstedge;
         group_offset += gridDim.x * GroupTC_HASH_EDGE_CHUNK) {
        // compute group start and end
        if (threadIdx.x == 0) {
            int src = src_list[group_offset];
            int src_start = beg_pos[src];
            int src_end = beg_pos[src + 1];
            group_start = ((src_start == group_offset) ? src_start : src_end);

            src = src_list[min(group_offset + GroupTC_HASH_EDGE_CHUNK, nocomputefirstedge) - 1];
            group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
        }

        // cache start
        for (int i = threadIdx.x; i < GroupTC_HASH_group_bucketnum; i += blockDim.x) bin_count[i] = 0;

        __syncthreads();

        for (int i = threadIdx.x; i < group_size; i += GroupTC_HASH_BLOCK_SIZE) {
            int temp_src = src_list[i + group_start];
            int temp_adj = adj_list[i + group_start];

            shared_src[i] = temp_src;
            shared_adj_start[i] = beg_pos[temp_adj];
            shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

            int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
            int index = atomicAdd(&bin_count[bin], 1);

            if (index < GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
                shared_partition[index * GroupTC_HASH_group_bucketnum + bin] = temp_adj;
            } else if (index < GroupTC_HASH_GROUP_BUCKET_SIZE) {
                index = index - GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
                partition[index * GroupTC_HASH_group_bucketnum + bin + BIN_START] = temp_adj;
            }
        }
        __syncthreads();

        // compute 2 hop neighbors
        int now = threadIdx.x / GroupTC_HASH_WARP_SIZE;
        int workid = threadIdx.x % GroupTC_HASH_WARP_SIZE;

        while (now < group_size) {
            int neighbor_degree = shared_adj_degree[now];
            while (now < group_size && workid >= neighbor_degree) {
                now += GroupTC_HASH_BLOCK_SIZE / GroupTC_HASH_WARP_SIZE;
                workid -= neighbor_degree;
                neighbor_degree = shared_adj_degree[now];
            }

            if (now < group_size) {
                int temp_src = shared_src[now];
                int temp_adj = adj_list[shared_adj_start[now] + workid];
                int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 3] == temp_adj : 0;

                if (len > GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
                    P_counter += tc::approach::GroupTC_HASH::linear_search_group(temp_adj, partition, len, bin, BIN_START);
                }
            }
            workid += GroupTC_HASH_WARP_SIZE;
        }
        __syncthreads();
    }

    atomicAdd(&G_counter, P_counter);

    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
    }
}

void tc::approach::GroupTC_HASH::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    int grid_size = 1024;
    int block_size = 1024;
    int chunk_size = 1;

    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    index_t *d_beg_pos = gpu_graph.beg_pos;
    vertex_t *d_src_list = gpu_graph.src_list;
    vertex_t *d_adj_list = gpu_graph.adj_list;

    index_t *h_beg_pos = (index_t *)malloc(sizeof(index_t) * (vertex_count + 1));
    HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

    int warpfirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_CTA, h_beg_pos) + 1;
    int warpfirstedge = h_beg_pos[warpfirstvertex];
    int nocomputefirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_WARP, h_beg_pos) + 1;
    int nocomputefirstedge = h_beg_pos[nocomputefirstvertex];

    int T_Group = 32;
    int nowindex[3];
    nowindex[0] = chunk_size * grid_size * block_size / T_Group;
    nowindex[1] = chunk_size * grid_size;
    nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

    int *BIN_MEM;
    unsigned long long *GLOBAL_COUNT;
    int *G_INDEX;

    HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * grid_size * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE));
    HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
    HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));

    HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

    unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);

    double total_kernel_use = 0;
    double startKernel, ee = 0;
    int block_kernel_grid_size = min(max(warpfirstvertex, 1), grid_size);
    int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (GroupTC_HASH_EDGE_CHUNK * 10), grid_size);
    int kernel_grid_size = max(max(block_kernel_grid_size, group_kernel_grid_size), 320);

    spdlog::info("kernel_grid_size {:d}", kernel_grid_size);

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
        startKernel = wtime();
        hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
        tc::approach::GroupTC_HASH::grouptc_hash<<<kernel_grid_size, GroupTC_HASH_BLOCK_SIZE>>>(
            d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, chunk_size, warpfirstvertex,
            warpfirstedge, nocomputefirstvertex, nocomputefirstedge);
        HRR(hipDeviceSynchronize());

        ee = wtime();
        total_kernel_use += ee - startKernel;
    }

    HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));

    // algorithm, dataset, iteration_count, avg compute time/s,
    spdlog::get("GroupTC-HASH_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", counter[0]);

    free(counter);
    free(h_beg_pos);
    HRR(hipFree(BIN_MEM));
    HRR(hipFree(GLOBAL_COUNT));
    HRR(hipFree(G_INDEX));
}

void tc::approach::GroupTC_HASH::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "GroupTC-HASH", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("GroupTC_HASH before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::GroupTC_HASH::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("GroupTC_HASH after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after GroupTC_HASH runs.");
        }
    }
}


// // 就是trust和groupTc-hash都按顶点度数把数据集分成了两部分，你可以测一测，他们分别在这两部分上的运行时间
// #include <hip/hip_runtime_api.h>
// #include <thrust/device_ptr.h>
// #include <thrust/functional.h>
// #include <thrust/reduce.h>
// #include <thrust/sort.h>

// #include <string>

// #include "approach/GroupTC-HASH/tc.h"
// #include "comm/comm.h"
// #include "comm/constant_comm.h"
// #include "comm/cuda_comm.h"
// #include "spdlog/spdlog.h"


// __device__ int tc::approach::GroupTC_HASH::linear_search_block(int neighbor, int *partition, int len, int bin, int BIN_START) {
//     for (;;) {
//         len -= GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
//         int i = bin + BIN_START;
//         int step = 0;
//         while (step < len) {
//             if (partition[i] == neighbor) {
//                 return 1;
//             }
//             i += GroupTC_HASH_block_bucketnum;
//             step += 1;
//         }
//         if (len + GroupTC_HASH_shared_BLOCK_BUCKET_SIZE < 99) break;
//         bin++;
//     }
//     return 0;
// }

// __device__ int tc::approach::GroupTC_HASH::linear_search_group(int neighbor, int *partition, int len, int bin, int BIN_START) {
//     len -= GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//     int i = bin + BIN_START;
//     int step = 0;
//     while (step < len) {
//         if (partition[i] == neighbor) {
//             return 1;
//         }
//         i += GroupTC_HASH_group_bucketnum;
//         step += 1;
//     }

//     return 0;
// }

// int tc::approach::GroupTC_HASH::my_binary_search(int len, int val, index_t *beg) {
//     int l = 0, r = len;
//     while (l < r - 1) {
//         int mid = (l + r) / 2;
//         if (beg[mid + 1] - beg[mid] > val)
//             l = mid;
//         else
//             r = mid;
//     }
//     if (beg[l + 1] - beg[l] <= val) return -1;
//     return l;
// }

// // 新的内核函数：处理高度数顶点
// // 加了边界检查的代码
// __global__ void tc::approach::GroupTC_HASH::grouptc_hash_high_degree(
//     vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count,
//     uint vertex_count, int *partition, unsigned long long *GLOBAL_COUNT, int T_Group,
//     int *G_INDEX, int CHUNK_SIZE, int warpfirstvertex) {

//     // hashTable bucket 计数器
//     __shared__ int bin_count[GroupTC_HASH_block_bucketnum];
//     // 共享内存中的 hashTable
//     __shared__ int shared_partition[GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE];

//     unsigned long long __shared__ G_counter;

//     if (threadIdx.x == 0) {
//         G_counter = 0;
//     }

//     int BIN_START = blockIdx.x * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE;
//     unsigned long long P_counter = 0;

//     // CTA for large degree vertex
//     int vertex = blockIdx.x * CHUNK_SIZE;
//     int vertex_end = vertex + CHUNK_SIZE;
//     __shared__ int ver;

//     while (vertex < warpfirstvertex) {
//         int group_start = beg_pos[vertex];
//         int end = beg_pos[vertex + 1];
//         int now = threadIdx.x + group_start;

//         // 初始化 hashTable bucket 计数器
//         for (int i = threadIdx.x; i < GroupTC_HASH_block_bucketnum; i += GroupTC_HASH_BLOCK_SIZE)
//             bin_count[i] = 0;
//         __syncthreads();

//         // 生成 hashTable
//         while (now < end) {
//             int temp = adj_list[now];
//             int bin = temp & GroupTC_HASH_BLOCK_MODULO;
//             int index;
//             index = atomicAdd(&bin_count[bin], 1);

//             if (index < GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                 int idx = index * GroupTC_HASH_block_bucketnum + bin;
//                 if (idx < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                     shared_partition[idx] = temp;
//                 } else {
//                     // 超出 shared_partition 数组范围，处理错误或丢弃数据
//                     // 可以选择记录错误或进行其他处理
//                 }
//             } else if (index < GroupTC_HASH_BLOCK_BUCKET_SIZE) {
//                 index = index - GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
//                 int idx = index * GroupTC_HASH_block_bucketnum + bin + BIN_START;
//                 int partition_size_per_block = GroupTC_HASH_block_bucketnum * (GroupTC_HASH_BLOCK_BUCKET_SIZE - GroupTC_HASH_shared_BLOCK_BUCKET_SIZE);
//                 if (idx - BIN_START < partition_size_per_block) {
//                     partition[idx] = temp;
//                 } else {
//                     // 超出 partition 数组范围，处理错误或丢弃数据
//                     // 可以选择记录错误或进行其他处理
//                 }
//             }
//             now += blockDim.x;
//         }
//         __syncthreads();

//         // 列表交集计算
//         now = beg_pos[vertex];
//         end = beg_pos[vertex + 1];
//         int superwarp_ID = threadIdx.x / 64;
//         int superwarp_TID = threadIdx.x % 64;
//         int workid = superwarp_TID;
//         now = now + superwarp_ID;

//         while (now < end) {
//             // 获取二跳邻居节点
//             int neighbor = adj_list[now];
//             int neighbor_start = beg_pos[neighbor];
//             int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;

//             // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点
//             while (now < end && workid >= neighbor_degree) {
//                 now += 16;
//                 workid -= neighbor_degree;
//                 if (now < end) {
//                     neighbor = adj_list[now];
//                     neighbor_start = beg_pos[neighbor];
//                     neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
//                 }
//             }

//             if (now < end && workid < neighbor_degree) {
//                 int temp_adj = adj_list[neighbor_start + workid];
//                 int bin = temp_adj & GroupTC_HASH_BLOCK_MODULO;

//                 int len = bin_count[bin];

//                 // 检查 len 是否大于 0，避免访问越界
//                 if (len > 0) {
//                     int idx0 = bin + GroupTC_HASH_block_bucketnum * 0;
//                     if (idx0 < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx0] == temp_adj;
//                     }
//                 }
//                 if (len > 1) {
//                     int idx1 = bin + GroupTC_HASH_block_bucketnum * 1;
//                     if (idx1 < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx1] == temp_adj;
//                     }
//                 }
//                 if (len > 2) {
//                     int idx2 = bin + GroupTC_HASH_block_bucketnum * 2;
//                     if (idx2 < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx2] == temp_adj;
//                     }
//                 }
//                 if (len > 3) {
//                     int idx3 = bin + GroupTC_HASH_block_bucketnum * 3;
//                     if (idx3 < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx3] == temp_adj;
//                     }
//                 }
//                 if (len > 4) {
//                     int idx4 = bin + GroupTC_HASH_block_bucketnum * 4;
//                     if (idx4 < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx4] == temp_adj;
//                     }
//                 }
//                 if (len > 5) {
//                     int idx5 = bin + GroupTC_HASH_block_bucketnum * 5;
//                     if (idx5 < GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx5] == temp_adj;
//                     }
//                 }

//                 if (len > GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                     P_counter += tc::approach::GroupTC_HASH::linear_search_block(temp_adj, partition, len, bin, BIN_START);
//                 }
//             }
//             workid += 64;
//         }

//         __syncthreads();
//         vertex++;
//         if (vertex == vertex_end) {
//             if (threadIdx.x == 0) {
//                 ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
//             }
//             __syncthreads();
//             vertex = ver;
//             vertex_end = vertex + CHUNK_SIZE;
//         }
//     }

//     atomicAdd(&G_counter, P_counter);

//     __syncthreads();
//     if (threadIdx.x == 0) {
//         atomicAdd(&GLOBAL_COUNT[0], G_counter);
//     }
// }


// // 新的内核函数：处理低度数顶点
// // 加了边界检查的代码
// __global__ void tc::approach::GroupTC_HASH::grouptc_hash_low_degree(
//     vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count,
//     uint vertex_count, int *partition, unsigned long long *GLOBAL_COUNT, int T_Group,
//     int *G_INDEX, int warpfirstedge, int nocomputefirstedge) {

//     extern __shared__ int shared_mem[];

//     int *bin_count = shared_mem;
//     int *shared_partition = bin_count + GroupTC_HASH_group_bucketnum;
//     int *shared_src = shared_partition + GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//     int *shared_adj_start = shared_src + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;
//     int *shared_adj_degree = shared_adj_start + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;

//     unsigned long long __shared__ G_counter;

//     if (threadIdx.x == 0) {
//         G_counter = 0;
//     }

//     int BIN_START = blockIdx.x * GroupTC_HASH_group_bucketnum * GroupTC_HASH_GROUP_BUCKET_SIZE;
//     unsigned long long P_counter = 0;

//     // EDGE CHUNK for small degree vertex
//     __shared__ int group_start;
//     __shared__ int group_size;

//     for (int group_offset = warpfirstedge + blockIdx.x * GroupTC_HASH_EDGE_CHUNK;
//          group_offset < nocomputefirstedge;
//          group_offset += gridDim.x * GroupTC_HASH_EDGE_CHUNK) {

//         // compute group start and end
//         if (threadIdx.x == 0) {
//             int src = src_list[group_offset];
//             int src_start = beg_pos[src];
//             int src_end = beg_pos[src + 1];
//             group_start = ((src_start == group_offset) ? src_start : src_end);

//             src = src_list[min(group_offset + GroupTC_HASH_EDGE_CHUNK, nocomputefirstedge) - 1];
//             group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
//         }

//         // cache start
//         for (int i = threadIdx.x; i < GroupTC_HASH_group_bucketnum; i += blockDim.x)
//             bin_count[i] = 0;

//         __syncthreads();

//         for (int i = threadIdx.x; i < group_size; i += GroupTC_HASH_BLOCK_SIZE) {
//             int temp_src = src_list[i + group_start];
//             int temp_adj = adj_list[i + group_start];

//             shared_src[i] = temp_src;
//             shared_adj_start[i] = beg_pos[temp_adj];
//             shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

//             int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
//             int index = atomicAdd(&bin_count[bin], 1);

//             if (index < GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                 int idx = index * GroupTC_HASH_group_bucketnum + bin;
//                 if (idx < GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                     shared_partition[idx] = temp_adj;
//                 } else {
//                     // 超出 shared_partition 数组范围，处理错误或丢弃数据
//                 }
//             } else if (index < GroupTC_HASH_GROUP_BUCKET_SIZE) {
//                 index = index - GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//                 int idx = index * GroupTC_HASH_group_bucketnum + bin + BIN_START;
//                 int partition_size_per_block = GroupTC_HASH_group_bucketnum * (GroupTC_HASH_GROUP_BUCKET_SIZE - GroupTC_HASH_shared_GROUP_BUCKET_SIZE);
//                 if (idx - BIN_START < partition_size_per_block) {
//                     partition[idx] = temp_adj;
//                 } else {
//                     // 超出 partition 数组范围，处理错误或丢弃数据
//                 }
//             }
//         }
//         __syncthreads();

//         // compute 2 hop neighbors
//         int now = threadIdx.x / GroupTC_HASH_WARP_SIZE;
//         int workid = threadIdx.x % GroupTC_HASH_WARP_SIZE;

//         while (now < group_size) {
//             int neighbor_degree = shared_adj_degree[now];
//             while (now < group_size && workid >= neighbor_degree) {
//                 now += GroupTC_HASH_BLOCK_SIZE / GroupTC_HASH_WARP_SIZE;
//                 workid -= neighbor_degree;
//                 if (now < group_size) {
//                     neighbor_degree = shared_adj_degree[now];
//                 }
//             }

//             if (now < group_size && workid < neighbor_degree) {
//                 int temp_src = shared_src[now];
//                 int temp_adj = adj_list[shared_adj_start[now] + workid];
//                 int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
//                 int len = bin_count[bin];

//                 if (len > 0) {
//                     int idx0 = bin + GroupTC_HASH_group_bucketnum * 0;
//                     if (idx0 < GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx0] == temp_adj;
//                     }
//                 }
//                 if (len > 1) {
//                     int idx1 = bin + GroupTC_HASH_group_bucketnum * 1;
//                     if (idx1 < GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx1] == temp_adj;
//                     }
//                 }
//                 if (len > 2) {
//                     int idx2 = bin + GroupTC_HASH_group_bucketnum * 2;
//                     if (idx2 < GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx2] == temp_adj;
//                     }
//                 }
//                 if (len > 3) {
//                     int idx3 = bin + GroupTC_HASH_group_bucketnum * 3;
//                     if (idx3 < GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                         P_counter += shared_partition[idx3] == temp_adj;
//                     }
//                 }

//                 if (len > GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                     P_counter += tc::approach::GroupTC_HASH::linear_search_group(temp_adj, partition, len, bin, BIN_START);
//                 }
//             }
//             workid += GroupTC_HASH_WARP_SIZE;
//         }
//         __syncthreads();
//     }

//     atomicAdd(&G_counter, P_counter);

//     __syncthreads();
//     if (threadIdx.x == 0) {
//         atomicAdd(&GLOBAL_COUNT[0], G_counter);
//     }
// }



// void tc::approach::GroupTC_HASH::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
//     std::string file = gpu_graph.input_dir;
//     int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
//     spdlog::info("Run algorithm {}", key_space);
//     spdlog::info("Dataset {}", file);
//     spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
//     int device = config.GetInteger(key_space, "device", 1);
//     HRR(hipSetDevice(device));

//     int grid_size = 1024;
//     int block_size = 1024;
//     int chunk_size = 1;

//     uint vertex_count = gpu_graph.vertex_count;
//     uint edge_count = gpu_graph.edge_count;
//     index_t *d_beg_pos = gpu_graph.beg_pos;
//     vertex_t *d_src_list = gpu_graph.src_list;
//     vertex_t *d_adj_list = gpu_graph.adj_list;

//     index_t *h_beg_pos = (index_t *)malloc(sizeof(index_t) * (vertex_count + 1));
//     HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

//     int warpfirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_CTA, h_beg_pos) + 1;
//     int warpfirstedge = h_beg_pos[warpfirstvertex];
//     int nocomputefirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_WARP, h_beg_pos) + 1;
//     int nocomputefirstedge = h_beg_pos[nocomputefirstvertex];

//     int T_Group = 32;
//     int nowindex[3];
//     nowindex[0] = chunk_size * grid_size * block_size / T_Group;
//     nowindex[1] = chunk_size * grid_size;
//     nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

//     int *BIN_MEM;
//     unsigned long long *GLOBAL_COUNT;
//     int *G_INDEX;

//     HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * grid_size * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE));
//     HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
//     HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));

//     HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

//     unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);

//     // double total_kernel_use = 0;
//     // double startKernel, ee = 0;
//     int block_kernel_grid_size = min(max(warpfirstvertex, 1), grid_size);
//     int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (GroupTC_HASH_EDGE_CHUNK * 10), grid_size);
//     int kernel_grid_size = max(max(block_kernel_grid_size, group_kernel_grid_size), 320);

//     spdlog::info("kernel_grid_size {:d}", kernel_grid_size);

//     // 定义 CUDA 事件
//     hipEvent_t start_high, stop_high;
//     hipEvent_t start_low, stop_low;
//     hipEventCreate(&start_high);
//     hipEventCreate(&stop_high);
//     hipEventCreate(&start_low);
//     hipEventCreate(&stop_low);

//     // 计时变量
//     float time_high_degree = 0.0f;
//     float time_low_degree = 0.0f;

//     for (int i = 0; i < iteration_count; i++) {
//         HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
//         hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);


//          // 如果需要，计算高度数顶点处理内核的共享内存大小
//         size_t shared_memory_size_high = sizeof(int) * (
//             GroupTC_HASH_block_bucketnum +  // bin_count
//             GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE  // shared_partition
//             // 如果还有其他共享内存数组，继续累加
//         );
//         // 调用高度数顶点处理内核并计时
//         // hipEventRecord(start_high, 0);
//         // tc::approach::GroupTC_HASH::grouptc_hash_high_degree<<<block_kernel_grid_size, GroupTC_HASH_BLOCK_SIZE>>>(
//         //     d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, chunk_size, warpfirstvertex);
//         // hipEventRecord(stop_high, 0);
//         // hipEventSynchronize(stop_high);
//         // float time_temp_high = 0.0f;
//         // hipEventElapsedTime(&time_temp_high, start_high, stop_high);
//         // time_high_degree += time_temp_high;
//         hipEventRecord(start_high, 0);
//         tc::approach::GroupTC_HASH::grouptc_hash_high_degree<<<block_kernel_grid_size, GroupTC_HASH_BLOCK_SIZE, shared_memory_size_high>>>(
//             d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, chunk_size, warpfirstvertex);
//         hipEventRecord(stop_high, 0);
//         hipEventSynchronize(stop_high);
//         float time_temp_high = 0.0f;
//         hipEventElapsedTime(&time_temp_high, start_high, stop_high);
//         time_high_degree += time_temp_high;

//         // 调用低度数顶点处理内核并计时
//         // hipEventRecord(start_low, 0);
//         // tc::approach::GroupTC_HASH::grouptc_hash_low_degree<<<group_kernel_grid_size, GroupTC_HASH_BLOCK_SIZE>>>(
//         //     d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, warpfirstedge, nocomputefirstedge);
//         // hipEventRecord(stop_low, 0);
//         // hipEventSynchronize(stop_low);
//         // float time_temp_low = 0.0f;
//         // hipEventElapsedTime(&time_temp_low, start_low, stop_low);
//         // time_low_degree += time_temp_low;
//         // 计算共享内存大小
//         size_t shared_memory_size_low = sizeof(int) * (
//             GroupTC_HASH_group_bucketnum +  // bin_count
//             GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE +  // shared_partition
//             GroupTC_HASH_shared_CHUNK_CACHE_SIZE * 3  // shared_src, shared_adj_start, shared_adj_degree
//         );
//         hipEventRecord(start_low, 0);
//         tc::approach::GroupTC_HASH::grouptc_hash_low_degree<<<group_kernel_grid_size, GroupTC_HASH_BLOCK_SIZE, shared_memory_size_low>>>(
//             d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, warpfirstedge, nocomputefirstedge);
//         hipEventRecord(stop_low, 0);
//         hipEventSynchronize(stop_low);
//         float time_temp_low = 0.0f;
//         hipEventElapsedTime(&time_temp_low, start_low, stop_low);
//         time_low_degree += time_temp_low;


//     }

//     // 计算平均时间
//     time_high_degree /= iteration_count;
//     time_low_degree /= iteration_count;

//     // 输出时间
//     spdlog::info("High degree vertices processing time: {:.6f} ms", time_high_degree);
//     spdlog::info("Low degree vertices processing time: {:.6f} ms", time_low_degree);

//     HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));

//     // 记录日志
//     spdlog::get("GroupTC-HASH_file_logger")
//         ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}\t{5:.6f}\t{6:.6f}", "GroupTC-HASH", gpu_graph.input_dir, counter[0], iteration_count,
//                (time_high_degree + time_low_degree) / 1000.0, time_high_degree / 1000.0, time_low_degree / 1000.0);

//     spdlog::info("Iter {0}, total kernel time {1:.6f} s", iteration_count, (time_high_degree + time_low_degree) / 1000.0);
//     spdlog::info("Triangle count {:d}", counter[0]);

//     free(counter);
//     free(h_beg_pos);
//     HRR(hipFree(BIN_MEM));
//     HRR(hipFree(GLOBAL_COUNT));
//     HRR(hipFree(G_INDEX));

//     // 销毁 CUDA 事件
//     hipEventDestroy(start_high);
//     hipEventDestroy(stop_high);
//     hipEventDestroy(start_low);
//     hipEventDestroy(stop_low);
// }

// void tc::approach::GroupTC_HASH::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
//     bool run = config.GetBoolean("comm", "GroupTC-HASH", false);
//     if (run) {
//         size_t free_byte, total_byte, available_byte;
//         HRR(hipMemGetInfo(&free_byte, &total_byte));
//         available_byte = total_byte - free_byte;
//         spdlog::debug("GroupTC_HASH before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

//         tc::approach::GroupTC_HASH::gpu_run(config, gpu_graph);

//         HRR(hipMemGetInfo(&free_byte, &total_byte));
//         spdlog::debug("GroupTC_HASH after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
//         if (available_byte != total_byte - free_byte) {
//             spdlog::warn("There is GPU memory that is not freed after GroupTC_HASH runs.");
//         }
//     }
// }
