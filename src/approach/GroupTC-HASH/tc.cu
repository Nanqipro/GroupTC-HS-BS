#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/GroupTC-HASH/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

__device__ int tc::approach::GroupTC_HASH::linear_search_block(int neighbor, int *partition, int len, int bin, int BIN_START) {
    for (;;) {
        len -= GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
        int i = bin + BIN_START;
        int step = 0;
        while (step < len) {
            if (partition[i] == neighbor) {
                return 1;
            }
            i += GroupTC_HASH_block_bucketnum;
            step += 1;
        }
        if (len + GroupTC_HASH_shared_BLOCK_BUCKET_SIZE < 99) break;
        bin++;
    }
    return 0;
}

__device__ int tc::approach::GroupTC_HASH::linear_search_group(int neighbor, int *partition, int len, int bin, int BIN_START) {
    len -= GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
    int i = bin + BIN_START;
    int step = 0;
    while (step < len) {
        if (partition[i] == neighbor) {
            return 1;
        }
        i += GroupTC_HASH_group_bucketnum;
        step += 1;
    }

    return 0;
}

int tc::approach::GroupTC_HASH::my_binary_search(int len, int val, index_t *beg) {
    int l = 0, r = len;
    while (l < r - 1) {
        int mid = (l + r) / 2;
        if (beg[mid + 1] - beg[mid] > val)
            l = mid;
        else
            r = mid;
    }
    if (beg[l + 1] - beg[l] <= val) return -1;
    return l;
}

__global__ void tc::approach::GroupTC_HASH::grouptc_hash(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count, uint vertex_count,
                                                         int *partition, unsigned long long *GLOBAL_COUNT, int T_Group, int *G_INDEX, int CHUNK_SIZE,
                                                         int warpfirstvertex, int warpfirstedge, int nocomputefirstvertex, int nocomputefirstedge) {
    // hashTable bucket 计数器
    __shared__ int bin_count[GroupTC_HASH_block_bucketnum];
    // 共享内存中的 hashTable
    __shared__ int shared_partition[GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE];
    unsigned long long __shared__ G_counter;

    if (threadIdx.x == 0) {
        G_counter = 0;
    }

    int BIN_START = blockIdx.x * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE;
    unsigned long long P_counter = 0;

    // CTA for large degree vertex
    int vertex = blockIdx.x * CHUNK_SIZE;
    int vertex_end = vertex + CHUNK_SIZE;
    __shared__ int ver;

    while (vertex < warpfirstvertex)
    // while (0)
    {
        // if (degree<=USE_CTA) break;
        int group_start = beg_pos[vertex];
        int end = beg_pos[vertex + 1];
        int now = threadIdx.x + group_start;
        // int MODULO = GroupTC_HASH_block_bucketnum - 1;
        // clean bin_count
        // 初始化 hashTable bucket 计数器
        for (int i = threadIdx.x; i < GroupTC_HASH_block_bucketnum; i += GroupTC_HASH_BLOCK_SIZE) bin_count[i] = 0;
        __syncthreads();

        // count hash bin
        // 生成 hashTable
        while (now < end) {
            int temp = adj_list[now];
            int bin = temp & GroupTC_HASH_BLOCK_MODULO;
            int index;
            index = atomicAdd(&bin_count[bin], 1);
            if (index < GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
                shared_partition[index * GroupTC_HASH_block_bucketnum + bin] = temp;
            } else if (index < GroupTC_HASH_BLOCK_BUCKET_SIZE) {
                index = index - GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
                partition[index * GroupTC_HASH_block_bucketnum + bin + BIN_START] = temp;
            }
            now += blockDim.x;
        }
        __syncthreads();

        // list intersection
        now = beg_pos[vertex];
        end = beg_pos[vertex + 1];
        int superwarp_ID = threadIdx.x / 64;
        int superwarp_TID = threadIdx.x % 64;
        int workid = superwarp_TID;
        now = now + superwarp_ID;
        // 获取二跳邻居节点
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += 16;
                workid -= neighbor_degree;
                neighbor = adj_list[now];
                neighbor_start = beg_pos[neighbor];
                neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
            }
            if (now < end) {
                int temp_adj = adj_list[neighbor_start + workid];
                int bin = temp_adj & GroupTC_HASH_BLOCK_MODULO;

                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 3] == temp_adj : 0;
                P_counter += len > 4 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 4] == temp_adj : 0;
                P_counter += len > 5 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 5] == temp_adj : 0;

                if (len > GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
                    P_counter += tc::approach::GroupTC_HASH::linear_search_block(temp_adj, partition, len, bin, BIN_START);
                }
            }
            // __syncthreads();
            workid += 64;
        }

        __syncthreads();
        // if (vertex>1) break;
        vertex++;
        if (vertex == vertex_end) {
            if (threadIdx.x == 0) {
                ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
            }
            __syncthreads();
            vertex = ver;
            vertex_end = vertex + CHUNK_SIZE;
        }
    }

    // EDGE CHUNK for small degree vertex
    __shared__ int group_start;
    __shared__ int group_size;

    int *shared_src = shared_partition + GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
    int *shared_adj_start = shared_src + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;
    int *shared_adj_degree = shared_adj_start + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;

    for (int group_offset = warpfirstedge + blockIdx.x * GroupTC_HASH_EDGE_CHUNK; group_offset < nocomputefirstedge;
         group_offset += gridDim.x * GroupTC_HASH_EDGE_CHUNK) {
        // compute group start and end
        if (threadIdx.x == 0) {
            int src = src_list[group_offset];
            int src_start = beg_pos[src];
            int src_end = beg_pos[src + 1];
            group_start = ((src_start == group_offset) ? src_start : src_end);

            src = src_list[min(group_offset + GroupTC_HASH_EDGE_CHUNK, nocomputefirstedge) - 1];
            group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
        }

        // cache start
        for (int i = threadIdx.x; i < GroupTC_HASH_group_bucketnum; i += blockDim.x) bin_count[i] = 0;

        __syncthreads();

        for (int i = threadIdx.x; i < group_size; i += GroupTC_HASH_BLOCK_SIZE) {
            int temp_src = src_list[i + group_start];
            int temp_adj = adj_list[i + group_start];

            shared_src[i] = temp_src;
            shared_adj_start[i] = beg_pos[temp_adj];
            shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

            int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
            int index = atomicAdd(&bin_count[bin], 1);

            if (index < GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
                shared_partition[index * GroupTC_HASH_group_bucketnum + bin] = temp_adj;
            } else if (index < GroupTC_HASH_GROUP_BUCKET_SIZE) {
                index = index - GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
                partition[index * GroupTC_HASH_group_bucketnum + bin + BIN_START] = temp_adj;
            }
        }
        __syncthreads();

        // compute 2 hop neighbors
        int now = threadIdx.x / GroupTC_HASH_WARP_SIZE;
        int workid = threadIdx.x % GroupTC_HASH_WARP_SIZE;

        while (now < group_size) {
            int neighbor_degree = shared_adj_degree[now];
            while (now < group_size && workid >= neighbor_degree) {
                now += GroupTC_HASH_BLOCK_SIZE / GroupTC_HASH_WARP_SIZE;
                workid -= neighbor_degree;
                neighbor_degree = shared_adj_degree[now];
            }

            if (now < group_size) {
                int temp_src = shared_src[now];
                int temp_adj = adj_list[shared_adj_start[now] + workid];
                int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
                int len = bin_count[bin];

                P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 0] == temp_adj : 0;
                P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 1] == temp_adj : 0;
                P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 2] == temp_adj : 0;
                P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 3] == temp_adj : 0;

                if (len > GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
                    P_counter += tc::approach::GroupTC_HASH::linear_search_group(temp_adj, partition, len, bin, BIN_START);
                }
            }
            workid += GroupTC_HASH_WARP_SIZE;
        }
        __syncthreads();
    }

    atomicAdd(&G_counter, P_counter);

    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
    }
}

void tc::approach::GroupTC_HASH::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    int grid_size = 1024;
    int block_size = 1024;
    int chunk_size = 1;

    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    index_t *d_beg_pos = gpu_graph.beg_pos;
    vertex_t *d_src_list = gpu_graph.src_list;
    vertex_t *d_adj_list = gpu_graph.adj_list;

    index_t *h_beg_pos = (index_t *)malloc(sizeof(index_t) * (vertex_count + 1));
    HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

    int warpfirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_CTA, h_beg_pos) + 1;
    int warpfirstedge = h_beg_pos[warpfirstvertex];
    int nocomputefirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_WARP, h_beg_pos) + 1;
    int nocomputefirstedge = h_beg_pos[nocomputefirstvertex];

    int T_Group = 32;
    int nowindex[3];
    nowindex[0] = chunk_size * grid_size * block_size / T_Group;
    nowindex[1] = chunk_size * grid_size;
    nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

    int *BIN_MEM;
    unsigned long long *GLOBAL_COUNT;
    int *G_INDEX;

    HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * grid_size * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE));
    HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
    HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));

    HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

    unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);

    double total_kernel_use = 0;
    double startKernel, ee = 0;
    int block_kernel_grid_size = min(max(warpfirstvertex, 1), grid_size);
    int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (GroupTC_HASH_EDGE_CHUNK * 10), grid_size);
    int kernel_grid_size = max(max(block_kernel_grid_size, group_kernel_grid_size), 320);

    spdlog::info("kernel_grid_size {:d}", kernel_grid_size);

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
        startKernel = wtime();
        hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
        tc::approach::GroupTC_HASH::grouptc_hash<<<kernel_grid_size, GroupTC_HASH_BLOCK_SIZE>>>(
            d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, chunk_size, warpfirstvertex,
            warpfirstedge, nocomputefirstvertex, nocomputefirstedge);
        HRR(hipDeviceSynchronize());

        ee = wtime();
        total_kernel_use += ee - startKernel;
    }

    HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));

    // algorithm, dataset, iteration_count, avg compute time/s,
    spdlog::get("GroupTC-HASH_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC-Hash", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", counter[0]);

    free(counter);
    free(h_beg_pos);
    HRR(hipFree(BIN_MEM));
    HRR(hipFree(GLOBAL_COUNT));
    HRR(hipFree(G_INDEX));
}

void tc::approach::GroupTC_HASH::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "GroupTC-HASH", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("GroupTC_HASH before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::GroupTC_HASH::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("GroupTC_HASH after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after GroupTC_HASH runs.");
        }
    }
}


// // GroupTC-Hash分两部分测时间   1构建索引时间   2搜索时间
// #include <hip/hip_runtime_api.h>
// #include <thrust/device_ptr.h>
// #include <thrust/functional.h>
// #include <thrust/reduce.h>
// #include <thrust/sort.h>

// #include <string>

// #include "approach/GroupTC-HASH/tc.h"
// #include "comm/comm.h"
// #include "comm/constant_comm.h"
// #include "comm/cuda_comm.h"
// #include "spdlog/spdlog.h"

// __device__ int tc::approach::GroupTC_HASH::linear_search_block(int neighbor, int *partition, int len, int bin, int BIN_START) {
//     for (;;) {
//         len -= GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
//         int i = bin + BIN_START;
//         int step = 0;
//         while (step < len) {
//             if (partition[i] == neighbor) {
//                 return 1;
//             }
//             i += GroupTC_HASH_block_bucketnum;
//             step += 1;
//         }
//         if (len + GroupTC_HASH_shared_BLOCK_BUCKET_SIZE < 99) break;
//         bin++;
//     }
//     return 0;
// }

// __device__ int tc::approach::GroupTC_HASH::linear_search_group(int neighbor, int *partition, int len, int bin, int BIN_START) {
//     len -= GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//     int i = bin + BIN_START;
//     int step = 0;
//     while (step < len) {
//         if (partition[i] == neighbor) {
//             return 1;
//         }
//         i += GroupTC_HASH_group_bucketnum;
//         step += 1;
//     }

//     return 0;
// }

// int tc::approach::GroupTC_HASH::my_binary_search(int len, int val, index_t *beg) {
//     int l = 0, r = len;
//     while (l < r - 1) {
//         int mid = (l + r) / 2;
//         if (beg[mid + 1] - beg[mid] > val)
//             l = mid;
//         else
//             r = mid;
//     }
//     if (beg[l + 1] - beg[l] <= val) return -1;
//     return l;
// }

// __global__ void tc::approach::GroupTC_HASH::grouptc_hash(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count, uint vertex_count,
//                                                          int *partition, unsigned long long *GLOBAL_COUNT, int T_Group, int *G_INDEX, int CHUNK_SIZE,
//                                                          int warpfirstvertex, int warpfirstedge, int nocomputefirstvertex, int nocomputefirstedge) {
//     // hashTable bucket 计数器
//     __shared__ int bin_count[GroupTC_HASH_block_bucketnum];
//     // 共享内存中的 hashTable
//     __shared__ int shared_partition[GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE];
//     unsigned long long __shared__ G_counter;

//     if (threadIdx.x == 0) {
//         G_counter = 0;
//     }

//     int BIN_START = blockIdx.x * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE;
//     unsigned long long P_counter = 0;

//     // CTA for large degree vertex
//     int vertex = blockIdx.x * CHUNK_SIZE;
//     int vertex_end = vertex + CHUNK_SIZE;
//     __shared__ int ver;

//     // while (vertex < warpfirstvertex)
//     // 高度数顶点
//     // 使用CTA（线程块）进行并行处理
//     // 包括初始化哈希表、计数桶、插入邻居等操作
//     while (0)
//     {
//         // if (degree<=USE_CTA) break;
//         int group_start = beg_pos[vertex];
//         int end = beg_pos[vertex + 1];
//         int now = threadIdx.x + group_start;
//         // int MODULO = GroupTC_HASH_block_bucketnum - 1;
//         // clean bin_count
//         // 初始化 hashTable bucket 计数器
//         for (int i = threadIdx.x; i < GroupTC_HASH_block_bucketnum; i += GroupTC_HASH_BLOCK_SIZE) bin_count[i] = 0;
//         __syncthreads();

//         // count hash bin
//         // 生成 hashTable
//         while (now < end) {
//             int temp = adj_list[now];
//             int bin = temp & GroupTC_HASH_BLOCK_MODULO;
//             int index;
//             index = atomicAdd(&bin_count[bin], 1);
//             if (index < GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                 shared_partition[index * GroupTC_HASH_block_bucketnum + bin] = temp;
//             } else if (index < GroupTC_HASH_BLOCK_BUCKET_SIZE) {
//                 index = index - GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
//                 partition[index * GroupTC_HASH_block_bucketnum + bin + BIN_START] = temp;
//             }
//             now += blockDim.x;
//         }
//         __syncthreads();

//         if(0){
//             // list intersection
//             now = beg_pos[vertex];
//             end = beg_pos[vertex + 1];
//             int superwarp_ID = threadIdx.x / 64;
//             int superwarp_TID = threadIdx.x % 64;
//             int workid = superwarp_TID;
//             now = now + superwarp_ID;
//             // 获取二跳邻居节点
//             int neighbor = adj_list[now];
//             int neighbor_start = beg_pos[neighbor];
//             int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
//             while (now < end) {
//                 // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
//                 while (now < end && workid >= neighbor_degree) {
//                     now += 16;
//                     workid -= neighbor_degree;
//                     neighbor = adj_list[now];
//                     neighbor_start = beg_pos[neighbor];
//                     neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
//                 }
//                 if (now < end) {
//                     int temp_adj = adj_list[neighbor_start + workid];
//                     int bin = temp_adj & GroupTC_HASH_BLOCK_MODULO;

//                     int len = bin_count[bin];

//                     P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 0] == temp_adj : 0;
//                     P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 1] == temp_adj : 0;
//                     P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 2] == temp_adj : 0;
//                     P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 3] == temp_adj : 0;
//                     P_counter += len > 4 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 4] == temp_adj : 0;
//                     P_counter += len > 5 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 5] == temp_adj : 0;

//                     if (len > GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                         P_counter += tc::approach::GroupTC_HASH::linear_search_block(temp_adj, partition, len, bin, BIN_START);
//                     }
//                 }
//                 // __syncthreads();
//                 workid += 64;
//             }
//         }
//         __syncthreads();
//         // if (vertex>1) break;
//         vertex++;
//         if (vertex == vertex_end) {
//             if (threadIdx.x == 0) {
//                 ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
//             }
//             __syncthreads();
//             vertex = ver;
//             vertex_end = vertex + CHUNK_SIZE;
//         }
//     }

//     // 处理低频顶点（低度数顶点）
//     // EDGE CHUNK for small degree vertex
//     __shared__ int group_start;
//     __shared__ int group_size;

//     int *shared_src = shared_partition + GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//     int *shared_adj_start = shared_src + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;
//     int *shared_adj_degree = shared_adj_start + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;

//     for (int group_offset = warpfirstedge + blockIdx.x * GroupTC_HASH_EDGE_CHUNK; group_offset < nocomputefirstedge;
//          group_offset += gridDim.x * GroupTC_HASH_EDGE_CHUNK) {
//         // compute group start and end
//         if (threadIdx.x == 0) {
//             int src = src_list[group_offset];
//             int src_start = beg_pos[src];
//             int src_end = beg_pos[src + 1];
//             group_start = ((src_start == group_offset) ? src_start : src_end);

//             src = src_list[min(group_offset + GroupTC_HASH_EDGE_CHUNK, nocomputefirstedge) - 1];
//             group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
//         }

//         // cache start
//         for (int i = threadIdx.x; i < GroupTC_HASH_group_bucketnum; i += blockDim.x) bin_count[i] = 0;

//         __syncthreads();

//         for (int i = threadIdx.x; i < group_size; i += GroupTC_HASH_BLOCK_SIZE) {
//             int temp_src = src_list[i + group_start];
//             int temp_adj = adj_list[i + group_start];

//             shared_src[i] = temp_src;
//             shared_adj_start[i] = beg_pos[temp_adj];
//             shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

//             int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
//             int index = atomicAdd(&bin_count[bin], 1);

//             if (index < GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                 shared_partition[index * GroupTC_HASH_group_bucketnum + bin] = temp_adj;
//             } else if (index < GroupTC_HASH_GROUP_BUCKET_SIZE) {
//                 index = index - GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//                 partition[index * GroupTC_HASH_group_bucketnum + bin + BIN_START] = temp_adj;
//             }
//         }
//         __syncthreads();

//         if(0){
//             // compute 2 hop neighbors
//             int now = threadIdx.x / GroupTC_HASH_WARP_SIZE;
//             int workid = threadIdx.x % GroupTC_HASH_WARP_SIZE;

//             while (now < group_size) {
//                 int neighbor_degree = shared_adj_degree[now];
//                 while (now < group_size && workid >= neighbor_degree) {
//                     now += GroupTC_HASH_BLOCK_SIZE / GroupTC_HASH_WARP_SIZE;
//                     workid -= neighbor_degree;
//                     neighbor_degree = shared_adj_degree[now];
//                 }

//                 if (now < group_size) {
//                     int temp_src = shared_src[now];
//                     int temp_adj = adj_list[shared_adj_start[now] + workid];
//                     int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
//                     int len = bin_count[bin];

//                     P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 0] == temp_adj : 0;
//                     P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 1] == temp_adj : 0;
//                     P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 2] == temp_adj : 0;
//                     P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 3] == temp_adj : 0;

//                     if (len > GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                         P_counter += tc::approach::GroupTC_HASH::linear_search_group(temp_adj, partition, len, bin, BIN_START);
//                     }
//                 }
//                 workid += GroupTC_HASH_WARP_SIZE;
//             }
//             __syncthreads();
//         }   
//     }
//     // 处理低度数顶点结束
//     atomicAdd(&G_counter, P_counter);
//     __syncthreads();
//     if (threadIdx.x == 0) {
//         atomicAdd(&GLOBAL_COUNT[0], G_counter);
//     }
// }

// void tc::approach::GroupTC_HASH::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
//     std::string file = gpu_graph.input_dir;
//     int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
//     spdlog::info("Run algorithm {}", key_space);
//     spdlog::info("Dataset {}", file);
//     spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
//     int device = config.GetInteger(key_space, "device", 1);
//     HRR(hipSetDevice(device));

//     int grid_size = 1024;
//     int block_size = 1024;
//     int chunk_size = 1;

//     uint vertex_count = gpu_graph.vertex_count;
//     uint edge_count = gpu_graph.edge_count;
//     index_t *d_beg_pos = gpu_graph.beg_pos;
//     vertex_t *d_src_list = gpu_graph.src_list;
//     vertex_t *d_adj_list = gpu_graph.adj_list;

//     index_t *h_beg_pos = (index_t *)malloc(sizeof(index_t) * (vertex_count + 1));
//     HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

//     int warpfirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_CTA, h_beg_pos) + 1;
//     int warpfirstedge = h_beg_pos[warpfirstvertex];
//     int nocomputefirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_WARP, h_beg_pos) + 1;
//     int nocomputefirstedge = h_beg_pos[nocomputefirstvertex];

//     int T_Group = 32;
//     int nowindex[3];
//     nowindex[0] = chunk_size * grid_size * block_size / T_Group;
//     nowindex[1] = chunk_size * grid_size;
//     nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

//     int *BIN_MEM;
//     unsigned long long *GLOBAL_COUNT;
//     int *G_INDEX;

//     HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * grid_size * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE));
//     HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
//     HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));

//     HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

//     unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);

//     double total_kernel_use = 0;
//     double startKernel, ee = 0;
//     int block_kernel_grid_size = min(max(warpfirstvertex, 1), grid_size);
//     int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (GroupTC_HASH_EDGE_CHUNK * 10), grid_size);
//     int kernel_grid_size = max(max(block_kernel_grid_size, group_kernel_grid_size), 320);

//     spdlog::info("kernel_grid_size {:d}", kernel_grid_size);

//     for (int i = 0; i < iteration_count; i++) {
//         HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
//         startKernel = wtime();
//         hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
//         tc::approach::GroupTC_HASH::grouptc_hash<<<kernel_grid_size, GroupTC_HASH_BLOCK_SIZE>>>(
//             d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, chunk_size, warpfirstvertex,
//             warpfirstedge, nocomputefirstvertex, nocomputefirstedge);
//         HRR(hipDeviceSynchronize());

//         ee = wtime();
//         total_kernel_use += ee - startKernel;
//     }

//     HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));

//     // algorithm, dataset, iteration_count, avg compute time/s,
//     spdlog::get("GroupTC-HASH_file_logger")
//         ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC-Hash", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

//     // spdlog::get("GroupTC-HASH_file_logger")
//     //     ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}\t{5}", "GroupTC-Hash", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count, edge_count);

//     // // 计算度数小于100的顶点数
//     // spdlog::get("GroupTC-HASH_file_logger")
//     //     ->info("{0}\t{1}\t{2}\t{3}\t{4}\t{5}", "GroupTC-Hash-count", gpu_graph.input_dir, warpfirstvertex, warpfirstedge, nocomputefirstvertex, nocomputefirstedge);

//     spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
//     spdlog::info("Triangle count {:d}", counter[0]);

//     free(counter);
//     free(h_beg_pos);
//     HRR(hipFree(BIN_MEM));
//     HRR(hipFree(GLOBAL_COUNT));
//     HRR(hipFree(G_INDEX));
// }

// void tc::approach::GroupTC_HASH::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
//     bool run = config.GetBoolean("comm", "GroupTC-HASH", false);
//     if (run) {
//         size_t free_byte, total_byte, available_byte;
//         HRR(hipMemGetInfo(&free_byte, &total_byte));
//         available_byte = total_byte - free_byte;
//         spdlog::debug("GroupTC_HASH before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

//         tc::approach::GroupTC_HASH::gpu_run(config, gpu_graph);

//         HRR(hipMemGetInfo(&free_byte, &total_byte));
//         spdlog::debug("GroupTC_HASH after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
//         if (available_byte != total_byte - free_byte) {
//             spdlog::warn("There is GPU memory that is not freed after GroupTC_HASH runs.");
//         }
//     }
// }



// // 就是trust和groupTc-hash都按顶点度数把数据集分成了两部分，你可以测一测，他们分别在这两部分上的运行时间
// #include <hip/hip_runtime_api.h>
// #include <thrust/device_ptr.h>
// #include <thrust/functional.h>
// #include <thrust/reduce.h>
// #include <thrust/sort.h>

// #include <string>

// #include "approach/GroupTC-HASH/tc.h"
// #include "comm/comm.h"
// #include "comm/constant_comm.h"
// #include "comm/cuda_comm.h"
// #include "spdlog/spdlog.h"

// __device__ int tc::approach::GroupTC_HASH::linear_search_block(int neighbor, int *partition, int len, int bin, int BIN_START) {
//     for (;;) {
//         len -= GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
//         int i = bin + BIN_START;
//         int step = 0;
//         while (step < len) {
//             if (partition[i] == neighbor) {
//                 return 1;
//             }
//             i += GroupTC_HASH_block_bucketnum;
//             step += 1;
//         }
//         if (len + GroupTC_HASH_shared_BLOCK_BUCKET_SIZE < 99) break;
//         bin++;
//     }
//     return 0;
// }

// __device__ int tc::approach::GroupTC_HASH::linear_search_group(int neighbor, int *partition, int len, int bin, int BIN_START) {
//     len -= GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//     int i = bin + BIN_START;
//     int step = 0;
//     while (step < len) {
//         if (partition[i] == neighbor) {
//             return 1;
//         }
//         i += GroupTC_HASH_group_bucketnum;
//         step += 1;
//     }

//     return 0;
// }

// int tc::approach::GroupTC_HASH::my_binary_search(int len, int val, index_t *beg) {
//     int l = 0, r = len;
//     while (l < r - 1) {
//         int mid = (l + r) / 2;
//         if (beg[mid + 1] - beg[mid] > val)
//             l = mid;
//         else
//             r = mid;
//     }
//     if (beg[l + 1] - beg[l] <= val) return -1;
//     return l;
// }

// __global__ void tc::approach::GroupTC_HASH::grouptc_hash(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count, uint vertex_count,
//                                                          int *partition, unsigned long long *GLOBAL_COUNT, int T_Group, int *G_INDEX, int CHUNK_SIZE,
//                                                          int warpfirstvertex, int warpfirstedge, int nocomputefirstvertex, int nocomputefirstedge) {
//     // hashTable bucket 计数器
//     __shared__ int bin_count[GroupTC_HASH_block_bucketnum];
//     // 共享内存中的 hashTable
//     __shared__ int shared_partition[GroupTC_HASH_block_bucketnum * GroupTC_HASH_shared_BLOCK_BUCKET_SIZE];
//     unsigned long long __shared__ G_counter;

//     if (threadIdx.x == 0) {
//         G_counter = 0;
//     }

//     int BIN_START = blockIdx.x * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE;
//     unsigned long long P_counter = 0;

//     // CTA for large degree vertex
//     int vertex = blockIdx.x * CHUNK_SIZE;
//     int vertex_end = vertex + CHUNK_SIZE;
//     __shared__ int ver;

//     // while (vertex < warpfirstvertex)
//     // 高度数顶点
//     // 使用CTA（线程块）进行并行处理
//     // 包括初始化哈希表、计数桶、插入邻居等操作
//     while (0)
//     {
//         // if (degree<=USE_CTA) break;
//         int group_start = beg_pos[vertex];
//         int end = beg_pos[vertex + 1];
//         int now = threadIdx.x + group_start;
//         // int MODULO = GroupTC_HASH_block_bucketnum - 1;
//         // clean bin_count
//         // 初始化 hashTable bucket 计数器
//         for (int i = threadIdx.x; i < GroupTC_HASH_block_bucketnum; i += GroupTC_HASH_BLOCK_SIZE) bin_count[i] = 0;
//         __syncthreads();

//         // count hash bin
//         // 生成 hashTable
//         while (now < end) {
//             int temp = adj_list[now];
//             int bin = temp & GroupTC_HASH_BLOCK_MODULO;
//             int index;
//             index = atomicAdd(&bin_count[bin], 1);
//             if (index < GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                 shared_partition[index * GroupTC_HASH_block_bucketnum + bin] = temp;
//             } else if (index < GroupTC_HASH_BLOCK_BUCKET_SIZE) {
//                 index = index - GroupTC_HASH_shared_BLOCK_BUCKET_SIZE;
//                 partition[index * GroupTC_HASH_block_bucketnum + bin + BIN_START] = temp;
//             }
//             now += blockDim.x;
//         }
//         __syncthreads();

//         // list intersection
//         now = beg_pos[vertex];
//         end = beg_pos[vertex + 1];
//         int superwarp_ID = threadIdx.x / 64;
//         int superwarp_TID = threadIdx.x % 64;
//         int workid = superwarp_TID;
//         now = now + superwarp_ID;
//         // 获取二跳邻居节点
//         int neighbor = adj_list[now];
//         int neighbor_start = beg_pos[neighbor];
//         int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
//         while (now < end) {
//             // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
//             while (now < end && workid >= neighbor_degree) {
//                 now += 16;
//                 workid -= neighbor_degree;
//                 neighbor = adj_list[now];
//                 neighbor_start = beg_pos[neighbor];
//                 neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
//             }
//             if (now < end) {
//                 int temp_adj = adj_list[neighbor_start + workid];
//                 int bin = temp_adj & GroupTC_HASH_BLOCK_MODULO;

//                 int len = bin_count[bin];

//                 P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 0] == temp_adj : 0;
//                 P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 1] == temp_adj : 0;
//                 P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 2] == temp_adj : 0;
//                 P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 3] == temp_adj : 0;
//                 P_counter += len > 4 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 4] == temp_adj : 0;
//                 P_counter += len > 5 ? shared_partition[bin + GroupTC_HASH_block_bucketnum * 5] == temp_adj : 0;

//                 if (len > GroupTC_HASH_shared_BLOCK_BUCKET_SIZE) {
//                     P_counter += tc::approach::GroupTC_HASH::linear_search_block(temp_adj, partition, len, bin, BIN_START);
//                 }
//             }
//             // __syncthreads();
//             workid += 64;
//         }

//         __syncthreads();
//         // if (vertex>1) break;
//         vertex++;
//         if (vertex == vertex_end) {
//             if (threadIdx.x == 0) {
//                 ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
//             }
//             __syncthreads();
//             vertex = ver;
//             vertex_end = vertex + CHUNK_SIZE;
//         }
//     }

//     // 处理低频顶点（低度数顶点）
//     // EDGE CHUNK for small degree vertex
//     __shared__ int group_start;
//     __shared__ int group_size;

//     int *shared_src = shared_partition + GroupTC_HASH_group_bucketnum * GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//     int *shared_adj_start = shared_src + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;
//     int *shared_adj_degree = shared_adj_start + GroupTC_HASH_shared_CHUNK_CACHE_SIZE;

//     for (int group_offset = warpfirstedge + blockIdx.x * GroupTC_HASH_EDGE_CHUNK; group_offset < nocomputefirstedge;
//          group_offset += gridDim.x * GroupTC_HASH_EDGE_CHUNK) {
//         // compute group start and end
//         if (threadIdx.x == 0) {
//             int src = src_list[group_offset];
//             int src_start = beg_pos[src];
//             int src_end = beg_pos[src + 1];
//             group_start = ((src_start == group_offset) ? src_start : src_end);

//             src = src_list[min(group_offset + GroupTC_HASH_EDGE_CHUNK, nocomputefirstedge) - 1];
//             group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
//         }

//         // cache start
//         for (int i = threadIdx.x; i < GroupTC_HASH_group_bucketnum; i += blockDim.x) bin_count[i] = 0;

//         __syncthreads();

//         for (int i = threadIdx.x; i < group_size; i += GroupTC_HASH_BLOCK_SIZE) {
//             int temp_src = src_list[i + group_start];
//             int temp_adj = adj_list[i + group_start];

//             shared_src[i] = temp_src;
//             shared_adj_start[i] = beg_pos[temp_adj];
//             shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

//             int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
//             int index = atomicAdd(&bin_count[bin], 1);

//             if (index < GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                 shared_partition[index * GroupTC_HASH_group_bucketnum + bin] = temp_adj;
//             } else if (index < GroupTC_HASH_GROUP_BUCKET_SIZE) {
//                 index = index - GroupTC_HASH_shared_GROUP_BUCKET_SIZE;
//                 partition[index * GroupTC_HASH_group_bucketnum + bin + BIN_START] = temp_adj;
//             }
//         }
//         __syncthreads();

//         // compute 2 hop neighbors
//         int now = threadIdx.x / GroupTC_HASH_WARP_SIZE;
//         int workid = threadIdx.x % GroupTC_HASH_WARP_SIZE;

//         while (now < group_size) {
//             int neighbor_degree = shared_adj_degree[now];
//             while (now < group_size && workid >= neighbor_degree) {
//                 now += GroupTC_HASH_BLOCK_SIZE / GroupTC_HASH_WARP_SIZE;
//                 workid -= neighbor_degree;
//                 neighbor_degree = shared_adj_degree[now];
//             }

//             if (now < group_size) {
//                 int temp_src = shared_src[now];
//                 int temp_adj = adj_list[shared_adj_start[now] + workid];
//                 int bin = (temp_src + temp_adj) & GroupTC_HASH_GROUP_MODULO;
//                 int len = bin_count[bin];

//                 P_counter += len > 0 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 0] == temp_adj : 0;
//                 P_counter += len > 1 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 1] == temp_adj : 0;
//                 P_counter += len > 2 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 2] == temp_adj : 0;
//                 P_counter += len > 3 ? shared_partition[bin + GroupTC_HASH_group_bucketnum * 3] == temp_adj : 0;

//                 if (len > GroupTC_HASH_shared_GROUP_BUCKET_SIZE) {
//                     P_counter += tc::approach::GroupTC_HASH::linear_search_group(temp_adj, partition, len, bin, BIN_START);
//                 }
//             }
//             workid += GroupTC_HASH_WARP_SIZE;
//         }
//         __syncthreads();
//     }
//     // 处理低度数顶点结束

//     atomicAdd(&G_counter, P_counter);

//     __syncthreads();
//     if (threadIdx.x == 0) {
//         atomicAdd(&GLOBAL_COUNT[0], G_counter);
//     }
// }

// void tc::approach::GroupTC_HASH::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
//     std::string file = gpu_graph.input_dir;
//     int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
//     spdlog::info("Run algorithm {}", key_space);
//     spdlog::info("Dataset {}", file);
//     spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
//     int device = config.GetInteger(key_space, "device", 1);
//     HRR(hipSetDevice(device));

//     int grid_size = 1024;
//     int block_size = 1024;
//     int chunk_size = 1;

//     uint vertex_count = gpu_graph.vertex_count;
//     uint edge_count = gpu_graph.edge_count;
//     index_t *d_beg_pos = gpu_graph.beg_pos;
//     vertex_t *d_src_list = gpu_graph.src_list;
//     vertex_t *d_adj_list = gpu_graph.adj_list;

//     index_t *h_beg_pos = (index_t *)malloc(sizeof(index_t) * (vertex_count + 1));
//     HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

//     int warpfirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_CTA, h_beg_pos) + 1;
//     int warpfirstedge = h_beg_pos[warpfirstvertex];
//     int nocomputefirstvertex = my_binary_search(vertex_count, GroupTC_HASH_USE_WARP, h_beg_pos) + 1;
//     int nocomputefirstedge = h_beg_pos[nocomputefirstvertex];

//     int T_Group = 32;
//     int nowindex[3];
//     nowindex[0] = chunk_size * grid_size * block_size / T_Group;
//     nowindex[1] = chunk_size * grid_size;
//     nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

//     int *BIN_MEM;
//     unsigned long long *GLOBAL_COUNT;
//     int *G_INDEX;

//     HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * grid_size * GroupTC_HASH_block_bucketnum * GroupTC_HASH_BLOCK_BUCKET_SIZE));
//     HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
//     HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));

//     HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

//     unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);

//     double total_kernel_use = 0;
//     double startKernel, ee = 0;
//     int block_kernel_grid_size = min(max(warpfirstvertex, 1), grid_size);
//     int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (GroupTC_HASH_EDGE_CHUNK * 10), grid_size);
//     int kernel_grid_size = max(max(block_kernel_grid_size, group_kernel_grid_size), 320);

//     spdlog::info("kernel_grid_size {:d}", kernel_grid_size);

//     for (int i = 0; i < iteration_count; i++) {
//         HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
//         startKernel = wtime();
//         hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
//         tc::approach::GroupTC_HASH::grouptc_hash<<<kernel_grid_size, GroupTC_HASH_BLOCK_SIZE>>>(
//             d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, chunk_size, warpfirstvertex,
//             warpfirstedge, nocomputefirstvertex, nocomputefirstedge);
//         HRR(hipDeviceSynchronize());

//         ee = wtime();
//         total_kernel_use += ee - startKernel;
//     }

//     HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));

//     // // algorithm, dataset, iteration_count, avg compute time/s,
//     // spdlog::get("GroupTC-HASH_file_logger")
//     //     ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC-Hash", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

//     spdlog::get("GroupTC-HASH_file_logger")
//         ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}\t{5}", "GroupTC-Hash", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count, edge_count);


//     spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
//     spdlog::info("Triangle count {:d}", counter[0]);

//     free(counter);
//     free(h_beg_pos);
//     HRR(hipFree(BIN_MEM));
//     HRR(hipFree(GLOBAL_COUNT));
//     HRR(hipFree(G_INDEX));
// }

// void tc::approach::GroupTC_HASH::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
//     bool run = config.GetBoolean("comm", "GroupTC-HASH", false);
//     if (run) {
//         size_t free_byte, total_byte, available_byte;
//         HRR(hipMemGetInfo(&free_byte, &total_byte));
//         available_byte = total_byte - free_byte;
//         spdlog::debug("GroupTC_HASH before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

//         tc::approach::GroupTC_HASH::gpu_run(config, gpu_graph);

//         HRR(hipMemGetInfo(&free_byte, &total_byte));
//         spdlog::debug("GroupTC_HASH after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
//         if (available_byte != total_byte - free_byte) {
//             spdlog::warn("There is GPU memory that is not freed after GroupTC_HASH runs.");
//         }
//     }
// }