#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "approach/TC-Check/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/sinks/basic_file_sink.h"
#include "spdlog/spdlog.h"

__global__ void tc::approach::TC_Check::calculate_triangles(uint edge_count, vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos, int* results,
                                                            int deviceCount, int deviceIdx) {
    int from = gridDim.x * blockDim.x * deviceIdx + blockDim.x * blockIdx.x + threadIdx.x;
    int step = deviceCount * gridDim.x * blockDim.x;
    unsigned long long count = 0;

    for (int i = from; i < edge_count; i += step) {
        int u = src_list[i], v = adj_list[i];

        int u_it = beg_pos[u], u_end = beg_pos[u + 1];
        int v_it = beg_pos[v], v_end = beg_pos[v + 1];

        int a = adj_list[u_it], b = adj_list[v_it];
        while (u_it < u_end && v_it < v_end) {
            int d = a - b;
            if (d <= 0) a = adj_list[++u_it];
            if (d >= 0) b = adj_list[++v_it];
            if (d == 0) {
                atomicAdd(results + u, 1);
                if (u == 253264) {
                    // printf("253264 %d\n", a);
                }
                ++count;
            }
        }
    }

    // results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void tc::approach::TC_Check::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = config.Get("comm", "dataset", "UNKNOWN");
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    int grid_size = NumberOfMPs() * 8;
    int block_size = 64;

    double t_start, total_kernel_use = 0;

    int* d_results;
    HRR(hipMalloc(&d_results, vertex_count * sizeof(int)));

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(d_results, 0, vertex_count * sizeof(int)));
        t_start = wtime();
        tc::approach::TC_Check::calculate_triangles<<<grid_size, block_size>>>(edge_count, d_src, d_adj, d_beg_pos, d_results);
        HRR(hipDeviceSynchronize());

        double ee = wtime();
        total_kernel_use += ee - t_start;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - t_start > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    int* h_count = (int*)malloc(sizeof(int) * vertex_count);

    HRR(hipMemcpy(h_count, d_results, sizeof(int) * vertex_count, hipMemcpyDeviceToHost));

    std::string output_file = config.Get(key_space, "dataset", "./logs/tc_check.txt");
    auto async_file = spdlog::basic_logger_mt("tc_check_logger", output_file);

    for (int i = 1; i < vertex_count; ++i) {
        if (h_count[i] != 0) {
            async_file->info("{} {}", i, h_count[i]);
        }
    }

    free(h_count);
    HRR(hipFree(d_results));
}

void tc::approach::TC_Check::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "TC_Check", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("TC_Check before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::TC_Check::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("TC_Check after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after TC_Check runs.");
        }
    }
}
