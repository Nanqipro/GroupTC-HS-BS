#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/GroupTC-OPT/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

__device__ int tc::approach::GroupTC_OPT::bin_search(vertex_t* arr, int len, int val) {
    uint32_t Y;
    int32_t bot = 0;
    int32_t top = len - 1;
    int32_t r;
    while (top >= bot) {
        r = (top + bot) / 2;
        Y = arr[r];

        if (val == Y) {
            return 1;
        }

        if (val < Y) {
            top = r - 1;
        } else {
            bot = r + 1;
        }
    }
    return 0;
}

__device__ int tc::approach::GroupTC_OPT::bin_search_less_branch(vertex_t* arr, int len, int val) {
    int ret = 0;
    int halfsize;
    int candidate;
    int temp = len;
    while (temp > 1) {
        halfsize = temp / 2;
        candidate = arr[ret + halfsize];
        ret += (candidate < val) ? halfsize : 0;
        temp -= halfsize;
    }
    ret += (arr[ret] < val);
    return ret < len && arr[ret] == val;
}

__device__ int tc::approach::GroupTC_OPT::bin_search_with_offset_and_less_branch(vertex_t* arr, int len, int val, int& offset) {
    int ret = 0;
    int halfsize;
    int candidate;
    int temp = len;
    while (temp > 1) {
        halfsize = temp / 2;
        candidate = arr[ret + halfsize];
        ret += (candidate < val) ? halfsize : 0;
        temp -= halfsize;
    }
    ret += (arr[ret] < val);
    offset = ret;
    return ret < len && arr[ret] == val;
}

template <const int GroupTC_OPT_SUBWARP_SIZE, const int GroupTC_OPT_WARP_STEP>
__global__ void tc::approach::GroupTC_OPT::grouptc_with_reduce(vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos, uint edge_count,
                                                               uint vertex_count, unsigned long long* GLOBAL_COUNT) {
    // 共享内存中的 hashTable
    __shared__ int sh_tb_start[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ int sh_tb_len[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ int sh_ele_start[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ int sh_ele_len[GroupTC_OPT_BLOCK_BUCKETNUM];

    unsigned long long P_counter = 0;

    int bid = blockIdx.x;
    int tid = threadIdx.x;

    for (int i = bid * GroupTC_OPT_BLOCK_BUCKETNUM; i < edge_count; i += gridDim.x * GroupTC_OPT_BLOCK_BUCKETNUM) {
        if (i + tid < edge_count) {
            int src = src_list[i + tid];
            int dst = adj_list[i + tid];
            int temp;

            int tb_start, tb_len, ele_start, ele_len;
            tb_start = i + tid + 1;
            // tb_start = beg_pos[src];
            tb_len = beg_pos[src + 1] - tb_start;
            ele_start = beg_pos[dst];
            ele_len = beg_pos[dst + 1] - ele_start;

            if (tb_len * 2 < ele_len) {
                temp = tb_start;
                tb_start = ele_start;
                ele_start = temp;

                temp = tb_len;
                tb_len = ele_len;
                ele_len = temp;
            }

            sh_tb_start[tid] = tb_start;
            sh_tb_len[tid] = tb_len;
            sh_ele_start[tid] = ele_start;
            sh_ele_len[tid] = ele_len;
        }

        __syncthreads();

        int now = tid / GroupTC_OPT_SUBWARP_SIZE;
        int end = min(edge_count - i, GroupTC_OPT_BLOCK_BUCKETNUM);
        int workid = tid % GroupTC_OPT_SUBWARP_SIZE;
        int offset = 0;
        int last_now = -1;

        // 获取二跳邻居节点
        int neighbor_degree = sh_ele_len[now];
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += GroupTC_OPT_WARP_STEP;
                if (now < end) {
                    workid -= neighbor_degree;
                    neighbor_degree = sh_ele_len[now];
                }
            }

            if (now < end) {
                offset = last_now == now ? offset : 0;
                P_counter += tc::approach::GroupTC_OPT::bin_search_with_offset_and_less_branch(
                    adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
                last_now = now;
            }
            workid += GroupTC_OPT_SUBWARP_SIZE;
        }
        __syncthreads();
    }

    GLOBAL_COUNT[bid * GroupTC_OPT_BLOCK_BUCKETNUM + tid] = P_counter;
}

template <const int GroupTC_OPT_SUBWARP_SIZE, const int GroupTC_OPT_WARP_STEP>
__global__ void tc::approach::GroupTC_OPT::grouptc_with_atomic(vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos, uint edge_count,
                                                               uint vertex_count, unsigned long long* GLOBAL_COUNT) {
    // 共享内存中的 hashTable
    __shared__ int sh_tb_start[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ int sh_tb_len[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ int sh_ele_start[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ int sh_ele_len[GroupTC_OPT_BLOCK_BUCKETNUM];
    __shared__ unsigned long long B_counter;

    unsigned long long P_counter = 0;

    int bid = blockIdx.x;
    int tid = threadIdx.x;
    if (tid == 0) {
        B_counter = 0;
    }
    __syncthreads();

    for (int i = bid * GroupTC_OPT_BLOCK_BUCKETNUM; i < edge_count; i += gridDim.x * GroupTC_OPT_BLOCK_BUCKETNUM) {
        if (i + tid < edge_count) {
            int src = src_list[i + tid];
            int dst = adj_list[i + tid];
            int temp;

            int tb_start, tb_len, ele_start, ele_len;
            tb_start = i + tid + 1;
            // tb_start = beg_pos[src];
            tb_len = beg_pos[src + 1] - tb_start;
            ele_start = beg_pos[dst];
            ele_len = beg_pos[dst + 1] - ele_start;

            if (tb_len * 2 < ele_len) {
                temp = tb_start;
                tb_start = ele_start;
                ele_start = temp;

                temp = tb_len;
                tb_len = ele_len;
                ele_len = temp;
            }

            sh_tb_start[tid] = tb_start;
            sh_tb_len[tid] = tb_len;
            sh_ele_start[tid] = ele_start;
            sh_ele_len[tid] = ele_len;
        }

        __syncthreads();

        int now = tid / GroupTC_OPT_SUBWARP_SIZE;
        int end = min(edge_count - i, GroupTC_OPT_BLOCK_BUCKETNUM);
        int workid = tid % GroupTC_OPT_SUBWARP_SIZE;
        int offset = 0;
        int last_now = -1;

        // 获取二跳邻居节点
        int neighbor_degree = sh_ele_len[now];
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += GroupTC_OPT_WARP_STEP;
                if (now < end) {
                    workid -= neighbor_degree;
                    neighbor_degree = sh_ele_len[now];
                }
            }

            if (now < end) {
                offset = last_now == now ? offset : 0;
                P_counter += tc::approach::GroupTC_OPT::bin_search_with_offset_and_less_branch(
                    adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
                last_now = now;
            }
            workid += GroupTC_OPT_SUBWARP_SIZE;
        }
        __syncthreads();
    }

    P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 16);
    P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 8);
    P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 4);
    P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 2);
    P_counter += __shfl_down_sync(0xFFFFFFFF, P_counter, 1);

    if (tid % 32 == 0) {
        atomicAdd(&B_counter, P_counter);
    }
    __syncthreads();
    if (tid == 0) {
        atomicAdd(GLOBAL_COUNT, B_counter);
    }
}

void tc::approach::GroupTC_OPT::gpu_run_with_reduce(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    int grid_size = edge_count / GroupTC_OPT_BLOCK_BUCKETNUM / 20;

    double t_start, total_kernel_use = 0;
    uint64_t count;

    unsigned long long* d_results;
    HRR(hipMalloc(&d_results, grid_size * GroupTC_OPT_BLOCK_BUCKETNUM * sizeof(unsigned long long)));

    int avg_degree = edge_count / vertex_count;

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(d_results, 0, grid_size * GroupTC_OPT_BLOCK_BUCKETNUM * sizeof(unsigned long long)));

        t_start = wtime();

        if (avg_degree > 32) {
            tc::approach::GroupTC_OPT::grouptc_with_reduce<64, GroupTC_OPT_BLOCK_BUCKETNUM / 64>
                <<<grid_size, GroupTC_OPT_BLOCK_BUCKETNUM>>>(d_src, d_adj, d_beg_pos, edge_count, vertex_count, d_results);
        } else {
            tc::approach::GroupTC_OPT::grouptc_with_reduce<32, GroupTC_OPT_BLOCK_BUCKETNUM / 32>
                <<<grid_size, GroupTC_OPT_BLOCK_BUCKETNUM>>>(d_src, d_adj, d_beg_pos, edge_count, vertex_count, d_results);
        }

        HRR(hipDeviceSynchronize());
        thrust::device_ptr<unsigned long long> ptr(d_results);
        count = thrust::reduce(ptr, ptr + (grid_size * GroupTC_OPT_BLOCK_BUCKETNUM));

        double ee = wtime();
        total_kernel_use += ee - t_start;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - t_start > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, iteration_count, avg compute time/s,
    spdlog::get("GroupTC-OPT_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC-OPT", gpu_graph.input_dir, count, iteration_count, total_kernel_use / iteration_count);

    spdlog::info("iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", count);

    HRR(hipFree(d_results));
}

void tc::approach::GroupTC_OPT::gpu_run_with_atomic(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    int grid_size = NumberOfMPs() * 8;

    double t_start, total_kernel_use = 0;
    unsigned long long* counter = (unsigned long long*)malloc(sizeof(unsigned long long));
    unsigned long long* results;

    HRR(hipMalloc(&results, sizeof(unsigned long long)));

    int avg_degree = edge_count / vertex_count;

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(results, 0, sizeof(unsigned long long)));
        hipProfilerStart();
        t_start = wtime();
        if (avg_degree > 32) {
            tc::approach::GroupTC_OPT::grouptc_with_atomic<64, GroupTC_OPT_BLOCK_BUCKETNUM / 64>
                <<<grid_size, GroupTC_OPT_BLOCK_BUCKETNUM>>>(d_src, d_adj, d_beg_pos, edge_count, vertex_count, results);
        } else {
            tc::approach::GroupTC_OPT::grouptc_with_atomic<32, GroupTC_OPT_BLOCK_BUCKETNUM / 32>
                <<<grid_size, GroupTC_OPT_BLOCK_BUCKETNUM>>>(d_src, d_adj, d_beg_pos, edge_count, vertex_count, results);
        }
        HRR(hipDeviceSynchronize());
        total_kernel_use += wtime() - t_start;
        hipProfilerStop();
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (total_kernel_use > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    HRR(hipMemcpy(counter, results, sizeof(unsigned long long), hipMemcpyDeviceToHost));

    // algorithm, dataset, iteration_count, avg compute time/s,
    spdlog::get("GroupTC-OPT_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC-OPT", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

    spdlog::info("iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", counter[0]);

    free(counter);
    HRR(hipFree(results));
}

void tc::approach::GroupTC_OPT::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "GroupTC-OPT", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("GroupTC_OPT before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        vertex_t edge_count = gpu_graph.edge_count;
        if (edge_count > 1e8) {
            tc::approach::GroupTC_OPT::gpu_run_with_reduce(config, gpu_graph);
        } else {
            tc::approach::GroupTC_OPT::gpu_run_with_atomic(config, gpu_graph);
        }

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("GroupTC_OPT after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after GroupTC_OPT runs.");
        }
    }
}
