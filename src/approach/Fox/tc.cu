#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <omp.h>

#include "approach/Fox/tc.h"
#include "comm/comm.h"
#include "comm/config_comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

__global__ void tc::approach::Fox::getEdgeWorkLoad(uint edge_count, uint16_t *d_edgeWorkLoad, vertex_t *d_src_list, vertex_t *d_adj_list,
                                                   uint *c_adjLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= edge_count) {
        return;
    }

    int srcLen = c_adjLen[d_src_list[idx]];
    int dstLen = c_adjLen[d_adj_list[idx]];
    int large = (srcLen > dstLen) ? srcLen : dstLen;
    uint16_t bWork = (srcLen + dstLen - large) * log2((double)large + 2);
    d_edgeWorkLoad[idx] = bWork;
    return;
}

uint tc::approach::Fox::binarySearchValue(uint16_t *array, int value, uint arrayLength, int direction) {
    uint s = 0, e = arrayLength - 1;
    uint rightPos;
    bool find = false;
    uint mid = (s + e) / 2;
    while (s <= e) {
        if (array[mid] == value) {
            rightPos = mid;
            find = true;
            break;
        } else if (array[mid] < value) {
            s = mid + 1;
        } else {
            e = mid - 1;
        }
        mid = (s + e) / 2;
    }
    if (!find) {
        return s;
    }
    long int tmpValue = rightPos + direction;
    while (tmpValue >= 0 && tmpValue < arrayLength && (int)array[tmpValue] == value) {
        rightPos += direction;
        tmpValue = rightPos + direction;
    }
    return rightPos;
}

__global__ void tc::approach::Fox::binSearchKernel(index_t *c_offset, vertex_t *d_src_reorder, vertex_t *d_adj_reorder, vertex_t *c_adj_list,
                                                   uint *c_adjLen, uint edge_count, uint c_edge_start_pos, uint c_edge_end_pos, int c_threadsPerEdge,
                                                   long *c_sums) {
    long idx = blockDim.x * blockIdx.x + threadIdx.x;
    long sum = 0;
    __shared__ long sh_sum[32];
    int edgeID = idx / c_threadsPerEdge + c_edge_start_pos;
    int inEdgeID = idx % c_threadsPerEdge;

    while (edgeID < c_edge_end_pos) {
        int src = d_src_reorder[edgeID];
        int dst = d_adj_reorder[edgeID];
        if (c_adjLen[src] < c_adjLen[dst]) {
            int tmp = src;
            src = dst;
            dst = tmp;
        }
        int srcAdjListLen = c_adjLen[src];
        int dstAdjListLen = c_adjLen[dst];
        vertex_t *srcAdjList = c_adj_list + c_offset[src];
        vertex_t *dstAdjList = c_adj_list + c_offset[dst];

        for (int i = 0; i < dstAdjListLen; i += c_threadsPerEdge) {
            int dstListIdx = i + inEdgeID;
            if (dstListIdx >= dstAdjListLen) {
                continue;
            }

            int targetValue = dstAdjList[dstListIdx];

            int s = 0, e = srcAdjListLen - 1;
            int mid = (s + e) / 2;
            while (s <= e) {
                if (srcAdjList[mid] == targetValue) {
                    sum++;
                    break;
                } else if (srcAdjList[mid] < targetValue) {
                    s = mid + 1;
                } else {
                    e = mid - 1;
                }
                mid = (s + e) / 2;
            }
        }
        idx += blockDim.x * gridDim.x;
        edgeID = idx / c_threadsPerEdge + c_edge_start_pos;
        inEdgeID = idx % c_threadsPerEdge;
    }
    int tIdx = threadIdx.x;

    sum += __shfl_down_sync(0xFFFFFFFF, sum, 16);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 8);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 4);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 2);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 1);

    if (tIdx % 32 == 0) sh_sum[tIdx / 32] = sum;
    __syncthreads();
    if (tIdx == 0) {
        sum = 0;
        for (int i = 0; i < 32; i++) {
            sum += sh_sum[i];
        }
        c_sums[blockIdx.x] = sum;
    }
    return;
}

void tc::approach::Fox::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    long int triangleCount = 0;
    uint *d_adjLength;
    index_t *d_edgeOffset = gpu_graph.beg_pos;
    vertex_t *d_src_list = gpu_graph.src_list;
    vertex_t *d_adj_list = gpu_graph.adj_list;
    uint nodeNum = gpu_graph.vertex_count;
    uint edgeNum = gpu_graph.edge_count;

    HRR(hipMalloc(&d_adjLength, sizeof(uint) * (nodeNum + 1)));

    int block_size = 1024;
    int vertex_grid_size = (nodeNum - 1) / block_size + 1;
    int edge_grid_size = (edgeNum - 1) / block_size + 1;
    cuda_graph_comm::cal_out_degree_by_offset<<<vertex_grid_size, block_size>>>(edgeNum, nodeNum, d_adjLength, d_edgeOffset);
    HRR(hipDeviceSynchronize());

    cuda_graph_comm::set_value_by_index(d_edgeOffset, nodeNum + 1, (index_t)edgeNum + 1);
    cuda_graph_comm::set_value_by_index(d_adjLength, nodeNum, (uint)1024);
    cuda_graph_comm::set_value_by_index(d_adj_list, edgeNum, nodeNum);

    uint16_t *edgeWorkLoad;
    vertex_t *d_src_reorder;
    vertex_t *d_adj_reorder;

    size_t edge_workload_size = (size_t)sizeof(uint16_t) * edgeNum;
    size_t edge_src_size = (size_t)sizeof(vertex_t) * edgeNum;

    HRR(hipMalloc(&d_src_reorder, edge_src_size));
    HRR(hipMalloc(&d_adj_reorder, edge_src_size));

    double t_start = wtime();

    int iterations = config_comm::cPreprocessingIterations;
    for (int i = 0; i < iterations; i++) {
        uint16_t *d_edgeWorkLoad;
        HRR(hipMalloc(&d_edgeWorkLoad, edge_workload_size));
        tc::approach::Fox::getEdgeWorkLoad<<<edge_grid_size, block_size>>>(edgeNum, d_edgeWorkLoad, d_src_list, d_adj_list, d_adjLength);
        HRR(hipDeviceSynchronize());

        size_t free_byte, total_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Fox after get edge workload, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Fox after get reorder arr, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        HRR(hipMemcpy(d_src_reorder, d_src_list, edge_src_size, hipMemcpyDeviceToDevice));
        HRR(hipMemcpy(d_adj_reorder, d_adj_list, edge_src_size, hipMemcpyDeviceToDevice));

        vertex_t *h_src_list;
        vertex_t *h_adj_list;

        // The GPU space required for sorting is insufficient, so src_list and adj_list need to be transferred to CPU space first.
        if (edgeNum > constant_comm::kFoxMaxEdgeCount) {
            spdlog::info("Fox's sorting requires more GPU space, so src_list and adj_list are transferred to CPU space.");

            h_src_list = (vertex_t *)malloc(edge_src_size);
            h_adj_list = (vertex_t *)malloc(edge_src_size);
            HRR(hipMemcpy(h_src_list, d_src_list, edge_src_size, hipMemcpyDeviceToHost));
            HRR(hipMemcpy(h_adj_list, d_adj_list, edge_src_size, hipMemcpyDeviceToHost));
            HRR(hipFree(d_src_list));
            HRR(hipFree(d_adj_list));
        }

        uint16_t *d_edgeWorkLoad_copy;
        HRR(hipMalloc(&d_edgeWorkLoad_copy, edge_workload_size));
        HRR(hipMemcpy(d_edgeWorkLoad_copy, d_edgeWorkLoad, edge_workload_size, hipMemcpyDeviceToDevice));

        spdlog::debug("Fox after get edge workload copy, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        thrust::sort_by_key((thrust::device_ptr<uint16_t>)d_edgeWorkLoad_copy, (thrust::device_ptr<uint16_t>)(d_edgeWorkLoad_copy + edgeNum),
                            (thrust::device_ptr<vertex_t>)d_src_reorder);
        HRR(hipFree(d_edgeWorkLoad_copy));

        thrust::sort_by_key((thrust::device_ptr<uint16_t>)d_edgeWorkLoad, (thrust::device_ptr<uint16_t>)(d_edgeWorkLoad + edgeNum),
                            (thrust::device_ptr<vertex_t>)d_adj_reorder);

        spdlog::debug("Fox sort successed ...");

        // After sorting, src_list and adj_list are transferred back to GPU space.
        if (edgeNum > constant_comm::kFoxMaxEdgeCount) {
            spdlog::info("Fox's sorting is completed, src_list and adj_list are transferred back to the GPU space.");

            HRR(hipMalloc(&d_src_list, edge_src_size));
            HRR(hipMalloc(&d_adj_list, edge_src_size));
            HRR(hipMemcpy(d_src_list, h_src_list, edge_src_size, hipMemcpyHostToDevice));
            HRR(hipMemcpy(d_adj_list, h_adj_list, edge_src_size, hipMemcpyHostToDevice));
            gpu_graph.src_list = d_src_list;
            gpu_graph.adj_list = d_adj_list;
            free(h_src_list);
            free(h_adj_list);
        }

        edgeWorkLoad = (uint16_t *)malloc(edge_workload_size);
        HRR(hipMemcpy(edgeWorkLoad, d_edgeWorkLoad, edge_workload_size, hipMemcpyDeviceToHost));
        HRR(hipFree(d_edgeWorkLoad));
    }

    double t_end = wtime();

    // algorithm, dataset, iterations, avg compute time/s,
    auto preprocessing_logger = spdlog::get("Fox_preprocessing_file_logger");
    if (preprocessing_logger) {
        preprocessing_logger->info("{0}\t{1}\t{2}\t{3:.6f}", "Fox", gpu_graph.input_dir, iterations, (t_end - t_start) / iterations);
    } else {
        spdlog::warn("Logger 'Fox_preprocessing_file_logger' is not initialized.");
    }

    double total_kernel_use = 0;
    double startKernel, ee;
    for (int iter = 0; iter < iteration_count; iter++) {
        triangleCount = 0;
        int binMaxWork = edgeWorkLoad[edgeNum - 1];
        int workPerThreadB = 8;
        int maxThreadsPerEdge = 32;
        double maxPowerOf8 = log(binMaxWork) / log(workPerThreadB);
        int curBinB = maxThreadsPerEdge;
        uint *binStartPos = new uint[curBinB + 2];
        binStartPos[1] = 0;
        for (int i = 2; i <= curBinB; i *= 2) {
            double index = (double)(maxPowerOf8 - 1) * (i - 1) / (double)curBinB + 1;
            int partitionPoint = powl(workPerThreadB, index);
            binStartPos[i] = tc::approach::Fox::binarySearchValue(edgeWorkLoad, partitionPoint, edgeNum, -1);
            binStartPos[i / 2 + 1] = binStartPos[i];
        }

        binStartPos[curBinB + 1] = edgeNum;

        for (int i = 1; i <= curBinB; i *= 2) {
            // for each bin
            if (binStartPos[i + 1] - binStartPos[i] == 0) {
                continue;
            }

            uint c_edge_start_pos = binStartPos[i];
            uint c_edge_end_pos = binStartPos[i + 1];

            uint curGridSize = 20000;
            int maxBlockNumTC = 20000;
            int maxBlockSizeTC = 1024;
            int curThreadsPerEdge = i;
            long *d_sum;
            HRR(hipMalloc(&d_sum, sizeof(long) * maxBlockNumTC));
            HRR(hipMemset(d_sum, 0, sizeof(long) * maxBlockNumTC));
            startKernel = wtime();
            tc::approach::Fox::binSearchKernel<<<curGridSize, maxBlockSizeTC>>>(d_edgeOffset, d_src_reorder, d_adj_reorder, d_adj_list, d_adjLength,
                                                                                edgeNum, c_edge_start_pos, c_edge_end_pos, curThreadsPerEdge, d_sum);
            HRR(hipDeviceSynchronize());
            triangleCount += thrust::reduce((thrust::device_ptr<long>)d_sum, (thrust::device_ptr<long>)(d_sum + curGridSize));
            ee = wtime();
            total_kernel_use += ee - startKernel;
            HRR(hipFree(d_sum));
        }

        if (iter == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (total_kernel_use > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, triangle_count, iteration_count, avg kernel time/s
    auto logger = spdlog::get("Fox_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "Fox", gpu_graph.input_dir, triangleCount, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'Fox_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", triangleCount);

    free(edgeWorkLoad);
    HRR(hipFree(d_adjLength));
    HRR(hipFree(d_src_reorder));
    HRR(hipFree(d_adj_reorder));
}

void tc::approach::Fox::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "Fox", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("Fox before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::Fox::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Fox after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after Fox runs.");
        }
    }
}
