#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/Hu/tc.h"
#include "comm/comm.h"
#include "comm/config_comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

__global__ void tc::approach::Hu::getNodesWorkLoad(int startPos, int threadNum, long int *d_nodeWorkLoad, index_t *c_offset, vertex_t *c_row,
                                                   uint *c_adjLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= threadNum) return;
    int src = startPos + idx;
    vertex_t *srcList = c_row + c_offset[src];
    uint srcListLen = c_adjLen[src];
    long int totalLength = 0;
    for (int i = 0; i < srcListLen; i++) {
        totalLength += c_adjLen[srcList[i]];
    }
    d_nodeWorkLoad[idx] = totalLength * (unsigned)log2((double)srcListLen);
    return;
}

uint tc::approach::Hu::binarySearchValue(long int *array, long int value, uint arrayLength, int direction) {
    long int s = 0, e = arrayLength - 1;
    long int rightPos;
    bool find = false;
    long int mid = (s + e) / 2;
    while (s <= e) {
        if (array[mid] == value) {
            rightPos = mid;
            find = true;
            break;
        } else if (array[mid] < value) {
            s = mid + 1;
        } else {
            if (e == 0) break;
            e = mid - 1;
        }
        mid = (s + e) / 2;
    }
    if (!find) {
        return s;
    }
    long int tmpValue = rightPos + direction;
    while (tmpValue >= 0 && tmpValue < arrayLength && array[tmpValue] == value) {
        rightPos += direction;
        tmpValue = rightPos + direction;
    }
    return rightPos;
}

__global__ void tc::approach::Hu::triangleCountKernel(index_t *c_offset, vertex_t *c_row, uint *c_adjLen, int *c_blockStartNodeOffset,
                                                      long int *c_sum) {
    int uid = c_blockStartNodeOffset[blockIdx.x];
    int uidThre = c_blockStartNodeOffset[blockIdx.x + 1];
    if (uid == uidThre) return;
    unsigned vpos = c_offset[uid];
    int vid = c_row[vpos];
    int wpos = threadIdx.x;
    // int *srcList = c_row + c_offset[uid];
    // int *dstList = c_row + c_offset[c_row[vpos]];
    long int sum = 0;
    __shared__ int cachedMaxUid;
    __shared__ bool cacheWorked;
    __shared__ int sharedVid[HU_shareMemorySizeInBlock];
    __shared__ int lastCachedMaxUid;
    while (1) {
        __syncthreads();
        if (threadIdx.x == 0) {
            if (uid == c_blockStartNodeOffset[blockIdx.x]) {
                cachedMaxUid = uid;
            }
            lastCachedMaxUid = cachedMaxUid;
            cachedMaxUid--;
            cacheWorked = true;
            int cachedVid = 0;
            int cachedUid = 0;
            while (1) {
                cachedMaxUid++;
                cachedUid++;
                cachedVid += c_adjLen[cachedMaxUid];
                if (cachedVid >= HU_shareMemorySizeInBlock || cachedMaxUid >= uidThre) {
                    break;
                }
            }
            if (cachedUid == 1) {
                cacheWorked = false;
                cachedMaxUid++;
                cachedMaxUid = (uidThre > cachedMaxUid) ? cachedMaxUid : uidThre;
            } else {
                int len = cachedVid - c_adjLen[cachedMaxUid];
                memcpy(sharedVid, c_row + c_offset[lastCachedMaxUid], sizeof(int) * len);
            }
        }
        __syncthreads();

        while (vpos >= c_offset[uid + 1]) uid++;
        while (1) {
            while (wpos >= c_adjLen[vid]) {
                wpos -= c_adjLen[vid];
                vpos++;
                vid = c_row[vpos];
                while (vpos >= c_offset[uid + 1]) {
                    uid++;
                }
            }
            if (uid >= cachedMaxUid) {
                break;
            }
            vertex_t *dstList = c_row + c_offset[vid];
            int targetValue = dstList[wpos];
            if (!cacheWorked) {
                vertex_t *srcList = c_row + c_offset[uid];
                uint s = 0, e = c_adjLen[uid];
                uint mid = (s + e) >> 1;
                while (s + 1 < e) {
                    if (srcList[mid] <= targetValue) {
                        s = mid;
                    } else {
                        e = mid;
                    }
                    mid = (s + e) >> 1;
                }
                if (srcList[s] == targetValue) sum++;
            } else {
                int adjListOff = c_offset[uid] - c_offset[lastCachedMaxUid];
                int s = 0, e = c_adjLen[uid];
                int mid = (s + e) >> 1;
                while (s + 1 < e) {
                    if (sharedVid[adjListOff + mid] <= targetValue) {
                        s = mid;
                    } else {
                        e = mid;
                    }
                    mid = (s + e) >> 1;
                }
                if (sharedVid[adjListOff + s] == targetValue) sum++;
            }
            wpos += HU_threadsPerBlockInTC;
        }
        if (cachedMaxUid >= uidThre) break;
    }
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 16);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 8);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 4);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 2);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 1);
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadIdx.x % 32 == 0) {
        c_sum[idx >> 5] = sum;
    }
    return;
}

void tc::approach::Hu::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    long int triangleCount = 0;

    uint *d_adjLength;
    index_t *d_edgeOffset = gpu_graph.beg_pos;
    vertex_t *d_edgeRow = gpu_graph.adj_list;
    uint nodeNum = gpu_graph.vertex_count;
    uint edgeNum = gpu_graph.edge_count;

    HRR(hipMalloc(&d_adjLength, sizeof(uint) * (nodeNum + 1)));

    int block_size = 1024;
    int vertex_grid_size = (nodeNum - 1) / block_size + 1;
    cuda_graph_comm::cal_out_degree_by_offset<<<vertex_grid_size, block_size>>>(edgeNum, nodeNum, d_adjLength, d_edgeOffset);
    HRR(hipDeviceSynchronize());

    cuda_graph_comm::set_value_by_index(d_edgeOffset, nodeNum + 1, (index_t)edgeNum + 1);
    cuda_graph_comm::set_value_by_index(d_adjLength, nodeNum, (uint)1024);
    cuda_graph_comm::set_value_by_index(d_edgeRow, edgeNum, nodeNum);

    cuda_graph_comm::check_array("d_adjLength", d_adjLength, nodeNum + 1, nodeNum - 10, nodeNum + 1);

    long int *nodeWorkLoad;
    int maxThreadsPerBlock = 1024;
    int maxBlocksPerGrid = 64000;
    int maxThreadsPerKernel = maxBlocksPerGrid * maxThreadsPerBlock;
    long int *d_nodeWorkLoad;
    int *d_blockStartNodeOffset;

    int threadsPerKernelInTC = HU_threadsPerBlockInTC * HU_blocksPerKernelInTC;
    int maxWarpPerGrid = threadsPerKernelInTC / 32;
    int *blockStartNodeOffset;
    HRR(hipMalloc(&d_blockStartNodeOffset, sizeof(int) * (HU_blocksPerKernelInTC + 1)));

    double t_start = wtime();

    int iterations = config_comm::cPreprocessingIterations;
    for (int iter = 0; iter < iterations; iter++) {
        HRR(hipMalloc(&d_nodeWorkLoad, sizeof(long int) * maxThreadsPerKernel));

        nodeWorkLoad = new long int[nodeNum];

        for (int i = 0; i < (nodeNum + maxThreadsPerKernel - 1) / maxThreadsPerKernel; i++) {
            int curThread = maxThreadsPerKernel;
            int remainedNodes = nodeNum - i * maxThreadsPerKernel;
            curThread = (remainedNodes > curThread) ? curThread : remainedNodes;
            HRR(hipMemset(d_nodeWorkLoad, 0, sizeof(long) * curThread));
            tc::approach::Hu::getNodesWorkLoad<<<(curThread + maxThreadsPerBlock - 1) / maxThreadsPerBlock, maxThreadsPerBlock>>>(
                i * maxThreadsPerKernel, curThread, d_nodeWorkLoad, d_edgeOffset, d_edgeRow, d_adjLength);

            HRR(hipMemcpy(nodeWorkLoad + i * maxThreadsPerKernel, d_nodeWorkLoad, sizeof(long int) * curThread, hipMemcpyDeviceToHost));
        }
        HRR(hipFree(d_nodeWorkLoad));

        for (int i = 1; i < nodeNum; i++) nodeWorkLoad[i] += nodeWorkLoad[i - 1];

        long int workLoadStep = (nodeWorkLoad[nodeNum - 1] + HU_blocksPerKernelInTC - 1) / HU_blocksPerKernelInTC;

        blockStartNodeOffset = new int[HU_blocksPerKernelInTC + 1];
        blockStartNodeOffset[0] = 0;
#pragma omp parallel for
        for (int i = 1; i < HU_blocksPerKernelInTC; i++) {
            blockStartNodeOffset[i] = (int)tc::approach::Hu::binarySearchValue(nodeWorkLoad, i * workLoadStep + 1, nodeNum, -1);
        }
        blockStartNodeOffset[HU_blocksPerKernelInTC] = nodeNum;

        delete[] nodeWorkLoad;

        HRR(hipMemcpy((void *)d_blockStartNodeOffset, (void *)blockStartNodeOffset, sizeof(int) * (HU_blocksPerKernelInTC + 1),
                       hipMemcpyHostToDevice));

        delete[] blockStartNodeOffset;
    }

    double t_end = wtime();

    // algorithm, dataset, iterations, avg compute time/s,
    auto preprocessing_logger = spdlog::get("Hu_preprocessing_file_logger");
    if (preprocessing_logger) {
        preprocessing_logger->info("{0}\t{1}\t{2}\t{3:.6f}", "Hu", gpu_graph.input_dir, iterations, (t_end - t_start) / iterations);
    } else {
        spdlog::warn("Logger 'Hu_preprocessing_file_logger' is not initialized.");
    }

    long int *d_sum;
    HRR(hipMalloc(&d_sum, sizeof(long int) * maxWarpPerGrid));
    HRR(hipMemset(d_sum, 0, sizeof(long int) * maxWarpPerGrid));

    double total_kernel_use = 0;
    double startKernel, ee;
    for (int i = 0; i < iteration_count; i++) {
        startKernel = wtime();

        tc::approach::Hu::triangleCountKernel<<<HU_blocksPerKernelInTC, HU_threadsPerBlockInTC>>>(d_edgeOffset, d_edgeRow, d_adjLength,
                                                                                                  d_blockStartNodeOffset, d_sum);
        HRR(hipDeviceSynchronize());
        triangleCount = thrust::reduce((thrust::device_ptr<long>)d_sum, (thrust::device_ptr<long>)(d_sum + maxWarpPerGrid));

        ee = wtime();
        total_kernel_use += ee - startKernel;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - startKernel > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, iteration_count, avg compute time/s,
    auto logger = spdlog::get("Hu_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "Hu", gpu_graph.input_dir, triangleCount, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'Hu_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", triangleCount);

    HRR(hipFree(d_sum));
    HRR(hipFree(d_blockStartNodeOffset));
    HRR(hipFree(d_adjLength));
}

void tc::approach::Hu::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "Hu", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("Hu before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::Hu::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Hu after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after Hu runs.");
        }
    }
}
