#include "hip/hip_runtime.h"
#include "approach/Green/main.cuh"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

void tc::approach::Green::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    uint thread_count = config.GetUnsigned(key_space, "thread_count", 512);
    int64_t threadsPerIntsctn = config.GetInteger64(key_space, "threads_per_intsctn", 32);
    int64_t intsctnPerBlock = thread_count / threadsPerIntsctn;
    int64_t threadShift = std::log2(threadsPerIntsctn);

    int64_t vertexCount = (int64_t)gpu_graph.vertex_count;
    int64_t edgeCount = (int64_t)gpu_graph.edge_count;
    int64_t* offsetVector = (int64_t*)gpu_graph.beg_pos;
    int64_t* indexVector;

    int block_size = 1024;
    int edge_grid_size = (edgeCount - 1) / block_size + 1;
    HRR(hipMalloc((void**)&indexVector, sizeof(int64_t) * edgeCount));
    cuda_graph_comm::copy_32_to_64<<<edge_grid_size, block_size>>>(edgeCount, (int32_t*)gpu_graph.adj_list, indexVector);
    HRR(hipDeviceSynchronize());

    thrust::device_vector<int64_t> dTriangleOutputVector(vertexCount, 0);
    int64_t* const dTriangle = thrust::raw_pointer_cast(dTriangleOutputVector.data());

    uint blocks = 1000000;
    if (edgeCount / 10 < blocks) {
        blocks = edgeCount / 10;
    }

    uint64_t triangleCount;
    double total_kernel_use = 0;
    double startKernel, ee;

    for (int i = 0; i < iteration_count; i++) {
        startKernel = wtime();
        kernelCall(blocks, thread_count, vertexCount, offsetVector, indexVector, dTriangle, threadsPerIntsctn, intsctnPerBlock, threadShift);
        hipDeviceSynchronize();
        triangleCount = thrust::reduce(dTriangleOutputVector.begin(), dTriangleOutputVector.end());
        ee = wtime();
        total_kernel_use += ee - startKernel;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - startKernel > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, triangle_count, iteration_count, avg kernel time/s
    auto logger = spdlog::get("Green_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "Green", gpu_graph.input_dir, triangleCount, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'Green_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", triangleCount);

    HRR(hipFree(indexVector));
}

void tc::approach::Green::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "Green", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("Green before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::Green::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Green after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after Green runs.");
        }
    }
}