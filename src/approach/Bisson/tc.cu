#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <omp.h>

#include <cassert>

#include "approach/Bisson/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

__global__ void tc::approach::Bisson::triangleCountKernel(uint nodeNum, index_t *c_offset, vertex_t *c_row, uint *c_adjLen, long int *c_sum,
                                                          int *c_bitmap) {
    long int sum = 0;
    int curRowNum = blockIdx.x;
    int lane_id = threadIdx.x % 32;
    __shared__ int sh_bitMap[Bisson_shareMemorySizeInBlock];
    uint intSizePerBitmap = (nodeNum + 31) / 32;
    int *myBitmap = c_bitmap + blockIdx.x * intSizePerBitmap;
    while (1) {
        //__syncthreads();
        // int privateRowNum = (curRowNum < nonZeroSize) ? c_nonZeroRow[curRowNum] : totalNodeNum;
        int privateRowNum = curRowNum;
        if (privateRowNum >= nodeNum) {
            break;
        }
        // if (c_offset[privateRowNum+1] == c_offset[privateRowNum])
        //	continue;
        vertex_t *curNodeNbr = c_row + c_offset[privateRowNum];
        uint curNodeNbrLength = c_offset[privateRowNum + 1] - c_offset[privateRowNum];
        if (curNodeNbrLength > 256) {
            // if (1) {
            if (threadIdx.x == 0) {
                memset(myBitmap, 0, sizeof(int) * intSizePerBitmap);
                memset(sh_bitMap, 0, sizeof(int) * Bisson_shareMemorySizeInBlock);
            }
            __threadfence();
            for (int i = (curNodeNbrLength + blockDim.x - 1) / blockDim.x - 1; i >= 0; i--) {
                int curIndex = i * blockDim.x + threadIdx.x;
                int curNbr;
                if (curIndex < curNodeNbrLength) {
                    curNbr = curNodeNbr[curIndex];
                    atomicOr(myBitmap + (curNbr / 32), 1 << (31 - curNbr % 32));
                    atomicOr(sh_bitMap + (curNbr / Bisson_hIndex / 32), 1 << (31 - (curNbr / Bisson_hIndex) % 32));
                }
                __syncthreads();
                if (curIndex < curNodeNbrLength) {
                    vertex_t *twoHoopNbr = c_row + c_offset[curNbr];
                    uint twoHoopNbrLength = c_offset[curNbr + 1] - c_offset[curNbr];
                    for (int j = 0; j < twoHoopNbrLength; j++) {
                        vertex_t curValue = twoHoopNbr[j];
                        if (((sh_bitMap[curValue / Bisson_hIndex / 32] >> (31 - (curValue / Bisson_hIndex) % 32)) & 1) &&
                            ((myBitmap[curValue / 32] >> (31 - curValue % 32)) & 1)) {
                            sum++;
                        }
                    }
                }
            }
        } else {
            if (threadIdx.x == 0) memcpy(sh_bitMap, curNodeNbr, sizeof(int) * curNodeNbrLength);
            __threadfence();
            for (int i = lane_id; i < curNodeNbrLength; i += 32) {
                int curNbr = curNodeNbr[i];
                vertex_t *twoHoopNbr = c_row + c_offset[curNbr];
                int twoHoopNbrLength = c_offset[curNbr + 1] - c_offset[curNbr];
                for (int j = 0; j < twoHoopNbrLength; j++) {
                    int targetValue = twoHoopNbr[j];
                    int s = 0, e = curNodeNbrLength, mid;
                    while (s < e) {
                        mid = (s + e) / 2;
                        if (sh_bitMap[mid] > targetValue)
                            e = mid;
                        else if (sh_bitMap[mid] < targetValue)
                            s = mid + 1;
                        else {
                            sum++;
                            break;
                        }
                    }
                }
            }
        }
        curRowNum += gridDim.x;  // atomicAdd(&nextNode, 1);
                                 //__syncthreads();
                                 // if (privateRowNum != curRowNum)
                                 //	printf("private is %d, curRowNum is %d, block %d, index %d\n",privateRowNum,curRowNum,blockIdx.x,threadIdx.x);
    }

    sum += __shfl_down_sync(0xFFFFFFFF, sum, 16);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 8);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 4);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 2);
    sum += __shfl_down_sync(0xFFFFFFFF, sum, 1);
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadIdx.x % 32 == 0) {
        c_sum[idx >> 5] = sum;
    }
    return;
}

void tc::approach::Bisson::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    long int triangleCount = 0;
    uint *d_adjLength;
    index_t *d_edgeOffset = gpu_graph.beg_pos;
    vertex_t *d_edgeRow = gpu_graph.adj_list;
    uint nodeNum = gpu_graph.vertex_count;
    uint edgeNum = gpu_graph.edge_count;

    HRR(hipMalloc(&d_adjLength, sizeof(int) * (nodeNum + 1)));

    int block_size = 1024;
    int vertex_grid_size = (nodeNum - 1) / block_size + 1;
    cuda_graph_comm::cal_out_degree_by_offset<<<vertex_grid_size, block_size>>>(edgeNum, nodeNum, d_adjLength, d_edgeOffset);
    HRR(hipDeviceSynchronize());

    cuda_graph_comm::set_value_by_index(d_edgeOffset, nodeNum + 1, (index_t)edgeNum + 1);
    cuda_graph_comm::set_value_by_index(d_adjLength, nodeNum, (uint)1024);
    cuda_graph_comm::set_value_by_index(d_edgeRow, edgeNum, nodeNum);

    int bitPerInt = sizeof(int) * 8;
    int intSizePerBitmap = (nodeNum + bitPerInt - 1) / bitPerInt;
    int blockSize = 32;
    int blockNum = 30 * 2048 / blockSize;

    if (nodeNum > Bisson_hIndex * Bisson_shareMemorySizeInBlock * 32) {
        spdlog::error("The nodeNum is too large: {}", nodeNum);
        HRR(hipFree(d_adjLength));
        return;
    }
    if ((long long int)blockNum * intSizePerBitmap * sizeof(int) > (long long int)16 * MEMORY_G) {
        spdlog::error("The bitmap is too large: {} bytes", (long long int)blockNum * intSizePerBitmap * sizeof(int));
        HRR(hipFree(d_adjLength));
        return;
    }

    int *d_bitmaps;
    HRR(hipMalloc(&d_bitmaps, sizeof(int) * intSizePerBitmap * blockNum));

    long int *d_sum;
    unsigned maxWarpPerGrid = blockNum * blockSize / 32;
    HRR(hipMalloc(&d_sum, sizeof(long int) * maxWarpPerGrid));
    HRR(hipMemset(d_sum, 0, sizeof(long int) * maxWarpPerGrid));

    double total_kernel_use = 0;
    double startKernel, ee;
    for (int i = 0; i < iteration_count; i++) {
        startKernel = wtime();
        tc::approach::Bisson::triangleCountKernel<<<blockNum, blockSize>>>(nodeNum, d_edgeOffset, d_edgeRow, d_adjLength, d_sum, d_bitmaps);
        HRR(hipDeviceSynchronize());
        triangleCount = thrust::reduce((thrust::device_ptr<long>)d_sum, (thrust::device_ptr<long>)(d_sum + maxWarpPerGrid));
        ee = wtime();

        total_kernel_use += ee - startKernel;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - startKernel > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, triangle_count, iteration_count, avg kernel time/s
    auto logger = spdlog::get("Bisson_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "Bisson", gpu_graph.input_dir, triangleCount, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'Bisson_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", triangleCount);

    HRR(hipFree(d_sum));
    HRR(hipFree(d_adjLength));
    HRR(hipFree(d_bitmaps));
}

void tc::approach::Bisson::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "Bisson", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("Bisson before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::Bisson::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("Bisson after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after Bisson runs.");
        }
    }
}
