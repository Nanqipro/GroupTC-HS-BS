#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/TRUST/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

__device__ int tc::approach::TRUST::linear_search(int neighbor, int* shared_partition, int* partition, int* bin_count, int bin, int BIN_START) {
    for (;;) {
        int i = bin;
        int len = bin_count[i];
        int step = 0;
        int nowlen;
        if (len < TRUST_SHARED_BUCKET_SIZE)
            nowlen = len;
        else
            nowlen = TRUST_SHARED_BUCKET_SIZE;
        while (step < nowlen) {
            if (shared_partition[i] == neighbor) {
                return 1;
            }
            i += TRUST_BLOCK_BUCKETNUM;
            step += 1;
        }

        len -= TRUST_SHARED_BUCKET_SIZE;
        i = bin + BIN_START;
        step = 0;
        while (step < len) {
            if (partition[i] == neighbor) {
                return 1;
            }
            i += TRUST_BLOCK_BUCKETNUM;
            step += 1;
        }
        if (len + TRUST_SHARED_BUCKET_SIZE < 99) break;
        bin++;
    }
    return 0;
}

int tc::approach::TRUST::my_binary_search(int len, int val, index_t* beg) {
    int l = 0, r = len;
    while (l < r - 1) {
        int mid = (l + r) / 2;
        if (beg[mid + 1] - beg[mid] > val)
            l = mid;
        else
            r = mid;
    }
    if (beg[l + 1] - beg[l] <= val) return -1;
    return l;
}

__global__ void tc::approach::TRUST::trust(vertex_t* adj_list, index_t* beg_pos, uint edge_count, uint vertex_count, int* partition,
                                           unsigned long long* GLOBAL_COUNT, int* G_INDEX, int CHUNK_SIZE, int warpfirstvertex) {
    // hashTable bucket 计数器
    __shared__ int bin_count[TRUST_BLOCK_BUCKETNUM];
    // 共享内存中的 hashTable
    __shared__ int shared_partition[TRUST_BLOCK_BUCKETNUM * TRUST_SHARED_BUCKET_SIZE + 1];
    unsigned long long __shared__ G_counter;
    int WARPSIZE = 32;
    if (threadIdx.x == 0) {
        G_counter = 0;
    }

    int BIN_START = blockIdx.x * TRUST_BLOCK_BUCKETNUM * TRUST_BUCKET_SIZE;
    // __syncthreads();
    unsigned long long P_counter = 0;

    // CTA for large degree vertex
    int vertex = blockIdx.x * CHUNK_SIZE;
    int vertex_end = vertex + CHUNK_SIZE;
    __shared__ int ver;
    while (vertex < warpfirstvertex)
    // while (0)
    {
        // if (degree<=TRUST_USE_CTA) break;
        int start = beg_pos[vertex];
        int end = beg_pos[vertex + 1];
        int now = threadIdx.x + start;
        int MODULO = TRUST_BLOCK_BUCKETNUM - 1;
        // int divide=(vert_count/blockDim.x);
        int BIN_OFFSET = 0;
        // clean bin_count
        // 初始化 hashTable bucket 计数器
        for (int i = threadIdx.x; i < TRUST_BLOCK_BUCKETNUM; i += blockDim.x) bin_count[i] = 0;
        __syncthreads();

        // start_time = clock64();
        // count hash bin
        // 生成 hashTable
        while (now < end) {
            int temp = adj_list[now];
            int bin = temp & MODULO;
            int index;
            index = atomicAdd(&bin_count[bin], 1);
            if (index < TRUST_SHARED_BUCKET_SIZE) {
                shared_partition[index * TRUST_BLOCK_BUCKETNUM + bin] = temp;
            } else if (index < TRUST_BUCKET_SIZE) {
                index = index - TRUST_SHARED_BUCKET_SIZE;
                partition[index * TRUST_BLOCK_BUCKETNUM + bin + BIN_START] = temp;
            }
            now += blockDim.x;
        }
        __syncthreads();

        now = beg_pos[vertex];
        end = beg_pos[vertex + 1];
        int superwarp_ID = threadIdx.x / 64;
        int superwarp_TID = threadIdx.x % 64;
        int workid = superwarp_TID;
        now = now + superwarp_ID;
        // 获取二跳邻居节点
        int neighbor = adj_list[now];
        int neighbor_start = beg_pos[neighbor];
        int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += 16;
                workid -= neighbor_degree;
                neighbor = adj_list[now];
                neighbor_start = beg_pos[neighbor];
                neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
            }
            if (now < end) {
                int temp = adj_list[neighbor_start + workid];
                int bin = temp & MODULO;
                P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
            }
            // __syncthreads();
            workid += 64;
        }

        __syncthreads();
        // if (vertex>1) break;
        vertex++;
        if (vertex == vertex_end) {
            if (threadIdx.x == 0) {
                ver = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
            }
            __syncthreads();
            vertex = ver;
            vertex_end = vertex + CHUNK_SIZE;
        }
        // __syncthreads();
    }

    // warp method
    int WARPID = threadIdx.x / WARPSIZE;
    int WARP_TID = threadIdx.x % WARPSIZE;
    vertex = warpfirstvertex + ((WARPID + blockIdx.x * blockDim.x / WARPSIZE)) * CHUNK_SIZE;
    vertex_end = vertex + CHUNK_SIZE;
    while (vertex < vertex_count) {
        int degree = beg_pos[vertex + 1] - beg_pos[vertex];
        if (degree < TRUST_USE_WARP) break;
        int start = beg_pos[vertex];
        int end = beg_pos[vertex + 1];
        int now = WARP_TID + start;
        int MODULO = TRUST_WARP_BUCKETNUM - 1;
        int BIN_OFFSET = WARPID * TRUST_WARP_BUCKETNUM;
        // clean bin_count

        for (int i = BIN_OFFSET + WARP_TID; i < BIN_OFFSET + TRUST_WARP_BUCKETNUM; i += WARPSIZE) bin_count[i] = 0;
        // bin_count[threadIdx.x]=0;
        //__syncwarp();

        // count hash bin
        while (now < end) {
            int temp = adj_list[now];
            int bin = temp & MODULO;
            bin += BIN_OFFSET;
            int index;
            index = atomicAdd(&bin_count[bin], 1);
            if (index < TRUST_SHARED_BUCKET_SIZE) {
                shared_partition[index * TRUST_BLOCK_BUCKETNUM + bin] = temp;
            } else if (index < TRUST_BUCKET_SIZE) {
                index = index - TRUST_SHARED_BUCKET_SIZE;
                partition[index * TRUST_BLOCK_BUCKETNUM + bin + BIN_START] = temp;
            }
            now += WARPSIZE;
        }
        //__syncwarp();

        now = beg_pos[vertex];
        end = beg_pos[vertex + 1];

        int workid = WARP_TID;
        while (now < end) {
            int neighbor = adj_list[now];
            int neighbor_start = beg_pos[neighbor];
            int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;

            while (now < end && workid >= neighbor_degree) {
                now++;
                workid -= neighbor_degree;
                neighbor = adj_list[now];
                neighbor_start = beg_pos[neighbor];
                neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
            }
            if (now < end) {
                int temp = adj_list[neighbor_start + workid];
                int bin = temp & MODULO;
                P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
            }
            //__syncwarp();
            now = __shfl_sync(0xffffffff, now, 31);
            workid = __shfl_sync(0xffffffff, workid, 31);
            workid += WARP_TID + 1;

            // workid+=WARPSIZE;
        }
        //__syncwarp();
        vertex++;
        if (vertex == vertex_end) {
            if (WARP_TID == 0) {
                vertex = atomicAdd(&G_INDEX[2], CHUNK_SIZE);
            }
            //__syncwarp();
            vertex = __shfl_sync(0xffffffff, vertex, 0);
            vertex_end = vertex + CHUNK_SIZE;
        }
    }

    atomicAdd(&G_counter, P_counter);

    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
    }
}

void tc::approach::TRUST::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    int grid_size = 1024;
    int block_size = 1024;
    int chunk_size = 1;

    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;

    index_t* h_beg_pos = (index_t*)malloc(sizeof(index_t) * (vertex_count + 1));
    HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

    int warpfirstvertex = my_binary_search(vertex_count, TRUST_USE_CTA, h_beg_pos) + 1;

    int* BIN_MEM;
    unsigned long long* GLOBAL_COUNT;
    int* G_INDEX;

    index_t* d_beg_pos = gpu_graph.beg_pos;
    vertex_t* d_adj_list = gpu_graph.adj_list;

    unsigned long long* counter = (unsigned long long*)malloc(sizeof(unsigned long long) * 10);

    HRR(hipMalloc((void**)&BIN_MEM, sizeof(int) * grid_size * TRUST_BLOCK_BUCKETNUM * TRUST_BUCKET_SIZE));
    HRR(hipMalloc((void**)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
    HRR(hipMalloc((void**)&G_INDEX, sizeof(int) * 3));

    int T_Group = 32;
    int nowindex[3];
    nowindex[0] = chunk_size * grid_size * block_size / T_Group;
    nowindex[1] = chunk_size * grid_size;
    nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

    HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

    double total_kernel_use = 0;
    double startKernel, ee = 0;
    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
        startKernel = wtime();
        hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
        tc::approach::TRUST::trust<<<grid_size, block_size>>>(d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, G_INDEX,
                                                              chunk_size, warpfirstvertex);
        HRR(hipDeviceSynchronize());

        ee = wtime();
        total_kernel_use += ee - startKernel;
    }

    HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));

    // algorithm, dataset, triangle_count, iteration_count, avg kernel time/s
    spdlog::get("TRUST_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "TRUST", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", counter[0]);

    free(counter);
    free(h_beg_pos);
    HRR(hipFree(BIN_MEM));
    HRR(hipFree(GLOBAL_COUNT));
    HRR(hipFree(G_INDEX));
}

void tc::approach::TRUST::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "TRUST", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("TRUST before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::TRUST::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("TRUST after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after TRUST runs.");
        }
    }
}
