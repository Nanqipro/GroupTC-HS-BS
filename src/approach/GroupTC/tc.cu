#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/GroupTC/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

// 实现一个二分查找，成功返回1，否则返回0
__device__ int tc::approach::GroupTC::bin_search(vertex_t* arr, int len, int val) {
    uint32_t Y;
    int32_t bot = 0;
    int32_t top = len - 1;
    // 中点
    int32_t r;
    while (top >= bot) {

        r = (top + bot) / 2;
        Y = arr[r];

        if (val == Y) {
            return 1;
        }

        if (val < Y) {
            top = r - 1;
        } else {
            bot = r + 1;
        }
    }
    return 0;
}

// 优化后的二分搜索算法--目的是减少分支预测失误以提高性能
__device__ int tc::approach::GroupTC::bin_search_less_branch(vertex_t* arr, int len, int val) {
    // 用于跟踪当前位置
    int ret = 0;

    int halfsize;

    int candidate;
    int temp = len;

    while (temp > 1) {
        halfsize = temp / 2;
        candidate = arr[ret + halfsize];
        ret += (candidate < val) ? halfsize : 0;
        temp -= halfsize;
    }

    ret += (arr[ret] < val);

    return ret < len && arr[ret] == val;
}

__device__ int tc::approach::GroupTC::bin_search_with_offset_and_less_branch(vertex_t* arr, int len, int val, int& offset) {
    int ret = 0;
    int halfsize;
    int candidate;
    int temp = len;
    while (temp > 1) {

        halfsize = temp / 2;

        candidate = arr[ret + halfsize];

        ret += (candidate < val) ? halfsize : 0;
        temp -= halfsize;
    }
    ret += (arr[ret] < val);

    // offset 的作用是用来存储二分查找过程中得到的结果位置，即查找到的元素的索引，或者如果没有找到该元素时，表示它应当插入的位置。
    offset = ret;

    return ret < len && arr[ret] == val;
}

/**
 * @brief 计算GROUP_TC（Triangle Count）的GPU内核函数
 * 
 * 该函数通过遍历边列表，利用共享内存中的哈希表来加速查找二跳邻居节点的过程，从而计算出团的数量
 * 
 * @param src_list 边的源节点数组
 * @param adj_list 邻接表，存储边的目标节点
 * @param beg_pos 每个节点在邻接表中开始位置的索引数组
 * @param edge_count 边的数量
 * @param vertex_count 节点的数量
 * @param GLOBAL_COUNT 用于存储每个线程块计算结果的全局计数器数组
 */
__global__ void tc::approach::GroupTC::grouptc(vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos, uint edge_count, uint vertex_count,
                                               unsigned long long* GLOBAL_COUNT) {
    // 定义共享内存，用于存储块内的哈希表数据
    __shared__ int sh_tb_start[GroupTC_BLOCK_BUCKETNUM]; // 块中存储每个线程的邻接表起点
    __shared__ int sh_tb_len[GroupTC_BLOCK_BUCKETNUM]; // 块中存储每个线程邻接表的长度
    __shared__ int sh_ele_start[GroupTC_BLOCK_BUCKETNUM]; // 块中存储每个线程的二跳邻居的起点
    __shared__ int sh_ele_len[GroupTC_BLOCK_BUCKETNUM]; // 块中存储每个线程的二跳邻居列表长度
    // 定义一个局部变量计数器，用于存储当前块的结果 （每个线程块的计算结果） 
    unsigned long long P_counter = 0;
    // 获取当前线程块的索引 (bid) 和当前线程的索引 (tid)
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    // 以线程块为单位进行处理
    for (int i = bid * GroupTC_BLOCK_BUCKETNUM; i < edge_count; i += gridDim.x * GroupTC_BLOCK_BUCKETNUM) {
        // 检查是否超出边界范围
        if (i + tid < edge_count) {
            // 获取当前边的源节点和目的节点
            vertex_t src = src_list[i + tid];
            vertex_t dst = adj_list[i + tid];

            int temp;

            // 初始化起点和长度，用于一阶邻居和二阶邻居
            int tb_start, tb_len, ele_start, ele_len;

            // i + tid + 1 结合了当前线程块（block）和线程（thread）的索引，保证每个线程都能处理一条唯一的边，并且 +1 的操作避免了自查找。
            // 这个方式的核心在于，每个线程块处理不同的边，并且在块内由每个线程处理不同的边起点，确保每个线程有各自的起始位置，不会发生重复。

            tb_start = i + tid + 1; // 当前线程处理的邻居起点（可通过邻接表偏移调整）
            // tb_start = beg_pos[src];
            tb_len = beg_pos[src + 1] - tb_start;// 当前源节点邻接表的长度
            ele_start = beg_pos[dst]; // 当前目的节点的邻接表起点
            ele_len = beg_pos[dst + 1] - ele_start; // 当前目的节点邻接表的长度

            // 优化处理：如果源节点邻居较少，将源和目的节点的邻接表交换
            if (tb_len * 2 < ele_len) {
                temp = tb_start;
                tb_start = ele_start;
                ele_start = temp;

                temp = tb_len;
                tb_len = ele_len;
                ele_len = temp;
            }


            // 将当前线程的邻接表数据存储到共享内存中
            sh_tb_start[tid] = tb_start;
            sh_tb_len[tid] = tb_len;
            sh_ele_start[tid] = ele_start;
            sh_ele_len[tid] = ele_len;
        }
        __syncthreads();

        // 计算当前线程属于哪个子组（sub-warp），即现在处理哪一块数据
        int now = tid / GroupTC_SUBWARP_SIZE;
        int end = min(edge_count - i, GroupTC_BLOCK_BUCKETNUM);
        int workid = tid % GroupTC_SUBWARP_SIZE;
        int offset = 0;
        int last_now = -1;

        // 获取二跳邻居节点
        int neighbor_degree = sh_ele_len[now];
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += GroupTC_WARP_STEP;
                if (now < end) {
                    workid -= neighbor_degree;
                    neighbor_degree = sh_ele_len[now];
                }
            }

            if (now < end) {
                offset = last_now == now ? offset : 0;
                P_counter += tc::approach::GroupTC::bin_search_with_offset_and_less_branch(
                    adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
                last_now = now;
            }
            workid += GroupTC_SUBWARP_SIZE;
        }
        __syncthreads();
    }

    GLOBAL_COUNT[bid * GroupTC_BLOCK_BUCKETNUM + tid] = P_counter;
}

void tc::approach::GroupTC::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    vertex_t vertex_count = gpu_graph.vertex_count;
    vertex_t edge_count = gpu_graph.edge_count;
    int grid_size = NumberOfMPs() * 8;

    double t_start, total_kernel_use = 0;
    uint64_t count;

    unsigned long long* d_results;
    HRR(hipMalloc(&d_results, grid_size * GroupTC_BLOCK_BUCKETNUM * sizeof(unsigned long long)));

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(d_results, 0, grid_size * GroupTC_BLOCK_BUCKETNUM * sizeof(unsigned long long)));

        t_start = wtime();

        tc::approach::GroupTC::grouptc<<<grid_size, GroupTC_BLOCK_BUCKETNUM>>>(d_src, d_adj, d_beg_pos, edge_count, vertex_count, d_results);
        HRR(hipDeviceSynchronize());

        thrust::device_ptr<unsigned long long> ptr(d_results);
        count = thrust::reduce(ptr, ptr + (grid_size * GroupTC_BLOCK_BUCKETNUM));

        double ee = wtime();
        total_kernel_use += ee - t_start;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - t_start > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, iteration_count, avg compute time/s,
    spdlog::get("GroupTC_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC", gpu_graph.input_dir, count, iteration_count, total_kernel_use / iteration_count);

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", count);

    HRR(hipFree(d_results));
}

void tc::approach::GroupTC::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "GroupTC", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("GroupTC before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::GroupTC::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("GroupTC after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after GroupTC runs.");
        }
    }
}
