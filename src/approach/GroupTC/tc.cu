#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/GroupTC/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

__device__ int tc::approach::GroupTC::bin_search(vertex_t* arr, int len, int val) {
    uint32_t Y;
    int32_t bot = 0;
    int32_t top = len - 1;
    int32_t r;
    while (top >= bot) {
        r = (top + bot) / 2;
        Y = arr[r];

        if (val == Y) {
            return 1;
        }

        if (val < Y) {
            top = r - 1;
        } else {
            bot = r + 1;
        }
    }
    return 0;
}

__device__ int tc::approach::GroupTC::bin_search_less_branch(vertex_t* arr, int len, int val) {
    int ret = 0;
    int halfsize;
    int candidate;
    int temp = len;
    while (temp > 1) {
        halfsize = temp / 2;
        candidate = arr[ret + halfsize];
        ret += (candidate < val) ? halfsize : 0;
        temp -= halfsize;
    }
    ret += (arr[ret] < val);
    return ret < len && arr[ret] == val;
}

__device__ int tc::approach::GroupTC::bin_search_with_offset_and_less_branch(vertex_t* arr, int len, int val, int& offset) {
    int ret = 0;
    int halfsize;
    int candidate;
    int temp = len;
    while (temp > 1) {
        halfsize = temp / 2;
        candidate = arr[ret + halfsize];
        ret += (candidate < val) ? halfsize : 0;
        temp -= halfsize;
    }
    ret += (arr[ret] < val);
    offset = ret;
    return ret < len && arr[ret] == val;
}

__global__ void tc::approach::GroupTC::grouptc(vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos, uint edge_count, uint vertex_count,
                                               unsigned long long* GLOBAL_COUNT) {
    // 共享内存中的 hashTable
    __shared__ int sh_tb_start[GroupTC_BLOCK_BUCKETNUM];
    __shared__ int sh_tb_len[GroupTC_BLOCK_BUCKETNUM];
    __shared__ int sh_ele_start[GroupTC_BLOCK_BUCKETNUM];
    __shared__ int sh_ele_len[GroupTC_BLOCK_BUCKETNUM];

    unsigned long long P_counter = 0;

    int bid = blockIdx.x;
    int tid = threadIdx.x;

    for (int i = bid * GroupTC_BLOCK_BUCKETNUM; i < edge_count; i += gridDim.x * GroupTC_BLOCK_BUCKETNUM) {
        if (i + tid < edge_count) {
            vertex_t src = src_list[i + tid];
            vertex_t dst = adj_list[i + tid];
            int temp;

            int tb_start, tb_len, ele_start, ele_len;
            tb_start = i + tid + 1;
            // tb_start = beg_pos[src];
            tb_len = beg_pos[src + 1] - tb_start;
            ele_start = beg_pos[dst];
            ele_len = beg_pos[dst + 1] - ele_start;

            if (tb_len * 2 < ele_len) {
                temp = tb_start;
                tb_start = ele_start;
                ele_start = temp;

                temp = tb_len;
                tb_len = ele_len;
                ele_len = temp;
            }

            sh_tb_start[tid] = tb_start;
            sh_tb_len[tid] = tb_len;
            sh_ele_start[tid] = ele_start;
            sh_ele_len[tid] = ele_len;
        }

        __syncthreads();

        int now = tid / GroupTC_SUBWARP_SIZE;
        int end = min(edge_count - i, GroupTC_BLOCK_BUCKETNUM);
        int workid = tid % GroupTC_SUBWARP_SIZE;
        int offset = 0;
        int last_now = -1;

        // 获取二跳邻居节点
        int neighbor_degree = sh_ele_len[now];
        while (now < end) {
            // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
            while (now < end && workid >= neighbor_degree) {
                now += GroupTC_WARP_STEP;
                if (now < end) {
                    workid -= neighbor_degree;
                    neighbor_degree = sh_ele_len[now];
                }
            }

            if (now < end) {
                offset = last_now == now ? offset : 0;
                P_counter += tc::approach::GroupTC::bin_search_with_offset_and_less_branch(
                    adj_list + (sh_tb_start[now] + offset), sh_tb_len[now] - offset, adj_list[sh_ele_start[now] + workid], offset);
                last_now = now;
            }
            workid += GroupTC_SUBWARP_SIZE;
        }
        __syncthreads();
    }

    GLOBAL_COUNT[bid * GroupTC_BLOCK_BUCKETNUM + tid] = P_counter;
}

void tc::approach::GroupTC::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    vertex_t vertex_count = gpu_graph.vertex_count;
    vertex_t edge_count = gpu_graph.edge_count;
    int grid_size = NumberOfMPs() * 8;

    double t_start, total_kernel_use = 0;
    uint64_t count;

    unsigned long long* d_results;
    HRR(hipMalloc(&d_results, grid_size * GroupTC_BLOCK_BUCKETNUM * sizeof(unsigned long long)));

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(d_results, 0, grid_size * GroupTC_BLOCK_BUCKETNUM * sizeof(unsigned long long)));

        t_start = wtime();

        tc::approach::GroupTC::grouptc<<<grid_size, GroupTC_BLOCK_BUCKETNUM>>>(d_src, d_adj, d_beg_pos, edge_count, vertex_count, d_results);
        HRR(hipDeviceSynchronize());

        thrust::device_ptr<unsigned long long> ptr(d_results);
        count = thrust::reduce(ptr, ptr + (grid_size * GroupTC_BLOCK_BUCKETNUM));

        double ee = wtime();
        total_kernel_use += ee - t_start;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - t_start > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, iteration_count, avg compute time/s,
    auto logger = spdlog::get("GroupTC_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC", gpu_graph.input_dir, count, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'GroupTC_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", count);

    HRR(hipFree(d_results));
}

void tc::approach::GroupTC::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "GroupTC", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("GroupTC before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::GroupTC::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("GroupTC after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after GroupTC runs.");
        }
    }
}
