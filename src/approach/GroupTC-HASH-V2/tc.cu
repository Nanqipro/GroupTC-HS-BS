#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <string>

#include "approach/GroupTC-HASH-V2/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "spdlog/spdlog.h"

typedef struct longint2 {
    long long int x, y;  // 两个 long 类型的成员
} longint2;

__device__ int tc::approach::GroupTC_HASH_V2::linear_search_block(int neighbor, int *partition, int len, int bin, int BIN_START) {
    for (;;) {
        len -= GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE;
        int i = bin + BIN_START;
        int step = 0;
        while (step < len) {
            if (partition[i] == neighbor) {
                return 1;
            }
            i += GroupTC_HASH_V2_block_bucketnum;
            step += 1;
        }
        if (len + GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE < 99) break;
        bin++;
    }
    return 0;
}

__device__ int tc::approach::GroupTC_HASH_V2::linear_search_group(int neighbor, int *partition, int len, int bin, int BIN_START) {
    len -= GroupTC_HASH_V2_shared_GROUP_BUCKET_SIZE;
    int i = bin + BIN_START;
    int step = 0;
    while (step < len) {
        if (partition[i] == neighbor) {
            return 1;
        }
        i += GroupTC_HASH_V2_group_bucketnum;
        step += 1;
    }

    return 0;
}

int tc::approach::GroupTC_HASH_V2::my_binary_search(int len, int val, index_t *beg) {
    int l = 0, r = len;
    while (l < r - 1) {
        int mid = (l + r) / 2;
        if (beg[mid + 1] - beg[mid] > val)
            l = mid;
        else
            r = mid;
    }
    if (beg[l + 1] - beg[l] <= val) return -1;
    return l;
}

template <const int GroupTC_HASH_V2_Group_SUBWARP_SIZE, const int GroupTC_HASH_V2_Group_WARP_STEP, const int CHUNK_SIZE>
__global__ void tc::approach::GroupTC_HASH_V2::grouptc_hash_v2(vertex_t *src_list, vertex_t *adj_list, index_t *beg_pos, uint edge_count,
                                                               uint vertex_count, int *partition, unsigned long long *GLOBAL_COUNT, int T_Group,
                                                               int *G_INDEX, int warpfirstvertex, int warpfirstedge, int nocomputefirstvertex,
                                                               int nocomputefirstedge) {
    // hashTable bucket 计数器
    __shared__ int bin_count[GroupTC_HASH_V2_block_bucketnum];
    // 共享内存中的 hashTable
    __shared__ int shared_partition[GroupTC_HASH_V2_block_bucketnum * GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE];

    //__shared__ unsigned int bloom_filter[GroupTC_HASH_V2_block_bucketnum];

    int BIN_START = blockIdx.x * GroupTC_HASH_V2_block_bucketnum * GroupTC_HASH_V2_BLOCK_BUCKET_SIZE;
    unsigned long long P_counter = 0;

    int __shared__ vertex;

    if (threadIdx.x == 0) {
        vertex = blockIdx.x;
    }
    __syncthreads();

    // unsigned int* bloom_filter = reinterpret_cast<unsigned int*>(shared_partition +
    // GroupTC_HASH_V2_block_bucketnum*GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE);

    while (vertex < warpfirstvertex) {
        // while (0) {
        // if (degree<=USE_CTA) break;
        int group_start = beg_pos[vertex];
        int end = beg_pos[vertex + 1];
        int now = threadIdx.x + group_start;
        // int MODULO = GroupTC_HASH_V2_block_bucketnum - 1;

        // clean bin_count
        // for (int i = threadIdx.x; i < GroupTC_HASH_V2_block_bucketnum; i += GroupTC_HASH_V2_BLOCK_SIZE){
        bin_count[threadIdx.x] = 0;
        // bloom_filter[threadIdx.x] = 0;
        // }
        __syncthreads();

        // count hash bin
        // 生成 hashTable
        while (now < end) {
            int temp = adj_list[now];

            // insert bloom filter
            // unsigned int index = temp & GroupTC_HASH_V2_BLOOM_FILTER_SIZE;
            // atomicOr(&bloom_filter[index / 32], 1 << (index % 32));

            // insert hash table
            int bin = temp & GroupTC_HASH_V2_BLOCK_MODULO;
            unsigned int index = atomicAdd(&bin_count[bin], 1);
            if (index < GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE) {
                shared_partition[index * GroupTC_HASH_V2_block_bucketnum + bin] = temp;
            } else if (index < GroupTC_HASH_V2_BLOCK_BUCKET_SIZE) {
                index = index - GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE;
                partition[index * GroupTC_HASH_V2_block_bucketnum + bin + BIN_START] = temp;
            }
            now += blockDim.x;
        }
        __syncthreads();

        if (1) {
            // list intersection
            now = beg_pos[vertex];
            end = beg_pos[vertex + 1];
            int superwarp_ID = threadIdx.x / GroupTC_HASH_V2_CTA_WARP_SIZE;
            int superwarp_TID = threadIdx.x % GroupTC_HASH_V2_CTA_WARP_SIZE;
            int workid = superwarp_TID;
            now = now + superwarp_ID;
            // 获取二跳邻居节点
            int neighbor = adj_list[now];
            int neighbor_start = beg_pos[neighbor];
            int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
            while (now < end) {
                // 如果当前一阶邻居节点已被处理完，找下一个一阶邻居节点去处理
                while (now < end && workid >= neighbor_degree) {
                    now += GroupTC_HASH_V2_BLOCK_SIZE / GroupTC_HASH_V2_CTA_WARP_SIZE;
                    workid -= neighbor_degree;
                    neighbor = adj_list[now];
                    neighbor_start = beg_pos[neighbor];
                    neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
                }
                if (now < end) {
                    int temp_adj = adj_list[neighbor_start + workid];

                    // unsigned int index = temp_adj & GroupTC_HASH_V2_BLOOM_FILTER_SIZE;

                    // bloom filter check
                    // if ((bloom_filter[index / 32] & (1 << (index % 32))) != 0) {

                    int bin = temp_adj & GroupTC_HASH_V2_BLOCK_MODULO;

                    int len = bin_count[bin];

                    P_counter += len > 0 && shared_partition[bin + GroupTC_HASH_V2_block_bucketnum * 0] == temp_adj;
                    P_counter += len > 1 && shared_partition[bin + GroupTC_HASH_V2_block_bucketnum * 1] == temp_adj;
                    P_counter += len > 2 && shared_partition[bin + GroupTC_HASH_V2_block_bucketnum * 2] == temp_adj;
                    P_counter += len > 3 && shared_partition[bin + GroupTC_HASH_V2_block_bucketnum * 3] == temp_adj;
                    P_counter += len > 4 && shared_partition[bin + GroupTC_HASH_V2_block_bucketnum * 4] == temp_adj;
                    P_counter += len > 5 && shared_partition[bin + GroupTC_HASH_V2_block_bucketnum * 5] == temp_adj;

                    if (len > GroupTC_HASH_V2_shared_BLOCK_BUCKET_SIZE) {
                        P_counter += tc::approach::GroupTC_HASH_V2::linear_search_block(temp_adj, partition, len, bin, BIN_START);
                    }
                    // }
                }
                // __syncthreads();
                workid += GroupTC_HASH_V2_CTA_WARP_SIZE;
            }
        }

        __syncthreads();
        if (threadIdx.x == 0) {
            vertex = atomicAdd(&G_INDEX[1], CHUNK_SIZE);
        }
        __syncthreads();
    }

    // EDGE CHUNK for small degree vertex
    __shared__ int group_start;
    __shared__ int group_size;

    int *shared_src = shared_partition + GroupTC_HASH_V2_group_bucketnum * GroupTC_HASH_V2_shared_GROUP_BUCKET_SIZE;
    int *shared_adj_start = shared_src + GroupTC_HASH_V2_shared_CHUNK_CACHE_SIZE;
    int *shared_adj_degree = shared_adj_start + GroupTC_HASH_V2_shared_CHUNK_CACHE_SIZE;

    if (1) {
        for (int group_offset = warpfirstedge + blockIdx.x * GroupTC_HASH_V2_EDGE_CHUNK; group_offset < nocomputefirstedge;
             group_offset += gridDim.x * GroupTC_HASH_V2_EDGE_CHUNK) {
            // compute group start and end
            if (threadIdx.x == 0) {
                int src = src_list[group_offset];
                int src_start = beg_pos[src];
                int src_end = beg_pos[src + 1];
                group_start = ((src_start == group_offset) ? src_start : src_end);

                src = src_list[min(group_offset + GroupTC_HASH_V2_EDGE_CHUNK, nocomputefirstedge) - 1];
                group_size = min(beg_pos[src + 1], (index_t)nocomputefirstedge) - group_start;
            }

            // cache start
            for (int i = threadIdx.x; i < GroupTC_HASH_V2_group_bucketnum; i += blockDim.x) bin_count[i] = 0;

            __syncthreads();

            for (int i = threadIdx.x; i < group_size; i += GroupTC_HASH_V2_BLOCK_SIZE) {
                int temp_src = src_list[i + group_start];
                int temp_adj = adj_list[i + group_start];

                longint2 *point_int2 = reinterpret_cast<longint2 *>(beg_pos + temp_adj);
                longint2 pos2 = *point_int2;
                shared_src[i] = temp_src;
                shared_adj_start[i] = pos2.x;
                shared_adj_degree[i] = pos2.y - pos2.x;

                // if (shared_adj_start[i] != beg_pos[temp_adj]) {
                //     printf("shared_adj_start[%d] = %d, beg_pos[%d] = %d\n", i, shared_adj_start[i], temp_adj, beg_pos[temp_adj]);
                // }
                // if(shared_adj_degree[i] != beg_pos[temp_adj + 1] - shared_adj_start[i]) {
                //     printf("shared_adj_degree[%d] = %d, beg_pos[%d + 1] - shared_adj_start[%d] = %d\n", i, shared_adj_degree[i], temp_adj,
                //     temp_adj, beg_pos[temp_adj + 1] - shared_adj_start[i]);
                // }

                // shared_src[i] = temp_src;
                // shared_adj_start[i] = beg_pos[temp_adj];
                // shared_adj_degree[i] = beg_pos[temp_adj + 1] - shared_adj_start[i];

                int bin = (temp_src + temp_adj) & GroupTC_HASH_V2_GROUP_MODULO;
                int index = atomicAdd(&bin_count[bin], 1);

                if (index < GroupTC_HASH_V2_shared_GROUP_BUCKET_SIZE) {
                    shared_partition[index * GroupTC_HASH_V2_group_bucketnum + bin] = temp_adj;
                } else if (index < GroupTC_HASH_V2_GROUP_BUCKET_SIZE) {
                    index = index - GroupTC_HASH_V2_shared_GROUP_BUCKET_SIZE;
                    partition[index * GroupTC_HASH_V2_group_bucketnum + bin + BIN_START] = temp_adj;
                }
            }
            __syncthreads();

            if (1) {
                // compute 2 hop neighbors
                int now = threadIdx.x / GroupTC_HASH_V2_Group_SUBWARP_SIZE;
                int workid = threadIdx.x % GroupTC_HASH_V2_Group_SUBWARP_SIZE;

                while (now < group_size) {
                    int neighbor_degree = shared_adj_degree[now];
                    while (now < group_size && workid >= neighbor_degree) {
                        now += GroupTC_HASH_V2_BLOCK_SIZE / GroupTC_HASH_V2_Group_SUBWARP_SIZE;
                        workid -= neighbor_degree;
                        neighbor_degree = shared_adj_degree[now];
                    }

                    if (now < group_size) {
                        int temp_src = shared_src[now];
                        int temp_adj = adj_list[shared_adj_start[now] + workid];
                        int bin = (temp_src + temp_adj) & GroupTC_HASH_V2_GROUP_MODULO;
                        int len = bin_count[bin];

                        P_counter += len > 0 && shared_partition[bin + GroupTC_HASH_V2_group_bucketnum * 0] == temp_adj;
                        P_counter += len > 1 && shared_partition[bin + GroupTC_HASH_V2_group_bucketnum * 1] == temp_adj;
                        P_counter += len > 2 && shared_partition[bin + GroupTC_HASH_V2_group_bucketnum * 2] == temp_adj;
                        P_counter += len > 3 && shared_partition[bin + GroupTC_HASH_V2_group_bucketnum * 3] == temp_adj;

                        if (len > GroupTC_HASH_V2_shared_GROUP_BUCKET_SIZE) {
                            P_counter += tc::approach::GroupTC_HASH_V2::linear_search_group(temp_adj, partition, len, bin, BIN_START);
                        }
                    }
                    workid += GroupTC_HASH_V2_Group_SUBWARP_SIZE;
                }
            }

            __syncthreads();
        }
    }

    GLOBAL_COUNT[blockIdx.x * blockDim.x + threadIdx.x] = P_counter;
}

void tc::approach::GroupTC_HASH_V2::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    int grid_size = 2048;
    int block_size = 1024;
    int chunk_size = 1;

    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    index_t *d_beg_pos = gpu_graph.beg_pos;
    vertex_t *d_src_list = gpu_graph.src_list;
    vertex_t *d_adj_list = gpu_graph.adj_list;

    index_t *h_beg_pos = (index_t *)malloc(sizeof(index_t) * (vertex_count + 1));
    HRR(hipMemcpy(h_beg_pos, gpu_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

    int warpfirstvertex = my_binary_search(vertex_count, GroupTC_HASH_V2_USE_CTA, h_beg_pos) + 1;
    int warpfirstedge = h_beg_pos[warpfirstvertex];
    int nocomputefirstvertex = my_binary_search(vertex_count, GroupTC_HASH_V2_USE_WARP, h_beg_pos) + 1;
    int nocomputefirstedge = h_beg_pos[nocomputefirstvertex];

    int T_Group = 32;
    int nowindex[3];
    nowindex[0] = chunk_size * grid_size * block_size / T_Group;
    nowindex[1] = chunk_size * grid_size;
    nowindex[2] = warpfirstvertex + chunk_size * (grid_size * block_size / T_Group);

    int *BIN_MEM;
    int *G_INDEX;
    unsigned long long *GLOBAL_COUNT;

    HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * grid_size * GroupTC_HASH_V2_block_bucketnum * GroupTC_HASH_V2_BLOCK_BUCKET_SIZE));
    HRR(hipMalloc((void **)&G_INDEX, sizeof(int) * 3));

    HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));

    double total_kernel_use = 0;
    double startKernel, ee = 0;
    int block_kernel_grid_size = min(max(warpfirstvertex, 1), grid_size);
    int group_kernel_grid_size = min((nocomputefirstedge - warpfirstedge) / (GroupTC_HASH_V2_EDGE_CHUNK * 10), grid_size);
    int kernel_grid_size = max(max(block_kernel_grid_size, group_kernel_grid_size), 320);

    uint64_t count;
    HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * kernel_grid_size * GroupTC_HASH_V2_BLOCK_SIZE));
    spdlog::info("kernel_grid_size {:d}", kernel_grid_size);

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemcpy(G_INDEX, &nowindex, sizeof(int) * 3, hipMemcpyHostToDevice));
        HRR(hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * kernel_grid_size * GroupTC_HASH_V2_BLOCK_SIZE));

        startKernel = wtime();
        tc::approach::GroupTC_HASH_V2::grouptc_hash_v2<64, GroupTC_HASH_V2_BLOCK_SIZE / 64, 1><<<kernel_grid_size, GroupTC_HASH_V2_BLOCK_SIZE>>>(
            d_src_list, d_adj_list, d_beg_pos, edge_count, vertex_count, BIN_MEM, GLOBAL_COUNT, T_Group, G_INDEX, warpfirstvertex, warpfirstedge,
            nocomputefirstvertex, nocomputefirstedge);
        HRR(hipDeviceSynchronize());
        thrust::device_ptr<unsigned long long> ptr(GLOBAL_COUNT);
        count = thrust::reduce(ptr, ptr + (kernel_grid_size * GroupTC_HASH_V2_BLOCK_SIZE));
        ee = wtime();

        total_kernel_use += ee - startKernel;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - startKernel > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, iteration_count, avg compute time/s,
    auto logger = spdlog::get("GroupTC-HASH-V2_file_logger");
    if (logger) {
        logger->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "GroupTC-HASH-V2", gpu_graph.input_dir, count, iteration_count, total_kernel_use / iteration_count);
    } else {
        spdlog::warn("Logger 'GroupTC-HASH-V2_file_logger' is not initialized.");
    }

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", count);

    free(h_beg_pos);
    HRR(hipFree(BIN_MEM));
    HRR(hipFree(GLOBAL_COUNT));
    HRR(hipFree(G_INDEX));
}

void tc::approach::GroupTC_HASH_V2::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "GroupTC-HASH-V2", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("GroupTC_HASH_V2 before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::GroupTC_HASH_V2::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("GroupTC_HASH_V2 after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after GroupTC_HASH_V2 runs.");
        }
    }
}
