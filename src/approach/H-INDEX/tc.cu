#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <iterator>
#include <queue>
#include <set>

#include "approach/H-INDEX/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

__device__ int tc::approach::H_INDEX::linear_search(int neighbor, int *partition1, int *bin_count, int bin, int BIN_OFFSET, int BIN_START,
                                                    int BUCKETS) {
    int len = bin_count[bin + BIN_OFFSET];
    // printf("\nPartStart: %d\n",BIN_START);
    int i = bin + BIN_START;
    int step = 0;
    while (step < len) {
        int test = partition1[i];
        // printf("Neighbor: %d, Test: %d\n",neighbor,test);
        if (test == neighbor) {
            return 1;
        } else {
            i += BUCKETS;
        }
        step += 1;
    }
    return 0;
}

__device__ int tc::approach::H_INDEX::max_count(int *bin_count, int start, int end, int len) {
    int max_count = bin_count[start];
    int min_count = bin_count[start];
    int zero_count = 0;
    for (int i = start; i < end; i++) {
        if (bin_count[i] > max_count) {
            max_count = bin_count[i];
        }
        if (bin_count[i] < min_count) {
            min_count = bin_count[i];
        }
        if (bin_count[i] == 0) {
            zero_count += 1;
        }
    }
    // printf("%d,%d,%d\n",zero_count,max_count,len);
    return max_count;
}

__global__ void tc::approach::H_INDEX::warp_hash_count(vertex_t *adj_list, index_t *beg_pos, vertex_t *edge_list, uint edge_count, uint vertex_count,
                                                       uint edge_list_count, int *partition, unsigned long long *GLOBAL_COUNT, long long E_START,
                                                       long long E_END, int device, int BUCKETS, int G_BUCKET_SIZE, int T_Group) {
    // Uncomment the lines below and change partition to Gpartition for using shared version
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int WARPSIZE = T_Group;
    int __shared__ bin_count[256 * 4];
    // int __shared__ partition[160*4];
    int PER_BLOCK_WARP = blockDim.x / WARPSIZE;
    int G_WARPID = tid / WARPSIZE;
    int WARPID = threadIdx.x / WARPSIZE;
    int __shared__ G_counter;
    G_counter = 0;
    int P_counter = 0;
    int BINsize = BUCKETS * G_BUCKET_SIZE;
    // int BINsize = BUCKETS*5;
    int BIN_START = G_WARPID * BINsize;
    // int BIN_START = WARPID*BINsize;
    long long i = G_WARPID * 2;
    long long RANGE = E_END - E_START;
    int BIN_OFFSET = WARPID * BUCKETS;
    // for(int i=0;i<edge_list_count; i+=2)
    // TODO: Static assignment to dynamic assignment of edges
    //  unsigned long long TT=0,HT=0,IT=0;
    //  unsigned long long __shared__ G_TT,G_HT,G_IT;
    //  G_TT=0,G_HT=0,G_IT=0;
    while (i < (RANGE)) {
        // if(threadIdx.x%32==0){printf("Warp:%d, G_WArp: %d,i: %d \n",WARPID,G_WARPID,i);}
        // if (device==1){printf("Device: %d, i: %d\n",device,i);}
        /* TODO: Divide edge list to multiple blocks*/
        // unsigned long long start_time=clock64();
        int destination = edge_list[i];
        int source = edge_list[i + 1];
        int N1_start = beg_pos[destination];
        int N1_end = beg_pos[destination + 1];
        int L1 = N1_end - N1_start;
        int N2_start = beg_pos[source];
        int N2_end = beg_pos[source + 1];
        int L2 = N2_end - N2_start;

        // if ((L1==0))
        // {
        // 	//printf("continue %d\n",i);
        // 	continue;
        // }
        // // N2 is for hashing and N1 is lookup
        if (L1 > L2) {
            int temp = N1_start;
            N1_start = N2_start;
            N2_start = temp;
            temp = N1_end;
            N1_end = N2_end;
            N2_end = temp;
            temp = L2;
            L2 = L1;
            L1 = temp;
        }

        // unsigned long long hash_start=clock64();
        int id = threadIdx.x % WARPSIZE + BIN_OFFSET;
        int end = BIN_OFFSET + BUCKETS;
        // if(threadIdx.x%32==0){printf("End: %d\n",end);}
        //  We can remove this line

        __syncwarp();
        while (id < (end)) {
            bin_count[id] = 0;
            // printf("BIN: %d\n",id);
            id += WARPSIZE;
        }
        int start = threadIdx.x % WARPSIZE + N2_start;
        // BIN_OFFSET is for count of number of element of each bin for all 4 warps

        __syncwarp();
        // Hash one list
        while (start < N2_end) {
            int temp = adj_list[start];
            int bin = temp % BUCKETS;
            int index = atomicAdd(&bin_count[bin + BIN_OFFSET], 1);
            partition[index * BUCKETS + bin + BIN_START] = temp;
            //{printf("thread: %d,warp:%d, write: %d bin %d, index %d  at: %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}
            start += WARPSIZE;
        }
        __syncwarp();
        // unsigned long long hash_time=clock64()-hash_start;
        // int max_len_collision= max_count(bin_count,BIN_OFFSET,BIN_OFFSET+BUCKETS,L2);

        // unsigned long long intersection_start=clock64();
        start = threadIdx.x % WARPSIZE + N1_start;
        int count;
        // if(threadIdx.x==32){printf("start: %d, BIN_OFFSET: %d\n",start,BIN_OFFSET);}
        // P_counter=0;
        while (start < N1_end) {
            count = 0;
            int neighbor = adj_list[start];
            int bin = neighbor % BUCKETS;
            count = tc::approach::H_INDEX::linear_search(neighbor, partition, bin_count, bin, BIN_OFFSET, BIN_START, BUCKETS);
            P_counter += count;
            start += WARPSIZE;
            // printf("Tid: %d, Search:%d\n",threadIdx.x,neighbor);
        }
        // atomicAdd(&GLOBAL_COUNT[0],P_counter);

        __syncwarp();
        // unsigned long long intersection_time=clock64()-intersection_start;
        // if(threadIdx.x%32==0){printf("I: %d, Start:%d, End:%d, Count:%d\n",i,vertex,vertex1,G_counter);}
        i += gridDim.x * PER_BLOCK_WARP * 2;
        // unsigned long long total_time=clock64()-start_time;
        // if(threadIdx.x%32==0){
        // 	// printf("%d %d %d\n",total_time, hash_time, intersection_time);
        // 	TT+=total_time;
        // 	HT+=hash_time;
        // 	IT+=intersection_time;
        // }
    }
    atomicAdd(&G_counter, P_counter);
    // atomicAdd(&G_HT,HT);
    // atomicAdd(&G_TT,TT);
    // atomicAdd(&G_IT,IT);
    __syncthreads();
    if (threadIdx.x == 0) {
        // printf("%d\n",G_TT);
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
        // atomicAdd(&GLOBAL_COUNT[1],G_TT);
        // atomicAdd(&GLOBAL_COUNT[2],G_HT);
        // atomicAdd(&GLOBAL_COUNT[3],G_IT);
    }

    // if(threadIdx.x==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}
}

__global__ void tc::approach::H_INDEX::CTA_hash_count(vertex_t *adj_list, index_t *beg_pos, vertex_t *edge_list, uint edge_count, uint vertex_count,
                                                      uint edge_list_count, int *partition, unsigned long long *GLOBAL_COUNT, int E_START, int E_END,
                                                      int device, int BUCKETS, int BUCKET_SIZE, int T_Group) {
    int __shared__ bin_count[512];
    int G_WARPID = blockIdx.x;
    int __shared__ G_counter;
    G_counter = 0;
    int P_counter = 0;
    int BINsize = BUCKETS * BUCKET_SIZE;
    int i = G_WARPID * 2;
    int RANGE = E_END - E_START;
    int BIN_START = G_WARPID * BINsize;
    // for(int i=0;i<edge_list_count; i+=2)
    // TODO: Static assignment to dynamic assignment of edges

    while (i < (RANGE)) {
        /* TODO: Divide edge list to multiple blocks*/
        int destination = edge_list[i];
        int source = edge_list[i + 1];
        int N1_start = beg_pos[destination];
        int N1_end = beg_pos[destination + 1];
        int L1 = N1_end - N1_start;
        int N2_start = beg_pos[source];
        int N2_end = beg_pos[source + 1];
        int L2 = N2_end - N2_start;

        // N2 is for hashing and N1 is lookup
        if (L1 > L2) {
            int temp = N1_start;
            N1_start = N2_start;
            N2_start = temp;
            temp = N1_end;
            N1_end = N2_end;
            N2_end = temp;
            temp = L2;
            L2 = L1;
            L1 = temp;
        }

        int id = threadIdx.x;
        int end = BUCKETS;

        while (id < (end)) {
            bin_count[id] = 0;
            id += blockDim.x;
        }
        __syncthreads();
        int start = threadIdx.x + N2_start;

        // Hash one list
        while (start < N2_end) {
            int temp = adj_list[start];
            int bin = temp % BUCKETS;
            int index = atomicAdd(&bin_count[bin], 1);
            partition[index * BUCKETS + bin + BIN_START] = temp;
            //{printf("thread: %d,warp:%d, write: %d bin %d, index %d  at: %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}
            start += blockDim.x;
        }
        __syncthreads();
        start = threadIdx.x + N1_start;
        int count;
        // if(threadIdx.x==32){printf("start: %d, BIN_OFFSET: %d\n",start,BIN_OFFSET);}
        // P_counter=0;
        while (start < N1_end) {
            count = 0;
            int neighbor = adj_list[start];
            int bin = neighbor % BUCKETS;
            count = tc::approach::H_INDEX::linear_search(neighbor, partition, bin_count, bin, 0, BIN_START, BUCKETS);
            P_counter += count;
            start += blockDim.x;
            // printf("Tid: %d, Search:%d\n",threadIdx.x,neighbor);
        }
        // atomicAdd(&GLOBAL_COUNT[0], P_counter);

        // if(threadIdx.x%32==0){printf("I: %d, Start:%d, End:%d, Count:%d\n",i,vertex,vertex1,G_counter);}
        i += gridDim.x * 2;
    }
    atomicAdd(&G_counter, P_counter);
    __syncthreads();
    if (threadIdx.x == 0) {
        atomicAdd(&GLOBAL_COUNT[0], G_counter);
        // atomicAdd(&GLOBAL_COUNT[0], max_len_collision);
    }

    // if(threadIdx.x==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}
}

void tc::approach::H_INDEX::gpu_run(INIReader &config, GPUGraph &gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int grid_size = config.GetInteger(key_space, "grid_size", 1024);
    int block_size = config.GetInteger(key_space, "block_size", 1024);
    int BUCKETS = config.GetInteger(key_space, "buckets", 32);
    int select_thread_group = config.GetInteger(key_space, "select_thread_group", 0);
    int select_partition = config.GetInteger(key_space, "select_partition", 0);
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    int T_Group = 32;
    int BUCKET_SIZE = 1000;
    int PER_BLOCK_WARP = block_size / T_Group;
    int total = grid_size * PER_BLOCK_WARP * BUCKETS * BUCKET_SIZE;
    unsigned long long *counter = (unsigned long long *)malloc(sizeof(unsigned long long) * 10);

    uint vertex_count = gpu_graph.vertex_count;
    uint edge_count = gpu_graph.edge_count;
    index_t edge_list_count = edge_count * 2;

    int *BIN_MEM;
    unsigned long long *GLOBAL_COUNT;
    index_t *d_beg_pos = gpu_graph.beg_pos;
    vertex_t *d_adj_list = gpu_graph.adj_list;
    vertex_t *d_edge_list;

    int zip_block_size = 1024;
    int zip_edge_grid_size = (edge_count - 1) / zip_block_size + 1;

    HRR(hipMalloc((void **)&GLOBAL_COUNT, sizeof(unsigned long long) * 10));
    HRR(hipMalloc((void **)&BIN_MEM, sizeof(int) * total));
    HRR(hipMalloc((void **)&d_edge_list, (size_t)sizeof(vertex_t) * edge_list_count));

    cuda_graph_comm::zip_edge<<<zip_edge_grid_size, zip_block_size>>>(edge_count, vertex_count, d_edge_list, gpu_graph.src_list, d_adj_list);
    HRR(hipDeviceSynchronize());

    double total_kernel_use = 0;
    double startKernel, ee;
    for (int i = 0; i < iteration_count; i++) {
        hipMemset(GLOBAL_COUNT, 0, sizeof(unsigned long long) * 10);
        startKernel = wtime();
        if (select_thread_group == 1) {
            tc::approach::H_INDEX::CTA_hash_count<<<grid_size, block_size>>>(d_adj_list, d_beg_pos, d_edge_list, edge_count, vertex_count,
                                                                             edge_list_count, BIN_MEM, GLOBAL_COUNT, 0, edge_count * 2, 0, BUCKETS,
                                                                             BUCKET_SIZE, T_Group);
            HRR(hipDeviceSynchronize());
        } else {
            tc::approach::H_INDEX::warp_hash_count<<<grid_size, block_size>>>(d_adj_list, d_beg_pos, d_edge_list, edge_count, vertex_count,
                                                                              edge_list_count, BIN_MEM, GLOBAL_COUNT, 0, edge_count * 2, 0, BUCKETS,
                                                                              BUCKET_SIZE, T_Group);
            HRR(hipDeviceSynchronize());
        }
        HRR(hipMemcpy(counter, GLOBAL_COUNT, sizeof(unsigned long long) * 10, hipMemcpyDeviceToHost));
        ee = wtime();
        total_kernel_use += ee - startKernel;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - startKernel > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, iteration_count, avg compute time/s,
    spdlog::get("H-INDEX_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "H-INDEX", gpu_graph.input_dir, counter[0], iteration_count, total_kernel_use / iteration_count);

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", counter[0]);

    HRR(hipFree(BIN_MEM));
    HRR(hipFree(GLOBAL_COUNT));
    HRR(hipFree(d_edge_list));
}

void tc::approach::H_INDEX::start_up(INIReader &config, GPUGraph &gpu_graph, int argc, char **argv) {
    bool run = config.GetBoolean("comm", "H-INDEX", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("H_INDEX before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::H_INDEX::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("H_INDEX after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after H_INDEX runs.");
        }
    }
}
