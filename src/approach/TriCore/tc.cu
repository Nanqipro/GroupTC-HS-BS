#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "approach/TriCore/tc.h"
#include "comm/comm.h"
#include "comm/constant_comm.h"
#include "comm/cuda_comm.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

__global__ void tc::approach::TriCore::warp_binary_kernel(uint edge_count, vertex_t* src_list, vertex_t* adj_list, index_t* beg_pos,
                                                          unsigned long long* results) {
    // phase 1, partition
    uint64_t count = 0;
    __shared__ vertex_t local[TriCore_BLOCKSIZE];

    vertex_t i = threadIdx.x % 32;
    vertex_t p = threadIdx.x / 32;
    for (vertex_t tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32; tid < edge_count; tid += blockDim.x * gridDim.x / 32) {
        vertex_t node_m = src_list[tid];
        vertex_t node_n = adj_list[tid];

        vertex_t degree_m = beg_pos[node_m + 1] - beg_pos[node_m];
        vertex_t degree_n = beg_pos[node_n + 1] - beg_pos[node_n];
        vertex_t* a = adj_list + beg_pos[node_m];
        vertex_t* b = adj_list + beg_pos[node_n];
        if (degree_m < degree_n) {
            vertex_t temp = degree_m;
            degree_m = degree_n;
            degree_n = temp;
            vertex_t* aa = a;
            a = b;
            b = aa;
        }

        // initial cache
        local[p * 32 + i] = a[i * degree_m / 32];
        __syncthreads();

        // search
        vertex_t j = i;
        while (j < degree_n) {
            vertex_t X = b[j];
            vertex_t Y;
            // phase 1: cache
            int32_t bot = 0;
            int32_t top = 32;
            int32_t r;
            while (top > bot + 1) {
                r = (top + bot) / 2;
                Y = local[p * 32 + r];
                if (X == Y) {
                    count++;
                    bot = top + 32;
                }
                if (X < Y) {
                    top = r;
                }
                if (X > Y) {
                    bot = r;
                }
            }
            // phase 2
            bot = bot * degree_m / 32;
            top = top * degree_m / 32 - 1;
            while (top >= bot) {
                r = (top + bot) / 2;
                Y = a[r];
                if (X == Y) {
                    count++;
                }
                if (X <= Y) {
                    top = r - 1;
                }
                if (X >= Y) {
                    bot = r + 1;
                }
            }
            j += 32;
        }
        __syncthreads();
    }
    results[blockDim.x * blockIdx.x + threadIdx.x] = count;
}

void tc::approach::TriCore::gpu_run(INIReader& config, GPUGraph& gpu_graph, std::string key_space) {
    std::string file = gpu_graph.input_dir;
    int iteration_count = config.GetInteger(key_space, "iteration_count", 1);
    spdlog::info("Run algorithm {}", key_space);
    spdlog::info("Dataset {}", file);
    spdlog::info("Number of nodes: {0}, number of edges: {1}", gpu_graph.vertex_count, gpu_graph.edge_count);
    int device = config.GetInteger(key_space, "device", 1);
    HRR(hipSetDevice(device));

    vertex_t* d_src = gpu_graph.src_list;
    vertex_t* d_adj = gpu_graph.adj_list;
    index_t* d_beg_pos = gpu_graph.beg_pos;
    vertex_t edge_count = gpu_graph.edge_count;

    int grid_size = 1048576;
    int block_size = TriCore_BLOCKSIZE;
    double t_start, total_kernel_use = 0;
    uint64_t count;

    unsigned long long* d_results;
    HRR(hipMalloc(&d_results, grid_size * block_size * sizeof(unsigned long long)));

    for (int i = 0; i < iteration_count; i++) {
        HRR(hipMemset(d_results, 0, grid_size * block_size * sizeof(unsigned long long)));

        cuda_graph_comm::check_array("d_src", d_src, edge_count, edge_count - 10, edge_count);
        cuda_graph_comm::check_array("d_adj", d_adj, edge_count, edge_count - 10, edge_count);
        cuda_graph_comm::check_array("d_beg_pos", d_beg_pos, 10000, 0, 10);
        t_start = wtime();

        tc::approach::TriCore::warp_binary_kernel<<<grid_size, block_size>>>(edge_count, d_src, d_adj, d_beg_pos, d_results);

        HRR(hipDeviceSynchronize());
        thrust::device_ptr<unsigned long long> ptr(d_results);
        count = thrust::reduce(ptr, ptr + (grid_size * block_size));

        double ee = wtime();
        total_kernel_use += ee - t_start;
        if (i == 0) {
            spdlog::info("Iter 0, kernel use {:.6f} s", total_kernel_use);
            if (ee - t_start > 0.1 && iteration_count != 1) {
                iteration_count = 10;
            }
        }
    }

    // algorithm, dataset, triangle_count, iteration_count, avg kernel time/s
    spdlog::get("TriCore_file_logger")
        ->info("{0}\t{1}\t{2}\t{3}\t{4:.6f}", "TriCore", gpu_graph.input_dir, count, iteration_count, total_kernel_use / iteration_count);

    spdlog::info("Iter {0}, avg kernel use {1:.6f} s", iteration_count, total_kernel_use / iteration_count);
    spdlog::info("Triangle count {:d}", count);

    HRR(hipFree(d_results));
}

void tc::approach::TriCore::start_up(INIReader& config, GPUGraph& gpu_graph, int argc, char** argv) {
    bool run = config.GetBoolean("comm", "TriCore", false);
    if (run) {
        size_t free_byte, total_byte, available_byte;
        HRR(hipMemGetInfo(&free_byte, &total_byte));
        available_byte = total_byte - free_byte;
        spdlog::debug("TriCore before compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

        tc::approach::TriCore::gpu_run(config, gpu_graph);

        HRR(hipMemGetInfo(&free_byte, &total_byte));
        spdlog::debug("TriCore after compute, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
        if (available_byte != total_byte - free_byte) {
            spdlog::warn("There is GPU memory that is not freed after TriCore runs.");
        }
    }
}
