#include <algorithm>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "comm/constant_comm.h"
#include "datatransfer/data_transfer.h"
#include "graph/cpu_graph.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

bool DataTransfer::degree_less(vertex &a, vertex &b) { return a.degree < b.degree; }

bool DataTransfer::degree_greather(vertex &a, vertex &b) { return a.degree > b.degree; }

bool DataTransfer::edge_less(edge &a, edge &b) { return a.u < b.u || (a.u == b.u && a.v < b.v); }

bool DataTransfer::edge_greather(edge &a, edge &b) { return a.u > b.u || (a.u == b.u && a.v > b.v); }

DataTransfer::DataTransfer(std::string file, CPUGraph *graph) {
    init(file, graph);
}

DataTransfer::DataTransfer() {}

void DataTransfer::init(std::string file, CPUGraph *graph) {
    input_file = file;
    h_graph = graph;
    d_graph.init(*graph);
}

bool DataTransfer::check_init() {
    if (input_file.empty()) {
        spdlog::error("Input file is empty, please init the data transfer first.");
        return false;
    }
    return true;
}

int DataTransfer::compute_max_degree() {
    index_t *offset_arr = (index_t *)malloc(sizeof(index_t) * (h_graph->vertex_count + 1));
    HRR(hipMemcpy(offset_arr, d_graph.beg_pos, sizeof(index_t) * (h_graph->vertex_count + 1), hipMemcpyDeviceToHost));
    uint vertex_count = h_graph->vertex_count;

    int max_degre = 0;
    for (uint i = 1; i <= vertex_count; i++) {
        if (offset_arr[i] - offset_arr[i - 1] > max_degre) {
            max_degre = offset_arr[i] - offset_arr[i - 1];
        }
    }
    free(offset_arr);
    return max_degre;
}
