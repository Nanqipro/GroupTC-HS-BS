#include <algorithm>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "comm/constant_comm.h"
#include "datatransfer/data_transfer.h"
#include "graph/cpu_graph.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/spdlog.h"

bool DataTransfer::degree_less(vertex &a, vertex &b) { return a.degree < b.degree; }

bool DataTransfer::degree_greather(vertex &a, vertex &b) { return a.degree > b.degree; }

bool DataTransfer::edge_less(edge &a, edge &b) { return a.u < b.u || (a.u == b.u && a.v < b.v); }

bool DataTransfer::edge_greather(edge &a, edge &b) { return a.u > b.u || (a.u == b.u && a.v > b.v); }

DataTransfer::DataTransfer(std::string file, CPUGraph *graph) : input_file(file), d_graph(*graph) { h_graph = graph; }

int DataTransfer::compute_max_degree() {
    index_t *offset_arr = (index_t *)malloc(sizeof(index_t) * (h_graph->vertex_count + 1));
    HRR(hipMemcpy(offset_arr, d_graph.beg_pos, sizeof(index_t) * (h_graph->vertex_count + 1), hipMemcpyDeviceToHost));
    uint vertex_count = h_graph->vertex_count;

    int max_degre = 0;
    for (uint i = 1; i <= vertex_count; i++) {
        if (offset_arr[i] - offset_arr[i - 1] > max_degre) {
            max_degre = offset_arr[i] - offset_arr[i - 1];
        }
    }
    free(offset_arr);
    return max_degre;
}
