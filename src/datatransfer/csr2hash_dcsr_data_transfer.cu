#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "comm/config_comm.h"
#include "comm/constant_comm.h"
#include "datatransfer/csr2hash_dcsr_data_transfer.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/sinks/basic_file_sink.h"
#include "spdlog/spdlog.h"

void Csr2HashDcsrDataTransfer::transfer() {

    if (!check_init()) {
        return;
    }

    uint *d_degree_arr;
    vertex_t *d_id_arr;
    vertex_t *d_id_map_arr;

    uint vertex_count = d_graph.vertex_count;
    uint edge_count = d_graph.edge_count;
    vertex_t *d_src_arr = d_graph.src_list;
    vertex_t *d_adj_arr = d_graph.adj_list;
    index_t *d_offset_arr = d_graph.beg_pos;

    size_t size_degree_arr = sizeof(uint) * vertex_count;

    HRR(hipMalloc((void **)&d_degree_arr, size_degree_arr));
    HRR(hipMalloc((void **)&d_id_arr, size_degree_arr));
    HRR(hipMalloc((void **)&d_id_map_arr, size_degree_arr));

    HRR(hipMemset(d_degree_arr, 0, size_degree_arr));

    int block_size = 1024;
    // int vertex_grid_size = (vertex_count - 1) / block_size + 1;
    int edge_grid_size = (edge_count - 1) / block_size + 1;

    int max_degree = compute_max_degree();

    spdlog::info("CSR2TrustDCSR graph transfer start, graph max degree is {}", max_degree);


    double t_start = wtime();

    int iterations = config_comm::cPreprocessingIterations;
    for (int k = 0; k < iterations; k++) {
        cuda_graph_comm::cal_degree<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_src_arr, d_adj_arr);
        // HRR(hipDeviceSynchronize());

        cuda_graph_comm::redirect_edge<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_src_arr, d_adj_arr);
        // HRR(hipDeviceSynchronize());

        // HRR(hipMemset(d_degree_arr, 0, size_degree_arr));
        // cuda_graph_comm::cal_out_degree_by_src<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_src_arr);
        // HRR(hipDeviceSynchronize());

        // cuda_graph_comm::record_id_and_part_graph_by_degree<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_id_arr, d_degree_arr);
        // HRR(hipDeviceSynchronize());

        // thrust::device_ptr<vertex_t> d_id_ptr((vertex_t *)d_id_arr);
        // thrust::sort_by_key(d_degree_arr, d_degree_arr + vertex_count, d_id_ptr);

        // cuda_graph_comm::map_id<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_id_arr, d_id_map_arr);
        // HRR(hipDeviceSynchronize());

        // cuda_graph_comm::reassign_id<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_id_map_arr, d_src_arr, d_adj_arr);
        // HRR(hipDeviceSynchronize());

        thrust::device_ptr<vertex_t> d_src_ptr((vertex_t *)d_src_arr);
        thrust::device_ptr<vertex_t> d_dst_ptr((vertex_t *)d_adj_arr);
        thrust::sort_by_key(d_src_ptr, d_src_ptr + edge_count, d_adj_arr);

        cuda_graph_comm::recal_offset<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_src_arr, d_offset_arr);
        // HRR(hipDeviceSynchronize());
    }
    HRR(hipDeviceSynchronize());

    double t_end = wtime();

    // algorithm, dataset, iterations, avg compute time/s,
    spdlog::get("csr2hash_dcsr_file_logger")
        ->info("{0}\t{1}\t{2}\t{3:.6f}", "csr2hash_dcsr", d_graph.input_dir, iterations, (t_end - t_start) / iterations);

    spdlog::info("Iterate {0} times, avg time consumption {1:.6f} s", iterations, (t_end - t_start) / iterations);

    cuda_graph_comm::check_array("d_src_arr", d_src_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_adj_arr", d_adj_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_offset_arr", d_offset_arr, vertex_count + 1, 0, 10);

    max_degree = compute_max_degree();
    spdlog::info("CSR2HashDCSR graph transfer finished, graph max degree is {}", max_degree);

    HRR(hipFree(d_degree_arr));
    HRR(hipFree(d_id_arr));
    HRR(hipFree(d_id_map_arr));
}
