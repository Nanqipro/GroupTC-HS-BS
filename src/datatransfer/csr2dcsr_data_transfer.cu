#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "comm/config_comm.h"
#include "comm/constant_comm.h"
#include "datatransfer/csr2dcsr_data_transfer.h"
#include "graph/cuda_graph_comm.h"
#include "spdlog/sinks/basic_file_sink.h"
#include "spdlog/spdlog.h"

void Csr2DcsrDataTransfer::transfer() {

    if (!check_init()) {
        return;
    }

    uint* d_degree_arr;
    vertex_t* d_edge_arr;

    uint vertex_count = d_graph.vertex_count;
    uint edge_count = d_graph.edge_count;
    vertex_t* d_src_arr = d_graph.src_list;
    vertex_t* d_adj_arr = d_graph.adj_list;
    index_t* d_offset_arr = d_graph.beg_pos;

    size_t size_degree_arr = sizeof(uint) * vertex_count;
    size_t size_src_arr = sizeof(vertex_t) * (edge_count + 10);
    size_t size_edge_arr = size_src_arr * 2;

    HRR(hipMalloc((void**)&d_degree_arr, size_degree_arr));
    HRR(hipMalloc((void**)&d_edge_arr, size_edge_arr));

    HRR(hipMemset(d_degree_arr, 0, size_degree_arr));

    int block_size = 1024;
    int grid_size = (edge_count - 1) / block_size + 1;

    int max_degree = compute_max_degree();

    spdlog::info("CSR2DCSR graph transfer start, graph max degree is {}", max_degree);

    double t_start = wtime();

    int iterations = config_comm::cPreprocessingIterations;
    for (int k = 0; k < iterations; k++) {
        cuda_graph_comm::cal_degree_and_zip_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_edge_arr, d_src_arr, d_adj_arr);
        // HRR(hipDeviceSynchronize());

        cuda_graph_comm::redirect_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_degree_arr, d_edge_arr);
        // HRR(hipDeviceSynchronize());

        vertex_t* h_src_arr;
        vertex_t* h_adj_arr;

        // The GPU space required for sorting is insufficient, so src_list and adj_list need to be transferred to CPU space first.
        if (edge_count > constant_comm::kDataTransferMaxEdgeCount) {
            spdlog::info("Csr2DcsrDataTransfer's sorting requires more GPU space, so src_list and adj_list are transferred to CPU space.");
            h_src_arr = (vertex_t*)malloc(size_src_arr);
            h_adj_arr = (vertex_t*)malloc(size_src_arr);
            HRR(hipMemcpy(h_src_arr, d_src_arr, size_src_arr, hipMemcpyDeviceToHost));
            HRR(hipMemcpy(h_adj_arr, d_adj_arr, size_src_arr, hipMemcpyDeviceToHost));
            HRR(hipFree(d_src_arr));
            HRR(hipFree(d_adj_arr));
        }

        thrust::device_ptr<uint64_t> sort_ptr((uint64_t*)d_edge_arr);
        thrust::sort(sort_ptr, sort_ptr + edge_count);

        // After sorting, src_list and adj_list are transferred back to GPU space.
        if (edge_count > constant_comm::kDataTransferMaxEdgeCount) {
            spdlog::info("Csr2DcsrDataTransfer's sorting is completed, src_list and adj_list are transferred back to the GPU space.");
            HRR(hipMalloc(&d_src_arr, size_src_arr));
            HRR(hipMalloc(&d_adj_arr, size_src_arr));
            HRR(hipMemcpy(d_src_arr, h_src_arr, size_src_arr, hipMemcpyHostToDevice));
            HRR(hipMemcpy(d_adj_arr, h_adj_arr, size_src_arr, hipMemcpyHostToDevice));
            d_graph.src_list = d_src_arr;
            d_graph.adj_list = d_adj_arr;
            free(h_src_arr);
            free(h_adj_arr);
        }

        cuda_graph_comm::unzip_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_edge_arr, d_src_arr, d_adj_arr);
        // HRR(hipDeviceSynchronize());

        cuda_graph_comm::recal_offset<<<grid_size, block_size>>>(edge_count, vertex_count, d_src_arr, d_offset_arr);
        // HRR(hipDeviceSynchronize());
    }
    HRR(hipDeviceSynchronize());
    double t_end = wtime();

    // algorithm, dataset, iterations, avg compute time/s,
    spdlog::get("csr2dcsr_file_logger")->info("{0}\t{1}\t{2}\t{3:.6f}", "csr2dcsr", d_graph.input_dir, iterations, (t_end - t_start) / iterations);

    spdlog::info("Iterate {0} times, avg time consumption {1:.6f} s", iterations, (t_end - t_start) / iterations);

    cuda_graph_comm::check_array("d_src_arr", d_graph.src_list, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_adj_arr", d_adj_arr, edge_count, 0, 10);
    cuda_graph_comm::check_array("d_offset_arr", d_offset_arr, vertex_count + 1, 0, 10);

    max_degree = compute_max_degree();
    spdlog::info("CSR2DCSR graph transfer finished, graph max degree is {}", max_degree);

    HRR(hipFree(d_degree_arr));
    HRR(hipFree(d_edge_arr));
}
