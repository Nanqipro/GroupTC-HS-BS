#include <unistd.h>

#include <string>

#include "main.h"

int main(int argc, char** argv) {
    std::string program_dir = get_program_dir();
    std::string config_file = program_dir + "/../config/config.ini";
    std::string config_dataset = "";

    // 存储命令行参数
    std::vector<std::string> args(argv, argv + argc);

    // 遍历命令行参数
    for (int i = 1; i < argc; ++i) {  // 从1开始，跳过程序名称
        std::string arg = args[i];
        if (arg.find("-config=") != std::string::npos) {
            config_file = arg.substr(8);
            spdlog::info("Config file is {}", config_file);
        }
        if (arg.find("-dataset=") != std::string::npos) {
            config_dataset = arg.substr(9);
            spdlog::info("Config_dataset file is {}", config_dataset);
        }
    }

    INIReader config(config_file);
    if (config.ParseError() < 0) {
        spdlog::info("Can't load {}", config_file);
        return 1;
    }

    // set comm config
    std::string dataset_file_path = config.Get("comm", "dataset_file_path", "UNKNOWN");
    std::string log_file_path = config.Get("comm", "log_file_path", "UNKNOWN");
    std::string str_datasets_1 = config.Get("comm", "datasets_1", "");
    std::string str_datasets_2 = config.Get("comm", "datasets_2", "");
    std::string log_level_str = config.Get("comm", "log_level", "info");
    spdlog::level::level_enum log_level = switch_log_level(log_level_str);
    spdlog::set_level(log_level);

    config_comm::cPreprocessingIterations = config.GetInteger("comm", "preprocessing_iterations", 1);

    int device = config.GetInteger("comm", "device", 0);
    HRR(hipSetDevice(device));
    spdlog::info("Use device {}", device);

    // init loggers
    init_loggers(log_file_path, config);

    std::vector<std::string> datasets_1 = get_datasets(str_datasets_1);
    std::vector<std::string> datasets_2 = get_datasets(str_datasets_2);
    std::vector<std::string> datasets = merge_vectors(datasets_1, datasets_2);

    for (auto dataset : datasets) {
        std::string input_file = dataset_file_path + dataset;
        if (config_dataset != "" && dataset != config_dataset) {
            continue;
        }

        // read dataset
        CPUGraph cpu_graph(input_file);
        if (cpu_graph.vertex_count <= 0 || cpu_graph.edge_count <= 0) {
            spdlog::warn("Invalid graph data, process next graph ...");
            continue;
        }

        if (cpu_graph.edge_count > constant_comm::kMaxGraphEdgeCount) {
            spdlog::info("Input graph is too large! Process next graph ...");
            continue;
        }

        // GPU warm up
        {
            int* d_warmup;
            size_t warmup_size = 1024 * 1024 * 1024;  // 1GB
            HRR(hipMalloc((void**)&d_warmup, warmup_size));
            HRR(hipMemset(d_warmup, 0, warmup_size));

            dim3 grid(1024), block(1024);
            cuda_graph_comm::warm_up<<<grid, block>>>(d_warmup, warmup_size / sizeof(int));

            HRR(hipDeviceSynchronize());
            HRR(hipFree(d_warmup));
        }
        
        // preprocess datasets and run algorithms
        {
            Csr2DcsrDataTransfer cddt(input_file, &cpu_graph);
            cddt.transfer();
            GPUGraph& dcsr = cddt.d_graph;

            tc::approach::Polak::start_up(config, dcsr, argc, argv);
            tc::approach::TriCore::start_up(config, dcsr, argc, argv);
            tc::approach::H_INDEX::start_up(config, dcsr, argc, argv);
            tc::approach::Green::start_up(config, dcsr, argc, argv);
            tc::approach::Hu::start_up(config, dcsr, argc, argv);
            // tc::approach::GroupTC_BS::start_up(config, dcsr, argc, argv);
            tc::approach::Fox::start_up(config, dcsr, argc, argv);
        }

        {
            Csr2RidDcsrDataTransfer crdt(input_file, &cpu_graph);
            crdt.transfer();
            GPUGraph& riddcsr = crdt.d_graph;

            tc::approach::Bisson::start_up(config, riddcsr, argc, argv);
            tc::approach::GroupTC::start_up(config, riddcsr, argc, argv);
            tc::approach::GroupTC_OPT::start_up(config, riddcsr, argc, argv);
        }

        {
            Csr2TrustDcsrDataTransfer ctdt(input_file, &cpu_graph);
            ctdt.transfer();
            GPUGraph& trustdcsr = ctdt.d_graph;

            tc::approach::GroupTC_Cuckoo::start_up(config, trustdcsr, argc, argv);
            tc::approach::TRUST::start_up(config, trustdcsr, argc, argv);
        }
        flush_loggers();
    }

    spdlog::info("All input graphs have been processed, the program ends.");

    return 0;
}

std::string get_program_dir() {
    char buffer[1024];
    ssize_t len = readlink("/proc/self/exe", buffer, sizeof(buffer) - 1);
    if (len != -1) {
        buffer[len] = '\0';
        std::string fullPath(buffer);
        std::string::size_type pos = fullPath.find_last_of("/");
        std::string program_directory = fullPath.substr(0, pos);
        return program_directory;
    }
    return "";
}

std::vector<std::string> get_datasets(const std::string& str, const std::string& delimiter) {
    std::vector<std::string> datasets;
    size_t pos = 0;
    std::string dataset;
    std::string str_copy = str;
    while ((pos = str_copy.find(delimiter)) != std::string::npos) {
        dataset = str_copy.substr(0, pos);
        dataset.erase(std::remove(dataset.begin(), dataset.end(), ' '), dataset.end());
        if (!dataset.empty()) {
            datasets.push_back(dataset);
        }
        str_copy.erase(0, pos + delimiter.length());
    }
    str_copy.erase(std::remove(str_copy.begin(), str_copy.end(), ' '), str_copy.end());
    if (!str_copy.empty()) {
        datasets.push_back(str_copy);
    }
    return datasets;
}

std::vector<std::string> merge_vectors(const std::vector<std::string>& vec1, const std::vector<std::string>& vec2) {
    std::vector<std::string> merged_vec;

    merged_vec.insert(merged_vec.end(), vec1.begin(), vec1.end());
    merged_vec.insert(merged_vec.end(), vec2.begin(), vec2.end());

    return merged_vec;
}

void init_loggers(std::string log_file_path, INIReader& config) {
    try {
        // 定义所有logger名称和对应的文件路径
        auto make_logger_pair = [&](std::string name, bool is_preprocessing = false) {
            if (!config.GetBoolean("comm", name, true)) {
                return std::make_pair(std::string(""), std::string(""));
            }
            if (is_preprocessing) {
                return std::make_pair(name + "_preprocessing_file_logger", name + "/preprocessing_time_output.txt");
            } else {
                return std::make_pair(name + "_file_logger", name + "/time_output.txt");
            }
        };

        const std::vector<std::pair<std::string, std::string>> loggers = {
            // CSR转换相关logger
            make_logger_pair("csr2dcsr"), make_logger_pair("csr2rid_dcsr"), make_logger_pair("csr2trust_dcsr"),

            // 预处理相关logger
            make_logger_pair("Fox", true), make_logger_pair("Hu", true),

            // 算法性能相关logger
            make_logger_pair("Bisson"), make_logger_pair("Fox"), make_logger_pair("Green"), make_logger_pair("GroupTC"),
            make_logger_pair("GroupTC-HASH"), make_logger_pair("GroupTC-OPT"), make_logger_pair("H-INDEX"), make_logger_pair("Hu"),
            make_logger_pair("Polak"), make_logger_pair("TriCore"), make_logger_pair("TRUST"), make_logger_pair("GroupTC-HASH-V2"),
            make_logger_pair("GroupTC-Cuckoo"), make_logger_pair("GroupTC-HS")};

        // 批量初始化所有logger
        for (const auto& logger : loggers) {
            if (logger.first != "") {
                init_file_logger(logger.first, log_file_path + logger.second, spdlog::level::info);
            }
        }

    } catch (const spdlog::spdlog_ex& ex) {
        spdlog::error("Log init failed: {}", ex.what());
    }
}

void init_file_logger(std::string logger_name, std::string file, spdlog::level::level_enum log_level) {
    std::shared_ptr<spdlog::logger> logger = spdlog::basic_logger_mt(logger_name, file);
    logger->set_level(log_level);
}

void flush_loggers() {
    // 获取所有已注册的logger并刷新
    spdlog::apply_all([](std::shared_ptr<spdlog::logger> logger) { logger->flush(); });
}

spdlog::level::level_enum switch_log_level(std::string log_level_str) {
    static const std::unordered_map<std::string, spdlog::level::level_enum> level_map = {
        {"trace", spdlog::level::trace}, {"debug", spdlog::level::debug},       {"info", spdlog::level::info}, {"warn", spdlog::level::warn},
        {"err", spdlog::level::err},     {"critical", spdlog::level::critical}, {"off", spdlog::level::off}};

    auto it = level_map.find(log_level_str);
    return it != level_map.end() ? it->second : spdlog::level::info;
}
