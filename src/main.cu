#include <unistd.h>
#include <string>
#include "main.h"

// 主线函数
int main(int argc, char** argv) {
    std::string program_dir = get_program_dir();
    std::string config_file = program_dir + "/../config/config.ini";
    std::string config_dataset = "";

    // 存储命令行参数
    std::vector<std::string> args(argv, argv + argc);

    // 遍历命令行参数
    for (int i = 1; i < argc; ++i) {  // 从1开始，跳过程序名称
        std::string arg = args[i];
        if (arg.find("-config=") != std::string::npos) {
            config_file = arg.substr(8);
            spdlog::info("Config file is {}", config_file);
        }
        if (arg.find("-dataset=") != std::string::npos) {
            config_dataset = arg.substr(9);
            spdlog::info("Config_dataset file is {}", config_dataset);
        }
    }

    INIReader config(config_file);
    if (config.ParseError() < 0) {
        spdlog::info("Can't load {}", config_file);
        return 1;
    }

    // set comm config
    std::string dataset_file_path = config.Get("comm", "dataset_file_path", "UNKNOWN");
    std::string log_file_path = config.Get("comm", "log_file_path", "UNKNOWN");
    std::string str_datasets_1 = config.Get("comm", "datasets_1", "");
    std::string str_datasets_2 = config.Get("comm", "datasets_2", "");
    std::string log_level_str = config.Get("comm", "log_level", "info");
    spdlog::level::level_enum log_level = switch_log_level(log_level_str);
    spdlog::set_level(log_level);

    config_comm::cPreprocessingIterations = config.GetInteger("comm", "preprocessing_iterations", 1);

    int device = config.GetInteger("comm", "device", 0);
    HRR(hipSetDevice(device));
    spdlog::info("Use device {}", device);

    // init loggers
    init_loggers(log_file_path);

    std::vector<std::string> datasets_1 = get_datasets(str_datasets_1);
    std::vector<std::string> datasets_2 = get_datasets(str_datasets_2);
    std::vector<std::string> datasets = merge_vectors(datasets_1, datasets_2);

    for (auto dataset : datasets) {
        std::string input_file = dataset_file_path + dataset;
        if (config_dataset != "" && dataset != config_dataset) {
            continue;
        }

        // read dataset
        CPUGraph cpu_graph(input_file);

        if (cpu_graph.edge_count > constant_comm::kMaxGraphEdgeCount) {
            spdlog::info("Input graph is too large! Process next graph ...");
            continue;
        }

        // preprocess datasets and run algorithms
        {
            Csr2DcsrDataTransfer cddt(input_file, &cpu_graph);
            cddt.transfer();
            GPUGraph& dcsr = cddt.d_graph;

            tc::approach::Polak::start_up(config, dcsr, argc, argv);
            tc::approach::TriCore::start_up(config, dcsr, argc, argv);
            tc::approach::H_INDEX::start_up(config, dcsr, argc, argv);
            tc::approach::Green::start_up(config, dcsr, argc, argv);
            tc::approach::Hu::start_up(config, dcsr, argc, argv);
            // tc::approach::TC_Check::start_up(config, dcsr, argc, argv);
            // tc::approach::GroupTC::start_up(config, dcsr, argc, argv);
            // tc::approach::GroupTC_OPT::start_up(config, dcsr, argc, argv);
            tc::approach::Fox::start_up(config, dcsr, argc, argv);
        }

        {
            Csr2RidDcsrDataTransfer crdt(input_file, &cpu_graph);
            crdt.transfer();
            GPUGraph& riddcsr = crdt.d_graph;

            tc::approach::Bisson::start_up(config, riddcsr, argc, argv);
            tc::approach::GroupTC::start_up(config, riddcsr, argc, argv);
            tc::approach::TC_Check::start_up(config, riddcsr, argc, argv);
            tc::approach::GroupTC_OPT::start_up(config, riddcsr, argc, argv);
        }

        {
            Csr2TrustDcsrDataTransfer ctdt(input_file, &cpu_graph);
            ctdt.transfer();
            GPUGraph& trustdcsr = ctdt.d_graph;

            tc::approach::GroupTC_HASH::start_up(config, trustdcsr, argc, argv);
            tc::approach::TRUST::start_up(config, trustdcsr, argc, argv);
        }

        flush_loggers();
    }

    spdlog::info("All input graphs have been processed, the program ends.");

    return 0;
}

// 支线函数
int main1(int argc, char** argv) {
    std::string config_file = "../config/config.ini";
    if (argc > 1) {
        config_file = argv[1];
    }

    INIReader config(config_file);
    if (config.ParseError() < 0) {
        spdlog::error("Can't load {}", config_file);
        return 1;
    }
    // comm config
    std::string input_file = config.Get("comm", "dataset", "UNKNOWN");
    std::string log_level_str = config.Get("comm", "log_level", "info");
    int device = config.GetInteger("comm", "device", 0);
    HRR(hipSetDevice(device));

    size_t free_byte, total_byte;
    // HRR(hipMemGetInfo(&free_byte, &total_byte));
    // spdlog::info("{:.2f}", float(total_byte) / MEMORY_G);
    // spdlog::info("{:.2f}", float(free_byte) / MEMORY_G);
    // spdlog::info("{:.2f}", float(total_byte - free_byte) / MEMORY_G);

    spdlog::level::level_enum log_level = switch_log_level(log_level_str);
    spdlog::set_level(log_level);
    CPUGraph cpu_graph(input_file);
    // GPUGraph gpu_graph(cpu_graph);

    HRR(hipMemGetInfo(&free_byte, &total_byte));
    spdlog::info("{:.2f}", float(total_byte) / MEMORY_G);
    spdlog::info("{:.2f}", float(free_byte) / MEMORY_G);
    spdlog::info("{:.2f}", float(total_byte - free_byte) / MEMORY_G);

    index_t* arr;
    index_t* key_arr;
    uint edge_count = cpu_graph.edge_count;
    size_t max_sort_len = 1e9 * 1.3;
    spdlog::info("{0} {1} {2}", edge_count, max_sort_len, (size_t)sizeof(index_t) * max_sort_len);
    HRR(hipMalloc(&arr, (size_t)sizeof(index_t) * max_sort_len));
    HRR(hipMalloc(&key_arr, (size_t)sizeof(index_t) * max_sort_len));
    // HRR(hipMemcpy(arr, cpu_graph.adj_list, sizeof(index_t) * 5e8, hipMemcpyHostToDevice));

    HRR(hipMemGetInfo(&free_byte, &total_byte));
    spdlog::info("{:.2f}", float(total_byte - free_byte) / MEMORY_G);

    thrust::device_ptr<index_t> sort_ptr((index_t*)arr);
    thrust::device_ptr<index_t> key_ptr((index_t*)key_arr);
    // thrust::sort_by_key(key_ptr, key_ptr + max_sort_len, sort_ptr);
    thrust::sort(sort_ptr, sort_ptr + max_sort_len);

    // // spdlog::info("sort big arr start.");
    // // cuda_graph_comm::sort_big_arr(arr, edge_count);
    // // spdlog::info("sort big arr end.");

    // HRR(hipFree(arr));
    return 0;
}

std::string get_program_dir() {
    char buffer[1024];
    ssize_t len = readlink("/proc/self/exe", buffer, sizeof(buffer) - 1);
    if (len != -1) {
        buffer[len] = '\0';
        std::string fullPath(buffer);
        std::string::size_type pos = fullPath.find_last_of("/");
        std::string program_directory = fullPath.substr(0, pos);
        return program_directory;
    }
    return "";
}

std::vector<std::string> get_datasets(const std::string& str, const std::string& delimiter) {
    std::vector<std::string> datasets;
    size_t pos = 0;
    std::string dataset;
    std::string str_copy = str;
    while ((pos = str_copy.find(delimiter)) != std::string::npos) {
        dataset = str_copy.substr(0, pos);
        dataset.erase(std::remove(dataset.begin(), dataset.end(), ' '), dataset.end());
        if (!dataset.empty()) {
            datasets.push_back(dataset);
        }
        str_copy.erase(0, pos + delimiter.length());
    }
    str_copy.erase(std::remove(str_copy.begin(), str_copy.end(), ' '), str_copy.end());
    if (!str_copy.empty()) {
        datasets.push_back(str_copy);
    }
    return datasets;
}

std::vector<std::string> merge_vectors(const std::vector<std::string>& vec1, const std::vector<std::string>& vec2) {
    std::vector<std::string> merged_vec;

    merged_vec.insert(merged_vec.end(), vec1.begin(), vec1.end());
    merged_vec.insert(merged_vec.end(), vec2.begin(), vec2.end());

    return merged_vec;
}

void init_loggers(std::string log_file_path) {
    try {
        init_file_logger("csr2dcsr_file_logger", log_file_path + "csr2dcsr/time_output.txt", spdlog::level::info);
        init_file_logger("csr2rid_dcsr_file_logger", log_file_path + "csr2rid_dcsr/time_output.txt", spdlog::level::info);
        init_file_logger("csr2trust_dcsr_file_logger", log_file_path + "csr2trust_dcsr/time_output.txt", spdlog::level::info);

        init_file_logger("Fox_preprocessing_file_logger", log_file_path + "Fox/preprocessing_time_output.txt", spdlog::level::info);
        init_file_logger("Hu_preprocessing_file_logger", log_file_path + "Hu/preprocessing_time_output.txt", spdlog::level::info);

        init_file_logger("Bisson_file_logger", log_file_path + "Bisson/time_output.txt", spdlog::level::info);
        init_file_logger("Fox_file_logger", log_file_path + "Fox/time_output.txt", spdlog::level::info);
        init_file_logger("Green_file_logger", log_file_path + "Green/time_output.txt", spdlog::level::info);
        init_file_logger("GroupTC_file_logger", log_file_path + "GroupTC/time_output.txt", spdlog::level::info);
        init_file_logger("GroupTC-HASH_file_logger", log_file_path + "GroupTC-HASH/time_output.txt", spdlog::level::info);
        init_file_logger("GroupTC-OPT_file_logger", log_file_path + "GroupTC-OPT/time_output.txt", spdlog::level::info);
        init_file_logger("H-INDEX_file_logger", log_file_path + "H-INDEX/time_output.txt", spdlog::level::info);
        init_file_logger("Hu_file_logger", log_file_path + "Hu/time_output.txt", spdlog::level::info);
        init_file_logger("Polak_file_logger", log_file_path + "Polak/time_output.txt", spdlog::level::info);
        init_file_logger("TriCore_file_logger", log_file_path + "TriCore/time_output.txt", spdlog::level::info);
        init_file_logger("TRUST_file_logger", log_file_path + "TRUST/time_output.txt", spdlog::level::info);

    } catch (const spdlog::spdlog_ex& ex) {
        spdlog::error("Log init failed: {}", ex.what());
    }
}

void init_file_logger(std::string logger_name, std::string file, spdlog::level::level_enum log_level) {
    std::shared_ptr<spdlog::logger> logger = spdlog::basic_logger_mt(logger_name, file);
    logger->set_level(log_level);
}

void flush_loggers() {
    spdlog::get("csr2dcsr_file_logger")->flush();
    spdlog::get("csr2rid_dcsr_file_logger")->flush();
    spdlog::get("csr2trust_dcsr_file_logger")->flush();

    spdlog::get("Fox_preprocessing_file_logger")->flush();
    spdlog::get("Hu_preprocessing_file_logger")->flush();

    spdlog::get("Bisson_file_logger")->flush();
    spdlog::get("Fox_file_logger")->flush();
    spdlog::get("Green_file_logger")->flush();
    spdlog::get("GroupTC_file_logger")->flush();
    spdlog::get("GroupTC-HASH_file_logger")->flush();
    spdlog::get("GroupTC-OPT_file_logger")->flush();
    spdlog::get("H-INDEX_file_logger")->flush();
    spdlog::get("Hu_file_logger")->flush();
    spdlog::get("Polak_file_logger")->flush();
    spdlog::get("TriCore_file_logger")->flush();
    spdlog::get("TRUST_file_logger")->flush();
}

spdlog::level::level_enum switch_log_level(std::string log_level_str) {
    if (log_level_str == "trace") {
        return spdlog::level::trace;
    } else if (log_level_str == "debug") {
        return spdlog::level::debug;
    } else if (log_level_str == "info") {
        return spdlog::level::info;
    } else if (log_level_str == "warn") {
        return spdlog::level::warn;
    } else if (log_level_str == "err") {
        return spdlog::level::err;
    } else if (log_level_str == "critical") {
        return spdlog::level::critical;
    } else if (log_level_str == "off") {
        return spdlog::level::off;
    } else {
        return spdlog::level::info;
    }
}
