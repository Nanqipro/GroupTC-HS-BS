#include "hip/hip_runtime.h"
#include "graph/cuda_graph_comm.h"

__global__ void cuda_graph_comm::copy_32_to_64(uint arr_len, int32_t* arr_32, int64_t* arr_64) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= arr_len) {
        return;
    }
    arr_64[i] = arr_32[i];
}

__global__ void cuda_graph_comm::zip_edge(uint edge_count, uint vertex_count, vertex_t* d_edge_arr, vertex_t* d_src_arr, vertex_t* d_adj_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_src_arr[i];
    int adj = d_adj_arr[i];
    d_edge_arr[i * 2 + 1] = src;
    d_edge_arr[i * 2] = adj;
}

__global__ void cuda_graph_comm::cal_degree_and_zip_edge(uint edge_count, uint vertex_count, uint* d_degree_arr, vertex_t* d_edge_arr,
                                                         vertex_t* d_src_arr, vertex_t* d_adj_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_src_arr[i];
    int adj = d_adj_arr[i];
    d_edge_arr[i * 2 + 1] = src;
    d_edge_arr[i * 2] = adj;

    atomicAdd(d_degree_arr + src, 1);
    atomicAdd(d_degree_arr + adj, 1);
}

__global__ void cuda_graph_comm::redirect_edge(uint edge_count, uint vertex_count, uint* d_degree_arr, vertex_t* d_edge_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_edge_arr[i * 2 + 1];
    int adj = d_edge_arr[i * 2];
    // redirect edge
    if (d_degree_arr[src] > d_degree_arr[adj] || (d_degree_arr[src] == d_degree_arr[adj] && src > adj)) {
        d_edge_arr[i * 2 + 1] = adj;
        d_edge_arr[i * 2] = src;
    }
}

__global__ void cuda_graph_comm::redirect_edge(uint edge_count, uint vertex_count, uint* d_degree_arr, vertex_t* d_src_arr, vertex_t* d_adj_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_src_arr[i];
    int dst = d_adj_arr[i];
    // redirect edge
    if (d_degree_arr[src] > d_degree_arr[dst] || (d_degree_arr[src] == d_degree_arr[dst] && src > dst)) {
        d_adj_arr[i] = src;
        d_src_arr[i] = dst;
    }
}

__global__ void cuda_graph_comm::unzip_edge(uint edge_count, uint vertex_count, vertex_t* d_edge_arr, vertex_t* d_src_arr, vertex_t* d_adj_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    d_src_arr[i] = d_edge_arr[i * 2 + 1];
    d_adj_arr[i] = d_edge_arr[i * 2];
}

__global__ void cuda_graph_comm::recal_offset(uint edge_count, uint vertex_count, vertex_t* d_src_arr, index_t* d_offset_arr) {
    uint from = blockDim.x * blockIdx.x + threadIdx.x;
    uint step = gridDim.x * blockDim.x;
    for (uint i = from; i <= edge_count; i += step) {
        int64_t prev = i > 0 ? (int64_t)d_src_arr[i - 1] : -1;
        int64_t next = i < edge_count ? (int64_t)d_src_arr[i] : vertex_count;
        // 前一个元素小于后一个元素，才有可能出现 offset 的计算
        for (int64_t j = prev + 1; j <= next; ++j) d_offset_arr[j] = i;
    }
}

__global__ void cuda_graph_comm::record_id(uint edge_count, uint vertex_count, vertex_t* d_id_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }
    d_id_arr[i] = i;
}

__global__ void cuda_graph_comm::map_id(uint edge_count, uint vertex_count, vertex_t* d_id_arr, vertex_t* d_id_map_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }
    d_id_map_arr[d_id_arr[i]] = i;
}

__global__ void cuda_graph_comm::redirect_edge_and_reassign_id(uint edge_count, uint vertex_count, vertex_t* d_id_map_arr, vertex_t* d_edge_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_id_map_arr[d_edge_arr[i * 2 + 1]];
    int adj = d_id_map_arr[d_edge_arr[i * 2]];
    if (src > adj) {
        int temp = src;
        src = adj;
        adj = temp;
    }
    d_edge_arr[i * 2] = adj;
    d_edge_arr[i * 2 + 1] = src;
}

__global__ void cuda_graph_comm::cal_degree(uint edge_count, uint vertex_count, uint* d_degree_arr, vertex_t* d_src_arr, vertex_t* d_adj_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_src_arr[i];
    int adj = d_adj_arr[i];

    atomicAdd(d_degree_arr + src, 1);
    atomicAdd(d_degree_arr + adj, 1);
}

__global__ void cuda_graph_comm::cal_out_degree_by_src(uint edge_count, uint vertex_count, uint* d_degree_arr, vertex_t* d_src_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    atomicAdd(d_degree_arr + d_src_arr[i], 1);
}

__global__ void cuda_graph_comm::cal_out_degree_by_offset(uint edge_count, uint vertex_count, uint* d_degree_arr, index_t* d_offset_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }

    d_degree_arr[i] = d_offset_arr[i + 1] - d_offset_arr[i];
}

__global__ void cuda_graph_comm::record_id_and_part_graph_by_degree(uint edge_count, uint vertex_count, vertex_t* d_id_arr, uint* d_degree_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }
    d_id_arr[i] = i;
    int degree = d_degree_arr[i];
    if (degree < 2) {
        degree = 2;
    } else if (degree <= 100) {
        degree = 1;
    } else {
        degree = 0;
    }
    d_degree_arr[i] = degree;
}

__global__ void cuda_graph_comm::reassign_id(uint edge_count, uint vertex_count, vertex_t* d_id_map_arr, vertex_t* d_src_arr, vertex_t* d_adj_arr) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    d_src_arr[i] = d_id_map_arr[d_src_arr[i]];
    d_adj_arr[i] = d_id_map_arr[d_adj_arr[i]];
}

__global__ void cuda_graph_comm::check_order(vertex_t* arr, uint len) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= len - 1) {
        return;
    }
    if (arr[i + 1] < arr[i]) {
        // printf("xxxxxxxxxxxxxxx error order xxxxxxxxxxxxxxx\n");
    }
}

__global__ void cuda_graph_comm::warm_up(int* data, size_t n) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    for(size_t i = tid; i < n; i += blockDim.x * gridDim.x) {
        data[i] = i;
    }
}
