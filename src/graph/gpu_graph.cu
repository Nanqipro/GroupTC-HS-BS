#include "comm/constant_comm.h"
#include "graph/gpu_graph.h"
#include "spdlog/spdlog.h"

GPUGraph::GPUGraph(CPUGraph &graph) { init(graph); }

void GPUGraph::init(CPUGraph &h_graph) {
    input_dir = h_graph.input_dir;
    vertex_count = h_graph.vertex_count;
    edge_count = h_graph.edge_count;
    max_degree = h_graph.max_degree;
    has_edge_list = h_graph.has_edge_list;

    size_t free_byte, total_byte;
    HRR(hipMemGetInfo(&free_byte, &total_byte));
    spdlog::debug("GPUGraph before transfer, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);

    size_t beg_sz = sizeof(index_t) * (vertex_count + 1 + 10);
    size_t adj_sz = sizeof(vertex_t) * (edge_count + 10);

    spdlog::info("Load graph from host to device, input dir is: {}", h_graph.input_dir);
    spdlog::info("Graph vertices: {0}, edges: {1}, avg degrees: {2}", vertex_count, edge_count, (float)edge_count * 2 / vertex_count);

    /* Alloc GPU space */
    HRR(hipMalloc((void **)&src_list, adj_sz));
    HRR(hipMalloc((void **)&adj_list, adj_sz));
    HRR(hipMalloc((void **)&beg_pos, beg_sz));

    spdlog::debug("Load graph from host to device, alloc GPU space finished");

    /* copy it to GPU */
    HRR(hipMemcpy(src_list, h_graph.src_list, sizeof(vertex_t) * (edge_count), hipMemcpyHostToDevice));
    HRR(hipMemcpy(adj_list, h_graph.adj_list, sizeof(vertex_t) * (edge_count), hipMemcpyHostToDevice));
    HRR(hipMemcpy(beg_pos, h_graph.beg_pos, sizeof(index_t) * (vertex_count + 1), hipMemcpyHostToDevice));

    if (has_edge_list) {
        size_t edge_sz = adj_sz * 2;
        HRR(hipMalloc((void **)&edge_list, edge_sz));
        HRR(hipMemcpy(edge_list, h_graph.edge_list, edge_sz, hipMemcpyHostToDevice));
    }

    spdlog::debug("Load graph from host to device, copy finished");
    spdlog::info("Load graph from host to device finished");

    HRR(hipMemGetInfo(&free_byte, &total_byte));
    spdlog::debug("GPUGraph after transfer, used memory {:.2f} GB", float(total_byte - free_byte) / MEMORY_G);
}

GPUGraph::~GPUGraph() {
    HRR(hipFree(beg_pos));
    HRR(hipFree(src_list));
    HRR(hipFree(adj_list));

    if (has_edge_list) {
        HRR(hipFree(edge_list));
    }
}