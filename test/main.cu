#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <functional>  // 用于 std::function

// 自定义比较器，只比较整数的前n位
struct MostSignificantBitsComparator {
    int numBits;  // 要比较的位数
    
    __host__ __device__ __forceinline__
    MostSignificantBitsComparator(int numBits) : numBits(numBits) {}
    
    __host__ __device__ __forceinline__
    bool operator()(const unsigned long long int &a, const unsigned long long int &b) const {
        // 创建掩码，只保留前numBits位
        unsigned long long int mask = 0xFFFFFFFFFFFFFFFFULL;
        if (numBits < 64) {
            mask = mask << (64 - numBits);
        }
        
        // 应用掩码并比较
        unsigned long long int maskedA = a & mask;
        unsigned long long int maskedB = b & mask;
        
        return maskedA < maskedB;
    }
};

// 用于保存排序后的原始索引和值的结构体
struct KeyValuePair {
    unsigned long long int key;   // 原始值
    int value;                    // 原始索引
};

// 仿函数，根据键（前n位）进行比较
struct KeyValueComparator {
    int numBits;
    
    __host__ __device__ __forceinline__
    KeyValueComparator(int numBits) : numBits(numBits) {}
    
    __host__ __device__ __forceinline__
    bool operator()(const KeyValuePair &a, const KeyValuePair &b) const {
        // 创建掩码，只保留前numBits位
        unsigned long long int mask = 0xFFFFFFFFFFFFFFFFULL;
        if (numBits < 64) {
            mask = mask << (64 - numBits);
        }
        
        // 应用掩码并比较
        unsigned long long int maskedA = a.key & mask;
        unsigned long long int maskedB = b.key & mask;
        
        return maskedA < maskedB;
    }
};

// 对整数数组仅考虑前n位进行排序
void sortByMostSignificantBits(unsigned long long int* d_in, unsigned long long int* d_out, int n, int numBits) {
    // 使用thrust::device_ptr包装原始指针
    thrust::device_ptr<unsigned long long int> dev_ptr_in(d_in);
    thrust::device_ptr<unsigned long long int> dev_ptr_out(d_out);
    
    // 自定义比较器
    MostSignificantBitsComparator comp(numBits);
    
    // 复制数据到输出数组
    thrust::copy(dev_ptr_in, dev_ptr_in + n, dev_ptr_out);
    
    // 使用自定义比较器排序
    thrust::sort(
        thrust::device,
        dev_ptr_out, dev_ptr_out + n,
        comp);
}

// 对整数数组仅考虑前n位进行排序，同时保留原始索引
void sortByMostSignificantBitsWithIndices(unsigned long long int* d_keys_in, unsigned long long int* d_keys_out, 
                                        int* d_values_in, int* d_values_out, 
                                        int n, int numBits) {
    // 使用thrust::device_ptr包装原始指针
    thrust::device_ptr<unsigned long long int> dev_keys_in(d_keys_in);
    thrust::device_ptr<int> dev_values_in(d_values_in);
    thrust::device_ptr<unsigned long long int> dev_keys_out(d_keys_out);
    thrust::device_ptr<int> dev_values_out(d_values_out);
    
    // 创建仅考虑前numBits位的比较器函数
    MostSignificantBitsComparator comp(numBits);
    
    // 定义键值对
    auto zip_begin = thrust::make_zip_iterator(thrust::make_tuple(dev_keys_in, dev_values_in));
    auto zip_end = zip_begin + n;
    auto zip_out = thrust::make_zip_iterator(thrust::make_tuple(dev_keys_out, dev_values_out));
    
    // 复制数据到输出数组
    thrust::copy(dev_keys_in, dev_keys_in + n, dev_keys_out);
    thrust::copy(dev_values_in, dev_values_in + n, dev_values_out);
    
    // 使用自定义比较器，基于键排序
    thrust::stable_sort_by_key(
        thrust::device,
        dev_keys_out, dev_keys_out + n,
        dev_values_out,
        comp
    );
}

// 使用键值对结构体进行排序（当需要更复杂的自定义比较时使用）
void sortKeyValuePairs(KeyValuePair* d_pairs_in, KeyValuePair* d_pairs_out, int n, int numBits) {
    // 使用thrust::device_ptr包装原始指针
    thrust::device_ptr<KeyValuePair> dev_pairs_in(d_pairs_in);
    thrust::device_ptr<KeyValuePair> dev_pairs_out(d_pairs_out);
    
    // 自定义比较器
    KeyValueComparator comp(numBits);
    
    // 复制数据到输出数组
    thrust::copy(dev_pairs_in, dev_pairs_in + n, dev_pairs_out);
    
    // 使用自定义比较器排序
    thrust::sort(
        thrust::device,
        dev_pairs_out, dev_pairs_out + n,
        comp);
}

// 检查CUDA错误
#define CHECK_CUDA_ERROR(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// 测量 GPU 排序函数执行时间的帮助函数
void measureSortingTime(const char* sortName, std::function<void()> sortFunc) {
    // 创建CUDA事件
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    
    // 记录开始时间
    CHECK_CUDA_ERROR(hipEventRecord(start));
    
    // 执行排序函数
    sortFunc();
    
    // 记录结束时间
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    // 计算执行时间（毫秒）
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    
    // 输出排序函数名称和执行时间
    std::cout << "【" << sortName << "】执行时间: " << milliseconds << " ms" << std::endl;
    
    // 销毁CUDA事件
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
}

int main() {
    int n = 10000;            // 增加数组大小以便更好地测量性能
    int numBits = 8;          // 只考虑前8位进行排序，使结果更明显
    
    // 主机内存分配
    unsigned long long int* h_data = new unsigned long long int[n];
    unsigned long long int* h_result = new unsigned long long int[n];
    int* h_indices = new int[n];
    int* h_indices_result = new int[n];
    
    // 初始化数据和索引
    srand(time(NULL));
    std::cout << "原始数据（十进制和二进制）：" << std::endl;
    for (int i = 0; i < n; i++) {
        // 生成64位随机数，创建更大范围的数据
        h_data[i] = ((unsigned long long int)rand() << 48) | ((unsigned long long int)rand() << 32) | 
                    ((unsigned long long int)rand() << 16) | (unsigned long long int)rand();
        h_indices[i] = i;          // 初始索引
        
        // 输出十进制值和二进制表示
        std::cout << "[" << i << "] " << h_data[i] << " (";
        for (int bit = 63; bit >= 0; bit--) {
            std::cout << ((h_data[i] >> bit) & 1);
            if (bit == 64 - numBits) std::cout << "|"; // 标记前numBits位的分隔
        }
        std::cout << ")" << std::endl;
    }
    
    // 设备内存分配
    unsigned long long int *d_data, *d_result;
    int *d_indices, *d_indices_result;
    CHECK_CUDA_ERROR(hipMalloc(&d_data, n * sizeof(unsigned long long int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_result, n * sizeof(unsigned long long int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_indices, n * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_indices_result, n * sizeof(int)));
    
    // 复制数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_data, h_data, n * sizeof(unsigned long long int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_indices, h_indices, n * sizeof(int), hipMemcpyHostToDevice));
    
    std::cout << "\n执行基于前" << numBits << "位的排序..." << std::endl;
    
    // 方法1：只考虑前numBits位进行排序，并测量时间
    measureSortingTime("基本前n位排序", [&]() {
        sortByMostSignificantBits(d_data, d_result, n, numBits);
    });
    
    // 方法2：排序同时保留原始索引，并测量时间
    measureSortingTime("带索引的前n位排序", [&]() {
        sortByMostSignificantBitsWithIndices(d_data, d_result, d_indices, d_indices_result, n, numBits);
    });
    
    // 将结果复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_result, d_result, n * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(h_indices_result, d_indices_result, n * sizeof(int), hipMemcpyDeviceToHost));
    
    // 输出结果（只显示前10个结果，避免输出太多）
    std::cout << "\n排序后的结果（只考虑前" << numBits << "位）：" << std::endl;
    int display_count = std::min(n, 10);
    for (int i = 0; i < display_count; i++) {
        std::cout << "[" << i << "] " << h_result[i] << " (";
        for (int bit = 63; bit >= 0; bit--) {
            std::cout << ((h_result[i] >> bit) & 1);
            if (bit == 64 - numBits) std::cout << "|"; // 标记前numBits位的分隔
        }
        std::cout << ") 原始索引: " << h_indices_result[i] << std::endl;
    }
    if (n > 10) {
        std::cout << "... (只显示前10个结果)" << std::endl;
    }
    
    // 方法3：使用键值对结构体进行排序（示例）
    KeyValuePair* h_pairs = new KeyValuePair[n];
    for (int i = 0; i < n; i++) {
        h_pairs[i].key = h_data[i];
        h_pairs[i].value = i;
    }
    
    KeyValuePair *d_pairs, *d_pairs_result;
    CHECK_CUDA_ERROR(hipMalloc(&d_pairs, n * sizeof(KeyValuePair)));
    CHECK_CUDA_ERROR(hipMalloc(&d_pairs_result, n * sizeof(KeyValuePair)));
    CHECK_CUDA_ERROR(hipMemcpy(d_pairs, h_pairs, n * sizeof(KeyValuePair), hipMemcpyHostToDevice));
    
    // 执行键值对排序并测量时间
    measureSortingTime("键值对结构体排序", [&]() {
        sortKeyValuePairs(d_pairs, d_pairs_result, n, numBits);
    });
    
    // 复制结果回主机
    KeyValuePair* h_pairs_result = new KeyValuePair[n];
    CHECK_CUDA_ERROR(hipMemcpy(h_pairs_result, d_pairs_result, n * sizeof(KeyValuePair), hipMemcpyDeviceToHost));
    
    // 输出键值对排序结果（只显示前10个结果，避免输出太多）
    std::cout << "\n键值对排序结果（只考虑前" << numBits << "位）：" << std::endl;
    for (int i = 0; i < display_count; i++) {
        std::cout << "[" << i << "] 值: " << h_pairs_result[i].key << " 原始索引: " << h_pairs_result[i].value << std::endl;
    }
    if (n > 10) {
        std::cout << "... (只显示前10个结果)" << std::endl;
    }
    
    // 清理内存
    delete[] h_data;
    delete[] h_result;
    delete[] h_indices;
    delete[] h_indices_result;
    delete[] h_pairs;
    delete[] h_pairs_result;
    
    hipFree(d_data);
    hipFree(d_result);
    hipFree(d_indices);
    hipFree(d_indices_result);
    hipFree(d_pairs);
    hipFree(d_pairs_result);
    
    return 0;
}