#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <functional>  // 用于 std::function

// 自定义比较器，只比较整数的前n位
struct MostSignificantBitsComparator {
    int numBits;  // 要比较的位数
    
    __host__ __device__ __forceinline__
    MostSignificantBitsComparator(int numBits) : numBits(numBits) {}
    
    __host__ __device__ __forceinline__
    bool operator()(const unsigned long long int &a, const unsigned long long int &b) const {
        // 创建掩码，只保留前numBits位
        unsigned long long int mask = 0xFFFFFFFFFFFFFFFFULL;
        if (numBits < 64) {
            mask = mask << (64 - numBits);
        }
        
        // 应用掩码并比较
        unsigned long long int maskedA = a & mask;
        unsigned long long int maskedB = b & mask;
        
        return maskedA < maskedB;
    }
};

// 对整数数组仅考虑前n位进行排序
void sortByMostSignificantBits(unsigned long long int* d_in, unsigned long long int* d_out, int n, int numBits) {
    // 使用thrust::device_ptr包装原始指针
    thrust::device_ptr<unsigned long long int> dev_ptr_in(d_in);
    thrust::device_ptr<unsigned long long int> dev_ptr_out(d_out);
    
    // 自定义比较器
    MostSignificantBitsComparator comp(numBits);
    
    // 复制数据到输出数组
    thrust::copy(dev_ptr_in, dev_ptr_in + n, dev_ptr_out);
    
    // 使用自定义比较器排序
    thrust::sort(
        thrust::device,
        dev_ptr_out, dev_ptr_out + n,
        comp);
}

// 检查CUDA错误
#define CHECK_CUDA_ERROR(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// 测量 GPU 排序函数执行时间的帮助函数
void measureSortingTime(const char* sortName, std::function<void()> sortFunc) {
    // 创建CUDA事件
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    
    // 记录开始时间
    CHECK_CUDA_ERROR(hipEventRecord(start));
    
    // 执行排序函数
    sortFunc();
    
    // 记录结束时间
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    // 计算执行时间（毫秒）
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    
    // 输出排序函数名称和执行时间
    std::cout << "【" << sortName << "】执行时间: " << milliseconds << " ms" << std::endl;
    
    // 销毁CUDA事件
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
}

int main() {
    int n = 10000;            // 数组大小
    int numBits = 8;          // 只考虑前8位进行排序，使结果更明显
    
    // 主机内存分配
    unsigned long long int* h_data = new unsigned long long int[n];
    unsigned long long int* h_result = new unsigned long long int[n];
    
    // 初始化数据
    srand(time(NULL));
    std::cout << "原始数据（十进制和二进制）：" << std::endl;
    for (int i = 0; i < n; i++) {
        // 生成64位随机数，创建更大范围的数据
        h_data[i] = ((unsigned long long int)rand() << 48) | ((unsigned long long int)rand() << 32) | 
                    ((unsigned long long int)rand() << 16) | (unsigned long long int)rand();
        
        // 输出十进制值和二进制表示（只显示前10个元素）
        if (i < 10) {
            std::cout << "[" << i << "] " << h_data[i] << " (";
            for (int bit = 63; bit >= 0; bit--) {
                std::cout << ((h_data[i] >> bit) & 1);
                if (bit == 64 - numBits) std::cout << "|"; // 标记前numBits位的分隔
            }
            std::cout << ")" << std::endl;
        }
    }
    if (n > 10) {
        std::cout << "... (只显示前10个数据)" << std::endl;
    }
    
    // 设备内存分配
    unsigned long long int *d_data, *d_result;
    CHECK_CUDA_ERROR(hipMalloc(&d_data, n * sizeof(unsigned long long int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_result, n * sizeof(unsigned long long int)));
    
    // 复制数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_data, h_data, n * sizeof(unsigned long long int), hipMemcpyHostToDevice));
    
    std::cout << "\n执行基于前" << numBits << "位的排序..." << std::endl;
    
    // 执行排序并测量时间
    measureSortingTime("基于前n位排序", [&]() {
        sortByMostSignificantBits(d_data, d_result, n, numBits);
    });
    
    // 将结果复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_result, d_result, n * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
    
    // 输出结果（只显示前10个结果，避免输出太多）
    std::cout << "\n排序后的结果（只考虑前" << numBits << "位）：" << std::endl;
    int display_count = std::min(n, 10);
    for (int i = 0; i < display_count; i++) {
        std::cout << "[" << i << "] " << h_result[i] << " (";
        for (int bit = 63; bit >= 0; bit--) {
            std::cout << ((h_result[i] >> bit) & 1);
            if (bit == 64 - numBits) std::cout << "|"; // 标记前numBits位的分隔
        }
        std::cout << ")" << std::endl;
    }
    if (n > 10) {
        std::cout << "... (只显示前10个结果)" << std::endl;
    }
    
    // 清理内存
    delete[] h_data;
    delete[] h_result;
    
    hipFree(d_data);
    hipFree(d_result);
    
    return 0;
}
