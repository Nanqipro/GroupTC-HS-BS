#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <fstream>
#include <cstdio>
#include <vector>
#include <sstream>
#include <cmath>

#include "./common/comm.h"

using namespace std;

typedef struct edge
{
    index_t u, v;
} edge;

typedef struct vertex
{
    index_t old_id;
    int degree;
} vertex;

bool cmp1(vertex &a, vertex &b);
bool cmp2(vertex &a, vertex &b);
bool cmp3(edge &a, edge &b);
int printMaxDegree(string str);

__global__ void cal_degree(int edge_count, int vertex_count, int *d_degreeArr, int *d_edgeArr, int *d_srcArr, int *d_dstArr);
__global__ void redirect_edge(int edge_count, int vertex_count, int *d_degreeArr, int *d_edgeArr);
__global__ void unzip_edge(int edge_count, int vertex_count, int *d_edgeArr, int *d_srcArr, int *d_dstArr);
__global__ void cal_offset(int edge_count, int vertex_count, int *d_srcArr, index_t *d_offsetArr);
void comparasion(vertex_t *d_degreeArr,
                 vertex_t *d_srcArr,
                 vertex_t *d_dstArr,
                 vertex_t *d_edgeArr,
                 index_t *d_offsetArr,
                 vertex_t *degreeArr,
                 vertex_t *srcArr,
                 vertex_t *dstArr,
                 vertex_t *edgeArr,
                 index_t *offsetArr);

int vertex_count;
long long int edge_count;

index_t *offsetArr;
vertex_t *srcArr;
vertex_t *dstArr;

long long sizeEdgeList;
long long sizeOffsetList;

void loadgraph(string prefix)
{
    string s_begin = prefix + "begin.bin";
    string s_source = prefix + "source.bin";
    string s_adj = prefix + "adjacent.bin";

    char *begin_file = const_cast<char *>(s_begin.c_str());
    char *source_file = const_cast<char *>(s_source.c_str());
    char *adj_file = const_cast<char *>(s_adj.c_str());

    ifstream beginFile(begin_file, ios::in | ios::binary);
    ifstream sourceFile(source_file, ios::in | ios::binary);
    ifstream adjFile(adj_file, ios::in | ios::binary);

    vertex_count = fsize(begin_file) / sizeof(index_t) - 1;
    edge_count = fsize(adj_file) / sizeof(vertex_t);

    cout << "vertex: " << vertex_count << "   edge: " << edge_count << endl;
    sizeOffsetList = sizeof(index_t) * (vertex_count + 1);
    sizeEdgeList = sizeof(vertex_t) * edge_count;

    offsetArr = (index_t *)malloc(sizeOffsetList);
    srcArr = (vertex_t *)malloc(sizeEdgeList);
    dstArr = (vertex_t *)malloc(sizeEdgeList);

    beginFile.read((char *)&offsetArr[0], sizeOffsetList);
    sourceFile.read((char *)&srcArr[0], sizeEdgeList);
    adjFile.read((char *)&dstArr[0], sizeEdgeList);

    beginFile.close();
    sourceFile.close();
    adjFile.close();
}

void writeback(string prefix)
{
    ofstream beginFile((prefix + "begin.bin").c_str(), ios::out | ios::binary);
    ofstream sourceFile((prefix + "source.bin").c_str(), ios::out | ios::binary);
    ofstream adjFile((prefix + "adjacent.bin").c_str(), ios::out | ios::binary);

    // ofstream outFile((prefix + "graph.txt").c_str(), ios::out);

    // for (int i = 0; i < 100; i++)
    // {
    //     outFile << i << " " << srcArr[i] << endl;
    // }

    // outFile << "===========================================" << endl;

    // for (int i = 0; i < 100; i++)
    // {
    //     outFile << i << " " << offsetArr[i] << endl;
    // }
    // outFile.close();

    beginFile.write((char *)&offsetArr[0], sizeOffsetList);
    sourceFile.write((char *)&srcArr[0], sizeEdgeList);
    adjFile.write((char *)&dstArr[0], sizeEdgeList);

    beginFile.close();
    sourceFile.close();
    adjFile.close();

    free(srcArr);
    free(dstArr);
    free(offsetArr);
}

void compute()
{
    hipSetDevice(1);
    
    vertex_t *d_degreeArr;
    vertex_t *d_srcArr;
    vertex_t *d_dstArr;
    vertex_t *d_edgeArr;
    index_t *d_offsetArr;

    size_t sizeVertexArr = sizeof(vertex_t) * vertex_count;
    size_t sizeEdgeArr = sizeof(vertex_t) * edge_count;

    HRR(hipMalloc((void **)&d_degreeArr, sizeVertexArr));
    HRR(hipMalloc((void **)&d_srcArr, sizeEdgeArr));
    HRR(hipMalloc((void **)&d_dstArr, sizeEdgeArr));
    HRR(hipMalloc((void **)&d_edgeArr, sizeEdgeArr * 2));
    HRR(hipMalloc((void **)&d_offsetArr, sizeOffsetList));

    HRR(hipMemcpy(d_srcArr, srcArr, sizeEdgeArr, hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_dstArr, dstArr, sizeEdgeArr, hipMemcpyHostToDevice));
    HRR(hipMemset(d_degreeArr, 0, sizeVertexArr));

    int block_size = 1024;
    int grid_size = (edge_count - 1) / block_size + 1;

    printMaxDegree("before compute");
    double t_start = wtime();
    int iteration = 10;
    for (int k = 0; k < iteration; k++)
    {
        cal_degree<<<grid_size, block_size>>>(edge_count, vertex_count, d_degreeArr, d_edgeArr, d_srcArr, d_dstArr);
        // HRR(hipDeviceSynchronize());

        redirect_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_degreeArr, d_edgeArr);
        // HRR(hipDeviceSynchronize());

        thrust::device_ptr<uint64_t> sort_ptr((uint64_t *)d_edgeArr);
        thrust::sort(sort_ptr, sort_ptr + edge_count);

        unzip_edge<<<grid_size, block_size>>>(edge_count, vertex_count, d_edgeArr, d_srcArr, d_dstArr);
        // HRR(hipDeviceSynchronize());

        cal_offset<<<grid_size, block_size>>>(edge_count, vertex_count, d_srcArr, d_offsetArr);
        // HRR(hipDeviceSynchronize());
    }
    double t_end = wtime();

    cout << "compute time spent " << (t_end - t_start) / iteration << " s" << endl;

    HRR(hipMemcpy(offsetArr, d_offsetArr, sizeOffsetList, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(srcArr, d_srcArr, sizeEdgeArr, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(dstArr, d_dstArr, sizeEdgeArr, hipMemcpyDeviceToHost));
    printMaxDegree("after compute");

    hipFree(d_degreeArr);
    hipFree(d_offsetArr);
    hipFree(d_edgeArr);
    hipFree(d_srcArr);
    hipFree(d_dstArr);
}

__global__ void cal_degree(int edge_count, int vertex_count, int *d_degreeArr, int *d_edgeArr, int *d_srcArr, int *d_dstArr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int src = d_srcArr[i];
    int dst = d_dstArr[i];
    d_edgeArr[i * 2] = src;
    d_edgeArr[i * 2 + 1] = dst;

    atomicAdd(d_degreeArr + src, 1);
    atomicAdd(d_degreeArr + dst, 1);
}

__global__ void redirect_edge(int edge_count, int vertex_count, int *d_degreeArr, int *d_edgeArr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    int dst = d_edgeArr[i * 2];
    int src = d_edgeArr[i * 2 + 1];
    // redirect edge
    if (d_degreeArr[src] > d_degreeArr[dst] || (d_degreeArr[src] == d_degreeArr[dst] && src > dst))
    {
        d_edgeArr[i * 2] = src;
        d_edgeArr[i * 2 + 1] = dst;
    }
}

__global__ void unzip_edge(int edge_count, int vertex_count, int *d_edgeArr, int *d_srcArr, int *d_dstArr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count)
    {
        return;
    }
    d_srcArr[i] = d_edgeArr[i * 2 + 1];
    d_dstArr[i] = d_edgeArr[i * 2];
}

__global__ void cal_offset(int edge_count, int vertex_count, int *d_srcArr, index_t *d_offsetArr)
{
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i <= edge_count; i += step)
    {
        int prev = i > 0 ? d_srcArr[i - 1] : -1;
        int next = i < edge_count ? d_srcArr[i] : vertex_count;
        // 前一个元素小于后一个元素，才有可能出现 offset 的计算
        for (int j = prev + 1; j <= next; ++j)
            d_offsetArr[j] = i;
    }
}

void comparasion(vertex_t *d_degreeArr,
                 vertex_t *d_srcArr,
                 vertex_t *d_dstArr,
                 vertex_t *d_edgeArr,
                 index_t *d_offsetArr,
                 vertex_t *degreeArr,
                 vertex_t *srcArr,
                 vertex_t *dstArr,
                 vertex_t *edgeArr,
                 index_t *offsetArr)
{
    vertex_t *degreeArr2 = (vertex_t *)malloc(sizeof(vertex_t) * vertex_count);
    vertex_t *srcArr2 = (vertex_t *)malloc(sizeof(vertex_t) * edge_count);
    vertex_t *dstArr2 = (vertex_t *)malloc(sizeof(vertex_t) * edge_count);
    vertex_t *edgeArr2 = (vertex_t *)malloc(sizeof(vertex_t) * edge_count * 2);
    vertex_t *offsetArr2 = (vertex_t *)malloc(sizeof(index_t) * (vertex_count + 1));

    HRR(hipMemcpy(degreeArr2, d_degreeArr, sizeof(vertex_t) * vertex_count, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(srcArr2, d_srcArr, sizeof(vertex_t) * edge_count, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(dstArr2, d_dstArr, sizeof(vertex_t) * edge_count, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(edgeArr2, d_edgeArr, sizeof(vertex_t) * edge_count * 2, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(offsetArr2, d_offsetArr, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

    for (int i = 0; i < edge_count * 2; i++)
    {
        if (i < vertex_count && degreeArr2[i] != degreeArr[i])
        {
            cout << "degree " << i << "  " << degreeArr2[i] << "  " << degreeArr[i] << endl;
        }
        if (i < edge_count && srcArr2[i] != srcArr[i])
        {
            cout << "src " << i << "  " << srcArr2[i] << "  " << srcArr[i] << endl;
        }
        if (i < edge_count && dstArr2[i] != dstArr[i])
        {
            cout << "dst " << i << "  " << dstArr2[i] << "  " << dstArr[i] << endl;
        }
        if (edgeArr2[i] != edgeArr[i])
        {
            cout << "edge " << i << "  " << edgeArr2[i] << "  " << edgeArr[i] << endl;
        }
        if (i < vertex_count + 1 && offsetArr2[i] != offsetArr[i])
        {
            // cout << "offset " << i << "  " << offsetArr2[i] << "  " << offsetArr[i] << endl;
        }
    }

    free(degreeArr2);
    free(srcArr2);
    free(dstArr2);
    free(edgeArr2);
    free(offsetArr2);
}

void riddcsr(string inPrefix, string outPrefix)
{
    // 将边加载到图中
    loadgraph(inPrefix);
    cout << "loadok" << endl;

    compute();

    writeback(outPrefix);
    cout << "writebackok" << endl;
}

int main(int argc, char *argv[])
{
    string inPrefix = argv[1];
    string outPrefix = argv[2];

    cout << "inPath: " << inPrefix << endl;
    cout << "outPath: " << outPrefix << endl;
    riddcsr(inPrefix, outPrefix);
    cout << endl;
}

bool cmp1(vertex &a, vertex &b)
{
    return a.degree < b.degree;
}

bool cmp2(vertex &a, vertex &b)
{
    return a.degree > b.degree;
}

bool cmp3(edge &a, edge &b)
{
    return a.u < b.u || (a.u == b.u && a.v < b.v);
}

int printMaxDegree(string str)
{
    int maxDegre = 0;
    for (index_t i = 1; i <= vertex_count; i++)
    {
        if (offsetArr[i] - offsetArr[i - 1] > maxDegre)
        {
            maxDegre = offsetArr[i] - offsetArr[i - 1];
        }
    }
    cout << str << " max degree :" << maxDegre << endl;
    return maxDegre;
}