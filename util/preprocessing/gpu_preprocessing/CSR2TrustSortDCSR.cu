#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include "./common/comm.h"

using namespace std;

typedef struct edge {
    index_t u, v;
} edge;

typedef struct vertex {
    index_t old_id;
    int degree;
} vertex;

bool cmp1(vertex &a, vertex &b);
bool cmp2(vertex &a, vertex &b);
bool cmp3(edge &a, edge &b);
int printMaxDegree(string str);

/**
 * @brief 记录变换前的 id，用来生成 d_idMapArr
 */
__global__ void record_id(int edge_count, int vertex_count, int *d_idArr);

/**
 * @brief 计算顶点度数并压缩边
 */
__global__ void cal_degree(int edge_count, int vertex_count, int *d_degreeArr, int *d_srcArr, int *d_dstArr);

/**
 * @brief 记录 id 映射
 */
__global__ void map_id(int edge_count, int vertex_count, int *d_idArr, int *d_idMapArr);

/**
 * @brief 边变向
 */
__global__ void redirect_edge(int edge_count, int vertex_count, int *d_degreeArr, int *d_srcArr, int *d_dstArr);

/**
 * @brief 重新分配 id
 *
 */
__global__ void reassign_id(int edge_count, int vertex_count, int *d_idMapArr, int *d_srcArr, int *d_dstArr);

/**
 * @brief 计算 src 的出度
 */
__global__ void cal_src_out_degree(int edge_count, int vertex_count, int *d_degreeArr, int *d_srcArr);

/**
 * @brief 解压缩边
 */
__global__ void unzip_edge(int edge_count, int vertex_count, int *d_edgeArr, int *d_srcArr, int *d_dstArr);

/**
 * @brief 计算偏移数组
 */
__global__ void cal_offset(int edge_count, int vertex_count, int *d_srcArr, index_t *d_offsetArr);

__global__ void show_arr(int edge_count, int vertex_count, int *d_degreeArr);

void comparasion(vertex_t *d_degreeArr, vertex_t *d_srcArr, vertex_t *d_dstArr, vertex_t *d_edgeArr, index_t *d_offsetArr, vertex_t *degreeArr,
                 vertex_t *srcArr, vertex_t *dstArr, vertex_t *edgeArr, index_t *offsetArr);

int vertex_count;
long long int edge_count;

index_t *offsetArr;
vertex_t *srcArr;
vertex_t *dstArr;

long long sizeEdgeList;
long long sizeOffsetList;

void loadgraph(string prefix) {
    string s_begin = prefix + "begin.bin";
    string s_source = prefix + "source.bin";
    string s_adj = prefix + "adjacent.bin";

    char *begin_file = const_cast<char *>(s_begin.c_str());
    char *source_file = const_cast<char *>(s_source.c_str());
    char *adj_file = const_cast<char *>(s_adj.c_str());

    ifstream beginFile(begin_file, ios::in | ios::binary);
    ifstream sourceFile(source_file, ios::in | ios::binary);
    ifstream adjFile(adj_file, ios::in | ios::binary);

    vertex_count = fsize(begin_file) / sizeof(index_t) - 1;
    edge_count = fsize(adj_file) / sizeof(vertex_t);

    cout << "vertex: " << vertex_count << "   edge: " << edge_count << endl;
    sizeOffsetList = sizeof(index_t) * (vertex_count + 1);
    sizeEdgeList = sizeof(vertex_t) * edge_count;

    offsetArr = (index_t *)malloc(sizeOffsetList);
    srcArr = (vertex_t *)malloc(sizeEdgeList);
    dstArr = (vertex_t *)malloc(sizeEdgeList);

    beginFile.read((char *)&offsetArr[0], sizeOffsetList);
    sourceFile.read((char *)&srcArr[0], sizeEdgeList);
    adjFile.read((char *)&dstArr[0], sizeEdgeList);

    beginFile.close();
    sourceFile.close();
    adjFile.close();
}

void writeback(string prefix) {
    ofstream beginFile((prefix + "begin.bin").c_str(), ios::out | ios::binary);
    ofstream sourceFile((prefix + "source.bin").c_str(), ios::out | ios::binary);
    ofstream adjFile((prefix + "adjacent.bin").c_str(), ios::out | ios::binary);

    // ofstream outFile((prefix + "graph.txt").c_str(), ios::out);

    // for (int i = 0; i < edge_count; i++)
    // {
    //     if (srcArr[i] >= dstArr[i])
    //     {
    //         cout << i << " " << srcArr[i] << " " << dstArr[i] << endl;
    //     }
    //     // outFile << i << " " << srcArr[i] << endl;
    // }

    // for (int i = 1; i < vertex_count; i++)
    // {
    //     // if (offsetArr[i + 1] - offsetArr[i] > offsetArr[i] - offsetArr[i - 1])
    //     // {
    //     //     cout << i << " " << offsetArr[i + 1] - offsetArr[i] << " " << offsetArr[i] - offsetArr[i - 1] << endl;
    //     // }
    //     if (offsetArr[i + 1] - offsetArr[i] < offsetArr[i] - offsetArr[i - 1])
    //     {
    //         cout << i << " " << offsetArr[i + 1] - offsetArr[i] << " " << offsetArr[i] - offsetArr[i - 1] << endl;
    //     }
    //     // outFile << i << " " << srcArr[i] << endl;
    // }
    // for (int i = 0; i < 100; i++)
    // {
    //     cout << i << " " << offsetArr[i + 1] - offsetArr[i] << endl;
    // }

    // outFile << "===========================================" << endl;

    // for (int i = 0; i < 100; i++)
    // {
    //     outFile << i << " " << offsetArr[i] << endl;
    // }
    // outFile.close();

    beginFile.write((char *)&offsetArr[0], sizeOffsetList);
    sourceFile.write((char *)&srcArr[0], sizeEdgeList);
    adjFile.write((char *)&dstArr[0], sizeEdgeList);

    beginFile.close();
    sourceFile.close();
    adjFile.close();

    free(srcArr);
    free(dstArr);
    free(offsetArr);
}

void compute() {
    hipSetDevice(3);

    vertex_t *d_degreeArr;
    vertex_t *d_idArr;
    vertex_t *d_idMapArr;
    vertex_t *d_srcArr;
    vertex_t *d_dstArr;
    index_t *d_offsetArr;

    size_t sizeVertexArr = sizeof(vertex_t) * vertex_count;
    size_t sizeEdgeArr = sizeof(vertex_t) * edge_count;

    HRR(hipMalloc((void **)&d_degreeArr, sizeVertexArr));
    HRR(hipMalloc((void **)&d_idArr, sizeVertexArr));
    HRR(hipMalloc((void **)&d_idMapArr, sizeVertexArr));
    HRR(hipMalloc((void **)&d_srcArr, sizeEdgeArr));
    HRR(hipMalloc((void **)&d_dstArr, sizeEdgeArr));
    HRR(hipMalloc((void **)&d_offsetArr, sizeOffsetList));

    HRR(hipMemcpy(d_srcArr, srcArr, sizeEdgeArr, hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_dstArr, dstArr, sizeEdgeArr, hipMemcpyHostToDevice));
    HRR(hipMemset(d_degreeArr, 0, sizeVertexArr));

    int block_size = 1024;
    int vertex_grid_size = (vertex_count - 1) / block_size + 1;
    int edge_grid_size = (edge_count - 1) / block_size + 1;

    printMaxDegree("before compute");

    double t_start = wtime();
    int iteration = 100;
    for (int k = 0; k < iteration; k++) {
        cal_degree<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degreeArr, d_srcArr, d_dstArr);
        HRR(hipDeviceSynchronize());

        redirect_edge<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degreeArr, d_srcArr, d_dstArr);
        HRR(hipDeviceSynchronize());

        HRR(hipMemset(d_degreeArr, 0, sizeVertexArr));
        cal_src_out_degree<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_degreeArr, d_srcArr);
        HRR(hipDeviceSynchronize());
        // cout << "xxxxxxxxxxxxxx" << endl;

        record_id<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_idArr);
        HRR(hipDeviceSynchronize());

        // show_arr<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_degreeArr);
        // HRR(hipDeviceSynchronize());

        thrust::device_ptr<vertex_t> d_id_ptr((vertex_t *)d_idArr);
        thrust::sort_by_key(d_degreeArr, d_degreeArr + vertex_count, d_id_ptr, thrust::greater<vertex_t>());

        map_id<<<vertex_grid_size, block_size>>>(edge_count, vertex_count, d_idArr, d_idMapArr);
        HRR(hipDeviceSynchronize());

        reassign_id<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_idMapArr, d_srcArr, d_dstArr);
        HRR(hipDeviceSynchronize());

        thrust::device_ptr<vertex_t> d_dst_ptr((vertex_t *)d_dstArr);
        thrust::device_ptr<vertex_t> d_src_ptr((vertex_t *)d_srcArr);
        thrust::sort_by_key(d_src_ptr, d_src_ptr + edge_count, d_dstArr);

        cal_offset<<<edge_grid_size, block_size>>>(edge_count, vertex_count, d_srcArr, d_offsetArr);
        HRR(hipDeviceSynchronize());
    }
    double t_end = wtime();

    cout << "compute time spent " << (t_end - t_start) / iteration << " s, iterations " << iteration << endl;

    HRR(hipMemcpy(offsetArr, d_offsetArr, sizeOffsetList, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(srcArr, d_srcArr, sizeEdgeArr, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(dstArr, d_dstArr, sizeEdgeArr, hipMemcpyDeviceToHost));
    printMaxDegree("after compute");

    hipFree(d_degreeArr);
    hipFree(d_idArr);
    hipFree(d_idMapArr);
    hipFree(d_offsetArr);
    hipFree(d_srcArr);
    hipFree(d_dstArr);
}

__global__ void record_id(int edge_count, int vertex_count, int *d_idArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }
    d_idArr[i] = i;
}

__global__ void show_arr(int edge_count, int vertex_count, int *d_degreeArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }
    if (i < 10)
    // if (317079 - i <= 100)
    {
        // printf("%d\t%d\n", i, d_degreeArr[d_arr[i]]);
        // printf("%d\t%d\n", i, d_degreeArr[210749]);
        // printf("%d\t%d\n", i, d_degreeArr[210749]);
        // printf("%d\t%d\n", i, d_degreeArr[210752]);
        // printf("%d\t%d\n", i, d_degreeArr[2 * i + 1]);
        // printf("%d\t%d\t%d\n", i, d_arr[i], d_degreeArr[i]);
    }
}

__global__ void map_id(int edge_count, int vertex_count, int *d_idArr, int *d_idMapArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vertex_count) {
        return;
    }
    d_idMapArr[d_idArr[i]] = i;
}

__global__ void cal_degree(int edge_count, int vertex_count, int *d_degreeArr, int *d_srcArr, int *d_dstArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_srcArr[i];
    int dst = d_dstArr[i];

    atomicAdd(d_degreeArr + src, 1);
    atomicAdd(d_degreeArr + dst, 1);
}

__global__ void cal_src_out_degree(int edge_count, int vertex_count, int *d_degreeArr, int *d_srcArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    atomicAdd(d_degreeArr + d_srcArr[i], 1);
}

__global__ void redirect_edge(int edge_count, int vertex_count, int *d_degreeArr, int *d_srcArr, int *d_dstArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    int src = d_srcArr[i];
    int dst = d_dstArr[i];
    // redirect edge
    if (d_degreeArr[src] > d_degreeArr[dst] || (d_degreeArr[src] == d_degreeArr[dst] && src > dst)) {
        d_dstArr[i] = src;
        d_srcArr[i] = dst;
    }
}

__global__ void reassign_id(int edge_count, int vertex_count, int *d_idMapArr, int *d_srcArr, int *d_dstArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    d_srcArr[i] = d_idMapArr[d_srcArr[i]];
    d_dstArr[i] = d_idMapArr[d_dstArr[i]];
}

__global__ void unzip_edge(int edge_count, int vertex_count, int *d_edgeArr, int *d_srcArr, int *d_dstArr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= edge_count) {
        return;
    }
    d_srcArr[i] = d_edgeArr[i * 2 + 1];
    d_dstArr[i] = d_edgeArr[i * 2];
}

__global__ void cal_offset(int edge_count, int vertex_count, int *d_srcArr, index_t *d_offsetArr) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i <= edge_count; i += step) {
        int prev = i > 0 ? d_srcArr[i - 1] : -1;
        int next = i < edge_count ? d_srcArr[i] : vertex_count;
        // 前一个元素小于后一个元素，才有可能出现 offset 的计算
        for (int j = prev + 1; j <= next; ++j) d_offsetArr[j] = i;
    }
}

void comparasion(vertex_t *d_degreeArr, vertex_t *d_srcArr, vertex_t *d_dstArr, vertex_t *d_edgeArr, index_t *d_offsetArr, vertex_t *degreeArr,
                 vertex_t *srcArr, vertex_t *dstArr, vertex_t *edgeArr, index_t *offsetArr) {
    vertex_t *degreeArr2 = (vertex_t *)malloc(sizeof(vertex_t) * vertex_count);
    vertex_t *srcArr2 = (vertex_t *)malloc(sizeof(vertex_t) * edge_count);
    vertex_t *dstArr2 = (vertex_t *)malloc(sizeof(vertex_t) * edge_count);
    vertex_t *edgeArr2 = (vertex_t *)malloc(sizeof(vertex_t) * edge_count * 2);
    vertex_t *offsetArr2 = (vertex_t *)malloc(sizeof(index_t) * (vertex_count + 1));

    HRR(hipMemcpy(degreeArr2, d_degreeArr, sizeof(vertex_t) * vertex_count, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(srcArr2, d_srcArr, sizeof(vertex_t) * edge_count, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(dstArr2, d_dstArr, sizeof(vertex_t) * edge_count, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(edgeArr2, d_edgeArr, sizeof(vertex_t) * edge_count * 2, hipMemcpyDeviceToHost));
    HRR(hipMemcpy(offsetArr2, d_offsetArr, sizeof(index_t) * (vertex_count + 1), hipMemcpyDeviceToHost));

    for (int i = 0; i < edge_count * 2; i++) {
        if (i < vertex_count && degreeArr2[i] != degreeArr[i]) {
            cout << "degree " << i << "  " << degreeArr2[i] << "  " << degreeArr[i] << endl;
        }
        if (i < edge_count && srcArr2[i] != srcArr[i]) {
            cout << "src " << i << "  " << srcArr2[i] << "  " << srcArr[i] << endl;
        }
        if (i < edge_count && dstArr2[i] != dstArr[i]) {
            cout << "dst " << i << "  " << dstArr2[i] << "  " << dstArr[i] << endl;
        }
        if (edgeArr2[i] != edgeArr[i]) {
            cout << "edge " << i << "  " << edgeArr2[i] << "  " << edgeArr[i] << endl;
        }
        if (i < vertex_count + 1 && offsetArr2[i] != offsetArr[i]) {
            // cout << "offset " << i << "  " << offsetArr2[i] << "  " << offsetArr[i] << endl;
        }
    }

    free(degreeArr2);
    free(srcArr2);
    free(dstArr2);
    free(edgeArr2);
    free(offsetArr2);
}

void riddcsr(string inPrefix, string outPrefix) {
    // 将边加载到图中
    loadgraph(inPrefix);
    cout << "loadok" << endl;

    compute();

    writeback(outPrefix);
    cout << "writebackok" << endl;
}

int main(int argc, char *argv[]) {
    string inPrefix = argv[1];
    string outPrefix = argv[2];

    cout << "inPath: " << inPrefix << endl;
    cout << "outPath: " << outPrefix << endl;
    riddcsr(inPrefix, outPrefix);
    cout << endl;
}

bool cmp1(vertex &a, vertex &b) { return a.degree < b.degree; }

bool cmp2(vertex &a, vertex &b) { return a.degree > b.degree; }

bool cmp3(edge &a, edge &b) { return a.u < b.u || (a.u == b.u && a.v < b.v); }

int printMaxDegree(string str) {
    int maxDegre = 0;
    for (index_t i = 1; i <= vertex_count; i++) {
        if (offsetArr[i] - offsetArr[i - 1] > maxDegre) {
            maxDegre = offsetArr[i] - offsetArr[i - 1];
        }
    }
    cout << str << " max degree :" << maxDegre << endl;
    return maxDegre;
}